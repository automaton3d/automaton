#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for vector addition
__global__ void add(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

int main() {
    const int N = 1024;
    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Allocate GPU memory
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    // Copy data to GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel (N threads, 256 threads per block)
    add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Copy results back to CPU
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display results
    for (int i = 0; i < 10; i++) {
        std::cout << "c[" << i << "] = " << c[i] << "\n";
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
