#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "automaton.h"

__device__ hiprandState state;

__device__ void initCell(Cell* cell, int floor, int xyz)
{
    hiprand_init(0, xyz, 0, &state);
    int x = xyz & (SIDE-1);
    int y = (xyz >> ORDER) & (SIDE - 1);
    int z = (xyz >> (2 * ORDER));
    cell->floor = floor;                        // DEBUG
    //
    // Variable wrap is a hint for wrapping in other kernels
    //
    cell->wrap = 0;
    if (z == 0)
        cell->wrap |= 0x08;
    else if (z == SIDE - 1)
        cell->wrap |= 0x04;
    if (y == 0)
        cell->wrap |= 0x20;
    else if (y == SIDE - 1)
        cell->wrap |= 0x10;
    if (x == 0)
        cell->wrap |= 0x80;
    else if (x == SIDE - 1)
        cell->wrap |= 0x40;
    if (floor == SIDE2 - 1)
        cell->wrap |= 0x02;
    if (!cell->active)
        cell->wrap |= 0x01;
    //
    cell->t = 0;
    cell->noise = floor;
    cell->f = 0;
    cell->b = 0;
    cell->synch = -1;
    cell->charge = 0;
    cell->ctrl = 0;
    RESET(cell->p);
    RESET(cell->s);
    if (z == SIDE/2 && (x + SIDE * y) == floor)
    {
        cell->f = 1;
        if (x < SIDE / 2)
        {
            cell->charge |= D_MASK;
        }
        else
        {
            cell->charge &= ~D_MASK;
        }
        //
        unsigned char tiling = (x % 2) ^ (y % 2);
        if (tiling)
        {
            cell->charge |= Q_MASK;
        }
        else
        {
            cell->charge |= C_MASK | W_MASK;
        }
        //
        // Initialize spin and momentum
        //
        if (x == SIDE - 1)
        {
            cell->s[2] = (cell->charge & D_MASK) ? -SIDE / 2 : +SIDE / 2;
            cell->p[2] = (floor % 2) ? +SIDE / 2 : -SIDE / 2;
        }
        else
        {
            /*
            switch (floor % 6)
            {
            case 0:
                cell->s[0] = +SIDE / 2;
                cell->p[1] = +SIDE / 2;
                break;
            case 1:
                cell->s[0] = -SIDE / 2;
                cell->p[1] = -SIDE / 2;
                break;
            case 2:
                cell->s[1] = +SIDE / 2;
                cell->p[2] = +SIDE / 2;
                break;
            case 3:
                cell->s[1] = -SIDE / 2;
                cell->p[2] = -SIDE / 2;
                break;
            case 4:
                cell->s[2] = +SIDE / 2;
                cell->p[0] = +SIDE / 2;
                break;
            case 5:
                cell->s[2] = -SIDE / 2;
                cell->p[0] = -SIDE / 2;
                break;
            }
            */
            switch (hiprand(&state) % 3)
            {
            case 0: // xy
                cell->s[0] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[0] *= hiprand(&state) % SIDE/2;
                cell->s[1] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[1] *= hiprand(&state) % SIDE / 2;
                cell->s[2] = hiprand(&state) % 2 == 0 ? SIDE / 2 : -SIDE / 2;
                break;
            case 1: // yz
                cell->s[0] = hiprand(&state) % 2 == 0 ? SIDE / 2 : -SIDE / 2;
                cell->s[1] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[1] *= hiprand(&state) % SIDE / 2;
                cell->s[2] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[2] *= hiprand(&state) % SIDE / 2;
                break;
            case 2: // zx
                cell->s[0] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[0] *= hiprand(&state) % SIDE / 2;
                cell->s[1] = hiprand(&state) % 2 == 0 ? SIDE / 2 : -SIDE / 2;
                cell->s[2] = hiprand(&state) % 2 == 0 ? +1 : -1;
                cell->s[2] *= hiprand(&state) % SIDE / 2;
                break;
            }
            cell->p[0] = -cell->s[0];
            cell->p[1] = -cell->s[1];
            cell->p[2] = -cell->s[2];
        }
    }
    //
    RESET(cell->o);
    cell->flash = false;
    cell->sine = 0;
    cell->cosine = SIDE / 2;
}

__global__ void hologram(Cell* lattice)
{
    // Calculate 3d index
    //
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        // Build one lattice
        //
        Cell* cell = lattice + xyz;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = true;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
        //
        // Buid the dual lattice
        //
        cell += SIDE2 * SIDE3;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = false;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
    }
}

