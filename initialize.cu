#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "automaton.h"

__device__ void initCell(Cell* cell, int floor, int xyz)
{
    int x = xyz & (SIDE-1);
    int y = (xyz >> ORDER) & (SIDE - 1);
    int z = (xyz >> (2 * ORDER));
    cell->floor = floor;                        // DEBUG
    //
    // Variable type is a hint for wrapping in other kernels
    //
    cell->type = 0;
    if (z == 0)
        cell->type |= 0x08;
    else if (z == SIDE - 1)
        cell->type |= 0x04;
    if (y == 0)
        cell->type |= 0x20;
    else if (y == SIDE - 1)
        cell->type |= 0x10;
    if (x == 0)
        cell->type |= 0x80;
    else if (x == SIDE - 1)
        cell->type |= 0x40;
    if (floor == SIDE2 - 1)
        cell->type |= 0x02;
    if (!cell->active)
        cell->type |= 0x01;
    //
    cell->t = 0;
    cell->noise = floor;
    cell->f = 0;
    cell->b = 0;
    cell->synch = -1;
    cell->charge = 0;
    cell->ctrl = 0;
    RESET(cell->p);
    RESET(cell->s);
    if (z == SIDE/2 && (x + SIDE * y) == floor)
    {
        cell->f = 1;
        if (x < SIDE / 2)
        {
            cell->charge |= D_MASK;
        }
        else
        {
            cell->charge &= ~D_MASK;
        }
        //
        unsigned char tiling = (x % 2) ^ (y % 2);
        if (tiling)
        {
            cell->charge |= Q_MASK;
        }
        else
        {
            cell->charge |= C_MASK | W_MASK;
        }
        //
        // Initialize spin and momentum
        //
        if (x == SIDE - 1)
        {
            cell->s[2] = (cell->charge & D_MASK) ? -SIDE / 2 : +SIDE / 2;
            cell->p[2] = (floor % 2) ? +SIDE / 2 : -SIDE / 2;
        }
        else
        {
            switch (floor % 6)
            {
            case 0:
                cell->s[0] = +SIDE / 2;
                cell->p[1] = +SIDE / 2;
                break;
            case 1:
                cell->s[0] = -SIDE / 2;
                cell->p[1] = -SIDE / 2;
                break;
            case 2:
                cell->s[1] = +SIDE / 2;
                cell->p[2] = +SIDE / 2;
                break;
            case 3:
                cell->s[1] = -SIDE / 2;
                cell->p[2] = -SIDE / 2;
                break;
            case 4:
                cell->s[2] = +SIDE / 2;
                cell->p[0] = +SIDE / 2;
                break;
            case 5:
                cell->s[2] = -SIDE / 2;
                cell->p[0] = -SIDE / 2;
                break;
            }
        }
    }
    //
    COPY(cell->pole, cell->p);
    RESET(cell->o);
    cell->sine = 0;
    cell->cosine = SIDE / 2;
}

__global__ void hologram(Cell* lattice)
{
    // Calculate 3d index
    //
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        // Build one lattice
        //
        Cell* cell = lattice + xyz;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = true;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
        //
        // Buid the dual lattice
        //
        cell += SIDE2 * SIDE3;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = false;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
    }
}

