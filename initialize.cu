#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "automaton.h"

__device__ hiprandState state;

__device__ void initCell(Cell* cell, int floor, int xyz)
{
    hiprand_init(0, xyz, 0, &state);
    int x = xyz & (SIDE-1);
    int y = (xyz >> ORDER) & (SIDE - 1);
    int z = (xyz >> (2 * ORDER));
    cell->floor = floor;
    //
    // Variable wrap is a hint for wrapping in other kernels
    //
    cell->wrap = 0;
    if (z == 0)
        cell->wrap |= 0x08;
    else if (z == SIDE - 1)
        cell->wrap |= 0x04;
    if (y == 0)
        cell->wrap |= 0x20;
    else if (y == SIDE - 1)
        cell->wrap |= 0x10;
    if (x == 0)
        cell->wrap |= 0x80;
    else if (x == SIDE - 1)
        cell->wrap |= 0x40;
    if (floor == SIDE2 - 1)
        cell->wrap |= 0x02;
    if (!cell->active)
        cell->wrap |= 0x01;
    //
    // Initialize simple variables
    //
    cell->t = 0;
    cell->noise = floor;
    cell->f = 0;
    cell->b = 0;
    cell->synch = 0;
    cell->charge = 0;
    cell->ctrl = 0;
    cell->flash = 0;
    cell->sine = 0;
    cell->cosine = SIDE / 2;
    RESET(cell->p);
    RESET(cell->s);
    RESET(cell->o);
    //
    // Cell belongs to the hologram?
    //
    if (z == SIDE/2 && (x + SIDE * y) == floor)
    {
        // Initialize charges, spin and momentum
        //
        cell->f = 1;
        if (x < SIDE / 2)
        {
            cell->charge |= D_MASK;
        }
        //
        unsigned char tiling = (x % 2) ^ (y % 2);
        if (tiling)
        {
            cell->charge |= Q_MASK;
        }
        else
        {
            cell->charge |= C_MASK | W_MASK;
        }
        //
        // Initialize spin and momentum
        //
        if (x == SIDE - 1)
        {
            // Enforce monopole
            //
            cell->s[2] = (cell->charge & D_MASK) ? -SIDE / 2 : +SIDE / 2;
            cell->p[2] = (floor % 2) ? +SIDE / 2 : -SIDE / 2;
        }
        else
        {
            // Isotropic distribution
            //
            switch ((x + SIDE*y) % 6)
            {
            case 0:
                cell->p[0] = x - SIDE / 2;
                cell->p[1] = y - SIDE / 2;
                cell->p[2] = SIDE / 2;
                //
                cell->s[0] = y - SIDE / 2;
                cell->s[1] = x - SIDE / 2;
                cell->s[2] = -SIDE / 2;
                break;
            case 1:
                cell->p[0] = SIDE / 2;
                cell->p[1] = x - SIDE / 2;
                cell->p[2] = y - SIDE / 2;
                //
                cell->s[0] = -SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = x - SIDE / 2;
                break;
            case 2:
                cell->p[0] = y - SIDE / 2;
                cell->p[1] = SIDE / 2;
                cell->p[2] = x - SIDE / 2;
                //
                cell->s[0] = x - SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = SIDE / 2;
                break;
            case 3:
                cell->p[0] = y - SIDE / 2;
                cell->p[1] = x - SIDE / 2;
                cell->p[2] = -SIDE / 2;
                //
                cell->s[0] = x - SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = SIDE / 2;
                break;
            case 4:
                cell->p[0] = -SIDE / 2;
                cell->p[1] = y - SIDE / 2;
                cell->p[2] = x - SIDE / 2;
                //
                cell->s[0] = SIDE / 2;
                cell->s[1] = x - SIDE / 2;
                cell->s[2] = y - SIDE / 2;
                break;
            case 5:
                cell->p[0] = x - SIDE / 2;
                cell->p[1] = -SIDE / 2;
                cell->p[2] = y - SIDE / 2;
                //
                cell->s[0] = y - SIDE / 2;
                cell->s[1] = SIDE / 2;
                cell->s[2] = x - SIDE / 2;
                break;
            }
        }
    }
}

__global__ void hologram(Cell* lattice)
{
    // Calculate 3d index
    //
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        // Build one lattice
        //
        Cell* cell = lattice + xyz;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = true;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
        //
        // Buid the dual lattice
        //
        cell += SIDE2 * SIDE3;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = false;
            initCell(cell, floor, xyz);
            cell = nextV(cell);
        }
    }
}

