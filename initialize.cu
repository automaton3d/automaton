#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "automaton.h"

__device__ void initLattice(int idx, struct Cell* cell, bool active)
{
    int noise = 0;
    for (int z = 0; z < SIDE; z++)
    {
        for (int y = 0; y < SIDE; y++)
        {
            for (int x = 0; x < SIDE; x++)
            {
                cell->active = active;
                cell->t = 0;
                cell->noise = noise;
                cell->f = 0;
                cell->synch = 0;
                RESET(cell->p);
                RESET(cell->s);
                if (z == 0 && (x + SIDE * y) == idx)
                {
                    cell->f = 1;
                    if (x < SIDE / 2)
                    {
                        cell->d = true;
                    }
                    else
                    {
                        cell->d = false;
                    }
                    //
                    unsigned char tiling = (x % 2) ^ (y % 2);
                    if (tiling)
                    {
                        cell->c = 0;
                        cell->w = false;
                        cell->q = true;
                    }
                    else
                    {
                        cell->c = 7;
                        cell->w = true;
                        cell->q = false;
                    }
                    //
                    // Initialize spin and momentum
                    //
                    if (x == SIDE - 1)
                    {
                        cell->s[2] = (cell->d) ? -SIDE / 2 : +SIDE / 2;
                        cell->p[2] = (noise % 2) ? +SIDE / 2 : -SIDE / 2;
                    }
                    else
                    {
                        switch (noise % 6)
                        {
                        case 0:
                            cell->s[0] = +SIDE / 2;
                            cell->p[1] = +SIDE / 2;
                            break;
                        case 1:
                            cell->s[0] = -SIDE / 2;
                            cell->p[1] = -SIDE / 2;
                            break;
                        case 2:
                            cell->s[1] = +SIDE / 2;
                            cell->p[2] = +SIDE / 2;
                            break;
                        case 3:
                            cell->s[1] = -SIDE / 2;
                            cell->p[2] = -SIDE / 2;
                            break;
                        case 4:
                            cell->s[2] = +SIDE / 2;
                            cell->p[0] = +SIDE / 2;
                            break;
                        case 5:
                            cell->s[2] = -SIDE / 2;
                            cell->p[0] = -SIDE / 2;
                            break;
                        }
                    }
                }
                //
                cell->sine = 0;
                cell->cosine = SIDE / 2;
                //
                if (x == SIDE - 1)
                    cell->px = cell - (SIDE - 1);
                else
                    cell->px = cell + 1;
                //
                if (x == 0)
                    cell->nx = cell + (SIDE - 1);
                else
                    cell->nx = cell - 1;
                if (y == SIDE - 1)
                    cell->py = cell - (SIDE - 1) * SIDE;
                else
                    cell->py = cell + SIDE;
                if (y == 0)
                    cell->ny = cell + (SIDE - 1);
                else
                    cell->ny = cell - SIDE;
                if (z == SIDE - 1)
                    cell->pz = cell - (SIDE - 1) * SIDE2;
                else
                    cell->pz = cell + SIDE2;
                if (z == 0)
                    cell->nz = cell + (SIDE - 1) * SIDE2;
                else
                    cell->nz = cell - SIDE2;
                if (noise == SIDE2 - 1)
                    cell->v = cell - (SIDE2 - 1) * SIDE3;
                else
                    cell->v = cell + SIDE3;
                //
                // Neighbor
                //
                if(active)
                    cell->h = cell + SIDE3 * SIDE2;
                else
                    cell->h = cell - SIDE3 * SIDE2;
                //
                // Elevator
                //
                if (idx == SIDE2 - 1)
                    cell->v = cell - (SIDE2 - 1) * SIDE3;
                else
                    cell->v = cell + SIDE3;
                //
                noise++;
                cell++;
            }
        }
    }
}

__global__ void hologram(struct Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < SIDE2)
    {
        struct Cell* cell = lattice + idx * (long long) SIDE3;
        initLattice(idx, cell, true);
        cell += SIDE2 * SIDE3;
        initLattice(idx, cell, false);
    }
}

