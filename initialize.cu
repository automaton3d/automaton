#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "automaton.h"

__device__ void initLattice(int idx, Cell* cell, bool active)
{
    for (int z = 0; z < SIDE; z++)
    {
        for (int y = 0; y < SIDE; y++)
        {
            for (int x = 0; x < SIDE; x++)
            {
                cell->type = 0;
                if (z == 0)
                    cell->type |= 0x08;
                else if (z == SIDE - 1)
                    cell->type |= 0x04;
                if (y == 0)
                    cell->type |= 0x20;
                else if (y == SIDE - 1)
                    cell->type |= 0x10;
                if (x == 0)
                    cell->type |= 0x80;
                else if (x == SIDE - 1)
                    cell->type |= 0x40;
                if (idx == SIDE2 - 1)
                    cell->type |= 0x02;
                if (!active)
                    cell->type |= 0x01;
                //
                cell->active = active;
                cell->t = 0;
                cell->noise = idx;
                cell->f = 0;
                cell->b = 0;
                cell->synch = -1;
                cell->charge = 0;
                cell->ctrl = 0;
                RESET(cell->p);
                RESET(cell->s);
                if (z == 0 && (x + SIDE * y) == idx)
                {
                    cell->f = 1;
                    if (x < SIDE / 2)
                    {
                        cell->charge |= D_MASK;
                    }
                    else
                    {
                        cell->charge &= ~D_MASK;
                    }
                    //
                    unsigned char tiling = (x % 2) ^ (y % 2);
                    if (tiling)
                    {
                        cell->charge |= Q_MASK;
                    }
                    else
                    {
                        cell->charge |= C_MASK | W_MASK;
                    }
                    //
                    // Initialize spin and momentum
                    //
                    if (x == SIDE - 1)
                    {
                        cell->s[2] = (cell->charge & D_MASK) ? -SIDE / 2 : +SIDE / 2;
                        cell->p[2] = (idx % 2) ? +SIDE / 2 : -SIDE / 2;
                    }
                    else
                    {
                        switch (idx % 6)
                        {
                        case 0:
                            cell->s[0] = +SIDE / 2;
                            cell->p[1] = +SIDE / 2;
                            break;
                        case 1:
                            cell->s[0] = -SIDE / 2;
                            cell->p[1] = -SIDE / 2;
                            break;
                        case 2:
                            cell->s[1] = +SIDE / 2;
                            cell->p[2] = +SIDE / 2;
                            break;
                        case 3:
                            cell->s[1] = -SIDE / 2;
                            cell->p[2] = -SIDE / 2;
                            break;
                        case 4:
                            cell->s[2] = +SIDE / 2;
                            cell->p[0] = +SIDE / 2;
                            break;
                        case 5:
                            cell->s[2] = -SIDE / 2;
                            cell->p[0] = -SIDE / 2;
                            break;
                        }
                    }
                }
                //
                COPY(cell->pole, cell->p);
                RESET(cell->o);
                cell->z = z;
                cell->sine = 0;
                cell->cosine = SIDE / 2;
                //
                cell++;
            }
        }
    }
}

__global__ void hologram(Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < SIDE2)
    {
        Cell* cell = lattice + idx * (long long) SIDE3;
        initLattice(idx, cell, true);
        cell += SIDE2 * SIDE3;
        initLattice(idx, cell, false);
    }
}

