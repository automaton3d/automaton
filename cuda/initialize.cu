#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "automaton.cuh"

__device__ hiprandState state;

__device__ void initCell(Cell* cell, int xyz)
{
    hiprand_init(0, xyz, 0, &state);
    int x = xyz & (SIDE-1);
    int y = (xyz >> ORDER) & (SIDE - 1);
    int z = (xyz >> (2 * ORDER));
    //
    // Variable wrap is a hint for wrapping in other kernels
    //
    cell->wrap = 0;
    if (z == 0)
        cell->wrap |= 0x08;
    else if (z == SIDE - 1)
        cell->wrap |= 0x04;
    if (y == 0)
        cell->wrap |= 0x20;
    else if (y == SIDE - 1)
        cell->wrap |= 0x10;
    if (x == 0)
        cell->wrap |= 0x80;
    else if (x == SIDE - 1)
        cell->wrap |= 0x40;
    if (cell->floor == SIDE2 - 1)
        cell->wrap |= 0x02;
    if (!cell->active)
        cell->wrap |= 0x01;
    //
    // Initialize simple variables
    //
    cell->t = 0;
    cell->noise = cell->floor;
    cell->f = 0;
    cell->a = cell->floor;
    cell->sync = 0;
    cell->chrg = 0x00;
    cell->flash = 0;
    cell->v = 0;
    cell->u = SIDE2 / 2;
    RESET(cell->p);
    RESET(cell->s);
    RESET(cell->o);
    //
    // Cell belongs to the hologram?
    //
    if (z == SIDE/2 && (x + SIDE * y) == cell->floor)
    {
        // Initialize charges, spin and momentum
        //
        cell->f = 1;
        if (x < SIDE / 2)
        {
            cell->chrg |= D_MASK;
        }
        //
        unsigned char tiling = (x % 2) ^ (y % 2);
        if (tiling)
        {
            cell->chrg |= Q_MASK;
        }
        else
        {
            cell->chrg |= C_MASK | W_MASK;
        }
        //
        // Initialize spin and momentum
        //
        if (x == SIDE - 1)
        {
            // Enforce monopole
            //
            cell->s[2] = (cell->chrg & D_MASK) ? -SIDE / 2 : +SIDE / 2;
            cell->p[2] = (cell->floor % 2) ? +SIDE / 2 : -SIDE / 2;
        }
        else
        {
            // Isotropic distribution
            //
            switch ((x + SIDE*y) % 6)
            {
            case 0:
                cell->p[0] = x - SIDE / 2;
                cell->p[1] = y - SIDE / 2;
                cell->p[2] = SIDE / 2;
                //
                cell->s[0] = y - SIDE / 2;
                cell->s[1] = x - SIDE / 2;
                cell->s[2] = -SIDE / 2;
                break;
            case 1:
                cell->p[0] = SIDE / 2;
                cell->p[1] = x - SIDE / 2;
                cell->p[2] = y - SIDE / 2;
                //
                cell->s[0] = -SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = x - SIDE / 2;
                break;
            case 2:
                cell->p[0] = y - SIDE / 2;
                cell->p[1] = SIDE / 2;
                cell->p[2] = x - SIDE / 2;
                //
                cell->s[0] = x - SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = SIDE / 2;
                break;
            case 3:
                cell->p[0] = y - SIDE / 2;
                cell->p[1] = x - SIDE / 2;
                cell->p[2] = -SIDE / 2;
                //
                cell->s[0] = x - SIDE / 2;
                cell->s[1] = y - SIDE / 2;
                cell->s[2] = SIDE / 2;
                break;
            case 4:
                cell->p[0] = -SIDE / 2;
                cell->p[1] = y - SIDE / 2;
                cell->p[2] = x - SIDE / 2;
                //
                cell->s[0] = SIDE / 2;
                cell->s[1] = x - SIDE / 2;
                cell->s[2] = y - SIDE / 2;
                break;
            case 5:
                cell->p[0] = x - SIDE / 2;
                cell->p[1] = -SIDE / 2;
                cell->p[2] = y - SIDE / 2;
                //
                cell->s[0] = y - SIDE / 2;
                cell->s[1] = SIDE / 2;
                cell->s[2] = x - SIDE / 2;
                break;
            }
        }
    }
}

__shared__ Cell *mirror;

__global__ void hologram(Cell* lattice)
{
    // Calculate 3d index
    //
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        // Build one lattice
        //
        Cell* cell = lattice + xyz;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = true;
            cell->floor = floor;
            initCell(cell, xyz);
            cell = nextV(cell);
        }
        //
        // Buid the dual lattice
        //
        cell += SIDE2 * SIDE3;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            cell->active = false;
            cell->floor = floor;
            initCell(cell, xyz);
            cell = nextV(cell);
        }
    }
}

