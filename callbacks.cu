#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#include "hip/hip_runtime.h"
#include ""

#define GLEW_STATIC

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cglm/vec3.h"

#include "automaton.h"

extern unsigned int shaderProgram;
extern unsigned int vao;
extern vec3 colors[];
extern struct hipGraphicsResource* cuda_resource;
extern hipError_t cudaStatus;
extern struct Cell* dev_lattice;
extern float yaw, pitch;

int step = 0;

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
        /* Exit on escape key press */
        case '\x1B':
        {
            closeApp();
            exit(EXIT_SUCCESS);
            break;
        }
        case 'A':
        case 'a':
            yaw += 5.0;
            updateCamera();
            break;
        case 'S':
        case 's':
            yaw -= 5.0;
            updateCamera();
            break;
        case 'W':
        case 'w':
            pitch += 3.0;
            updateCamera();
            break;
        case 'D':
        case 'd':
            pitch -= 3.0;
            updateCamera();
            break;
    }
}

void display()
{
    cudaStatus = hipGraphicsMapResources(1, &cuda_resource, 0);
    if (cudaStatus != hipSuccess)
    {
        puts("mapping error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    size_t num_bytes;
    void* dev_color;
    cudaStatus = hipGraphicsResourceGetMappedPointer(&dev_color, &num_bytes, cuda_resource);
    if (cudaStatus != hipSuccess)
    {
        puts("pointer error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    // "dev_color" points to the GPU memory data store of colorVBO maped by cuda_resource 
    //
    interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interop error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    cudaStatus = hipGraphicsUnmapResources(1, &cuda_resource, 0);
    if (cudaStatus != hipSuccess)
    {
        puts("unmapping error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    glClearColor(0.5, 0.5, 0.5, 1);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glUseProgram(shaderProgram);
    glBindVertexArray(vao);
    glDrawArraysInstanced(GL_POINTS, 0, 6, SIDE3);
    glBindVertexArray(0);
    //
    // Draw text
    //
    glWindowPos2i(10, 600 - 30);
    glColor4f(0.0f, 0.0f, 1.0f, 1.0f);
    glColor3f(1, 0, 0);
    char s[12];
    sprintf(s, "Step: %d", step);
    glutBitmapString(GLUT_BITMAP_HELVETICA_18, (const unsigned char*) s);
    //
    glutSwapBuffers();
}

void animation()
{
    commute << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("commute error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    /*
    compare << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("compare error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    */
    replicate << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("replicate error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    /*
    interact << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interact error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    */
    expand << <GRID1, BLOCK1 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("expand error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    // Generate graphics
    //
    display();
    step++;
}

