#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#include "hip/hip_runtime.h"
#include ""

#define GLEW_STATIC

#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cglm/vec3.h"

#include "automaton.h"

extern unsigned int shaderProgram;
extern unsigned int vao;
extern vec3 colors[];
extern struct hipGraphicsResource* cuda_resource;
extern hipError_t cudaStatus;
extern Cell *host_lattice, *dev_lattice;
extern float yaw, pitch;
extern DWORD start;
int step = 0;

boolean flag;

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
        /* Exit on escape key press */
        case '\x1B':
        {
            closeApp();
            exit(EXIT_SUCCESS);
            break;
        }
        case 'A':
        case 'a':
            yaw += 5.0;
            updateCamera();
            break;
        case 'S':
        case 's':
            yaw -= 5.0;
            updateCamera();
            break;
        case 'W':
        case 'w':
            pitch += 3.0;
            updateCamera();
            break;
        case 'D':
        case 'd':
            pitch -= 3.0;
            updateCamera();
            break;
        case '1':
            flag = !flag;
            break;
    }
}

void display()
{
    hipGraphicsMapResources(1, &cuda_resource, 0);
    size_t num_bytes;
    void* dev_color;
    hipGraphicsResourceGetMappedPointer(&dev_color, &num_bytes, cuda_resource);
    //
    if(flag)
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, true);
    else
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, false);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interop error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    hipGraphicsUnmapResources(1, &cuda_resource, 0);
    glClearColor(0.5, 0.5, 0.5, 1);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glUseProgram(shaderProgram);
    glBindVertexArray(vao);
    glDrawArraysInstanced(GL_POINTS, 0, 6, SIDE3);
    glBindVertexArray(0);
    //
    // Draw text
    //
    glWindowPos2i(10, 600 - 30);
    glColor4f(0.0f, 0.0f, 1.0f, 1.0f);
    glColor3f(1, 0, 0);
    char s[25];
    DWORD time = GetTickCount() - start;
    sprintf(s, "step=%d, time=%0.1f", step, time/1000.0);
    glutBitmapString(GLUT_BITMAP_HELVETICA_18, (const unsigned char*) s);
    //
    glutSwapBuffers();
}

void printResults()
{
    hipMemcpy(host_lattice, dev_lattice, 2 * SIDE2 * SIDE3 * sizeof(Cell), hipMemcpyDeviceToHost);
    Cell* cell = host_lattice;
    int i = 0;
    for (int h = 0; h < 2; h++)
        for (int v = 0; v < SIDE2; v++)
            for (int z = 0; z < SIDE; z++)
                for (int y = 0; y < SIDE; y++)
                    for (int x = 0; x < SIDE; x++)
                    {
                        if (!cell->active && v == 60 && !ISNULL(cell->p))
                        {
                            printf("%d: [%d, %d, (%d,%d)] v=%d, h=%d noise=%d p=[%d,%d,%d] o=[%d,%d,%d]\n", cell->t, x, y, z, cell->z, v, h, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2]);
                        }
                        cell++;
                        i++;
                    }

    fflush(stdout);
}

void animation()
{
    commute << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("commute error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    compare << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("compare error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    replicate << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("replicate error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    interact << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interact error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    expand << <GRID1, BLOCK1 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("expand error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //printResults();
    //
    // Generate graphics
    //
    display();
    //
//    Sleep(1000);
    step++;
}


#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#include "hip/hip_runtime.h"
#include ""

#define GLEW_STATIC

#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cglm/vec3.h"
#include "automaton.h"

extern unsigned int shaderProgram;
extern unsigned int vao;
extern vec3 colors[];
extern struct hipGraphicsResource* cuda_resource;
extern hipError_t cudaStatus;
extern Cell *host_lattice, *dev_lattice;
extern float yaw, pitch;
extern DWORD start;
int step = 0;

boolean flag;

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
        /* Exit on escape key press */
        case '\x1B':
        {
            closeApp();
            exit(EXIT_SUCCESS);
            break;
        }
        case 'A':
        case 'a':
            yaw += 5.0;
            updateCamera();
            break;
        case 'S':
        case 's':
            yaw -= 5.0;
            updateCamera();
            break;
        case 'W':
        case 'w':
            pitch += 3.0;
            updateCamera();
            break;
        case 'D':
        case 'd':
            pitch -= 3.0;
            updateCamera();
            break;
        case '1':
            flag = !flag;
            break;
    }
}

void display()
{
    hipGraphicsMapResources(1, &cuda_resource, 0);
    size_t num_bytes;
    void* dev_color;
    hipGraphicsResourceGetMappedPointer(&dev_color, &num_bytes, cuda_resource);
    //
    if(flag)
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, true);
    else
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, false);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interop error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    hipGraphicsUnmapResources(1, &cuda_resource, 0);
    glClearColor(0.5, 0.5, 0.5, 1);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glUseProgram(shaderProgram);
    glBindVertexArray(vao);
    glDrawArraysInstanced(GL_POINTS, 0, 6, SIDE3);
    glBindVertexArray(0);
    //
    // Draw text
    //
    glWindowPos2i(10, 600 - 30);
    char s[25];
    DWORD time = GetTickCount() - start;
    sprintf(s, "step=%d, time=%0.1f", step, time/1000.0);
    glutBitmapString(GLUT_BITMAP_HELVETICA_18, (const unsigned char*) s);
    //
    glutSwapBuffers();
}

void printResults(bool full)
{
    hipMemcpy(host_lattice, dev_lattice, 2 * SIDE2 * SIDE3 * sizeof(Cell), hipMemcpyDeviceToHost);
    Cell* cell = host_lattice;
    int i = 0;
    for (int h = 0; h < 2; h++)
        for (int v = 0; v < SIDE2; v++)
            for (int z = 0; z < SIDE; z++)
                for (int y = 0; y < SIDE; y++)
                    for (int x = 0; x < SIDE; x++)
                    {
                        if (full && cell->active && cell->f>0)
                        {
                            printf("%d: [%d, %d, %d] v=%d, h=%d noise=%d p=[%d,%d,%d] o=[%d,%d,%d] f=%d t2=%d syn=%d\n", cell->t, x, y, z, v, h, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->f, cell->t* cell->t, cell->synch);
                        }
                        else if (!cell->active && v == 60 && (!ISNULL(cell->p) || cell->f > 0))
                        {
                            printf("%d: [%d, %d, %d] v=%d, h=%d noise=%d p=[%d,%d,%d] o=[%d,%d,%d] f=%d t2=%d syn=%d\n", cell->t, x, y, z, v, h, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->f, cell->t*cell->t, cell->synch);
                        }
                        cell++;
                        i++;
                    }
    printf("step %d\n", step);
    fflush(stdout);
}

void animation()
{
    commute << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("commute error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    compare << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("compare error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    replicate << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("replicate error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    interact << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interact error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    expand << <GRID1, BLOCK1 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("expand error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //printResults(false);
    //
    // Generate graphics
    //
    display();
    //
    //Sleep(1000);
    step++;
}

