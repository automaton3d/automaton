#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#include "hip/hip_runtime.h"
#include ""

#define GLEW_STATIC

#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cglm/vec3.h"
#include "automaton.h"

extern unsigned int shaderProgram;
extern unsigned int vao;
extern struct hipGraphicsResource* cuda_resource;
extern hipError_t cudaStatus;
extern Cell *host_lattice, *dev_lattice;
extern float yaw, pitch;
extern DWORD start;
int step = 0;

boolean flag;

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
        /* Exit on escape key press */
        case '\x1B':
        {
            closeApp();
            exit(EXIT_SUCCESS);
            break;
        }
        case 'A':
        case 'a':
            yaw += 5.0;
            updateCamera();
            break;
        case 'S':
        case 's':
            yaw -= 5.0;
            updateCamera();
            break;
        case 'W':
        case 'w':
            pitch += 3.0;
            updateCamera();
            break;
        case 'D':
        case 'd':
            pitch -= 3.0;
            updateCamera();
            break;
        case '1':
            flag = !flag;
            break;
    }
}

void display()
{
    cudaStatus = hipGraphicsMapResources(1, &cuda_resource, 0);
    if (cudaStatus != hipSuccess)
    {
        puts("map resources error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    size_t num_bytes;
    void* dev_color;
    cudaStatus = hipGraphicsResourceGetMappedPointer(&dev_color, &num_bytes, cuda_resource);
    if (cudaStatus != hipSuccess)
    {
        puts("get pointer error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    printf("numbytes=%d\n", num_bytes); fflush(stdout);
    //
    if(flag)
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, true);
    else
        interop << <GRID2, BLOCK2 >> > (dev_lattice, (vec3*)dev_color, false);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interop error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    hipGraphicsUnmapResources(1, &cuda_resource, 0);
    glClearColor(0.5, 0.5, 0.5, 1);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glUseProgram(shaderProgram);
    glBindVertexArray(vao);
    glDrawArraysInstanced(GL_POINTS, 0, 6, SIDE3);
    glBindVertexArray(0);
    //
    // Draw text
    //
    glWindowPos2i(10, 600 - 30);
    char s[25];
    DWORD time = GetTickCount() - start;
    sprintf(s, "step=%d, time=%0.1f", step, time/1000.0);
    glutBitmapString(GLUT_BITMAP_HELVETICA_18, (const unsigned char*) s);
    //
    glutSwapBuffers();
}

void printResults(bool full)
{
    hipMemcpy(host_lattice, dev_lattice, 2 * SIDE2 * SIDE3 * sizeof(Cell), hipMemcpyDeviceToHost);
    Cell* cell = host_lattice;
    printf("t=%d: (%c) [%d, %d, %d] v=%d, noise=%d p=[%d,%d,%d] o=[%d,%d,%d] pole=[%d,%d,%d] f=%d t2=%d syn=%d \t%s\n", cell->t, '?', 0, 0, 0, cell->b, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->pole[0], cell->pole[1], cell->pole[2], cell->f, cell->t * cell->t, cell->synch, "??");
    cell += SIDE2 * SIDE3;
    printf("t=%d: (%c) [%d, %d, %d] v=%d, noise=%d p=[%d,%d,%d] o=[%d,%d,%d] pole=[%d,%d,%d] f=%d t2=%d syn=%d \t%s\n", cell->t, '?', 0, 0, 0, cell->b, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->pole[0], cell->pole[1], cell->pole[2], cell->f, cell->t * cell->t, cell->synch, "??");
    cell = host_lattice;
    for (int v = 0; v < SIDE2; v++)
    {
        for (int z = 0; z < SIDE; z++)
            for (int y = 0; y < SIDE; y++)
                for (int x = 0; x < SIDE; x++)
                {
                    char act = cell->active ? 'A' : ' ';
                    char* arrow = ISNULL(cell->p) ? "" : "<---";
                    if (full)
                    {
                        if (cell->f > 0)
                            printf("t=%d: (%c) [%d, %d, %d] v=%d, noise=%d p=[%d,%d,%d] o=[%d,%d,%d] pole=[%d,%d,%d] f=%d t2=%d syn=%d \t%s\n", cell->t, act, x, y, z, cell->b, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->pole[0], cell->pole[1], cell->pole[2], cell->f, cell->t * cell->t, cell->synch, arrow);
                    }
                    else if (!cell->active && v == 60 && (!ISNULL(cell->p) || cell->f > 0))
                    {
                        printf("t=%d: (%c) [%d, %d, %d] v=%d, noise=%d p=[%d,%d,%d] o=[%d,%d,%d] pole=[%d,%d,%d] f=%d t2=%d syn=%d \t%s\n", cell->t, act, x, y, z, cell->b, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->pole[0], cell->pole[1], cell->pole[2], cell->f, cell->t * cell->t, cell->synch, arrow);
                    }
                    cell++;
                }
    }
    printf("step %d\n", step);
    fflush(stdout);
}

void animation()
{
    commute << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("commute error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    compare << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("compare error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    replicate << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("replicate error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    interact << <GRID2, BLOCK2 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interact error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    expand << <GRID1, BLOCK1 >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("expand error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    printResults(false);
    //
    // Generate graphics
    //
    display();
    //
    //Sleep(1000);
    step++;
}


