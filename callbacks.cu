#include "hip/hip_runtime.h"
#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#include "hip/hip_runtime.h"
#include ""

#define GLEW_STATIC

#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cglm/vec3.h"
#include "automaton.cuh"

extern unsigned int shaderProgram, axesProgram;
extern unsigned int gridVAO, axesVAO;
extern struct hipGraphicsResource* cuda_resource;
extern hipError_t cudaStatus;
extern Cell *host_lattice, *dev_lattice;
extern DWORD start;
int step = 0;

boolean flag;
int sublattice = FLOOR;

void display()
{
    glClearColor(0.5, 0.5, 0.5, 1);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    //
    glUseProgram(axesProgram);
    glBindVertexArray(axesVAO);
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);
    glDrawArraysInstanced(GL_LINES, 0, 12, SIDE3);
    //
    glUseProgram(shaderProgram);
    glBindVertexArray(gridVAO);
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);
    glDrawArraysInstanced(GL_TRIANGLES, 0, 36, SIDE3);
    glBindVertexArray(0);
    // 
    // Draw text
    //
    glWindowPos2i(10, 800 - 30);
    char s[100];
    DWORD time = GetTickCount() - start;
    sprintf(s, "step=%d, time=%0.1f light=%d floor=%d", step, time/1000.0, step/LIGHT, sublattice);
    glutBitmapString(GLUT_BITMAP_HELVETICA_18, (const unsigned char*) s);
    //
    glutSwapBuffers();
}

void printResults(bool full)
{
    hipMemcpy(host_lattice, dev_lattice, 2 * SIDE2 * SIDE3 * sizeof(Cell), hipMemcpyDeviceToHost);
    Cell* cell = host_lattice;
    cell += SIDE2 * SIDE3;
    cell = host_lattice;
    for (int v = 0; v < SIDE2; v++)
    {
        for (int z = 0; z < SIDE; z++)
            for (int y = 0; y < SIDE; y++)
                for (int x = 0; x < SIDE; x++)
                {
                    if(cell->f)
                        printf("t=%d: [%d, %d, %d] floor=%d, noise=%d p=[%d,%d,%d] o=[%d,%d,%d] f=%d\n", 
                            cell->t, x, y, z, cell->b, cell->noise, cell->p[0], cell->p[1], cell->p[2], cell->o[0], cell->o[1], cell->o[2], cell->f);
                    cell++;
                }
    }
    printf("step %d\n", step);
    fflush(stdout);
}

void updateVoxels()
{
    cudaStatus = hipGraphicsMapResources(1, &cuda_resource, 0);
    if (cudaStatus != hipSuccess)
    {
        puts("map resources error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    size_t num_bytes;
    void* dev_color;
    cudaStatus = hipGraphicsResourceGetMappedPointer(&dev_color, &num_bytes, cuda_resource);
    if (cudaStatus != hipSuccess)
    {
        puts("get pointer error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    if (flag)
        interop << <GRID, BLOCK >> > (dev_lattice, (vec3*)dev_color, -1);
    else
        interop << <GRID, BLOCK >> > (dev_lattice, (vec3*)dev_color, sublattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interop error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    hipGraphicsUnmapResources(1, &cuda_resource, 0);
   
}

void animation()
{
    commute << <GRID, BLOCK >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("commute error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    compare << <GRID, BLOCK >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("compare error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    replicate << <GRID, BLOCK >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("replicate error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    interact << <GRID, BLOCK >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("interact error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    expand << <GRID, BLOCK >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("expand error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    int h_count;
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    poincare << <GRID, BLOCK >> > (dev_lattice, d_count);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("Poincar� error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    //printf("BINGO! %d\n", h_count);

    if (h_count == 0)
        printf("Poincareh cycle: %ld\n", step);
    hipFree(d_count);
    //
    //printResults(false);
    //
    // Generate graphics
    //
    updateVoxels();
    display();
    //
    //Sleep(100);
    step++;
}

