#include "hip/hip_runtime.h"
/*
 * ratnum.c
 *
 *  Created on: 17 de out. de 2021
 *      Author: Alexandre
 */


#include	<stdio.h>
#include	<math.h>
#include	"ratnum.cuh"

static	void simplify(ratnum*);
static	long int euclid(long int, long int);


ratnum mkrat(long int a, long int b)
{
	ratnum r;
	r.num = a;
	r.den = b;
	simplify(&r);
	return r;
}

void showrat(ratnum x)
{
	double	rr;
	rr = (double)(x.num) / (double)(x.den);
	fprintf(stderr, "%ld/", x.num);
	fprintf(stderr, "%ld", x.den);
	fprintf(stderr, " (%20.10lf)\n", rr);
}

double rat2float(ratnum r)
{
	return (double)(r.num) / (double)(r.den);
}

ratnum addrat(ratnum a, ratnum b)
{
	ratnum r;
	r.num = a.num * b.den + b.num * a.den;
	r.den = a.den * b.den;
	simplify(&r);
	return r;
}

ratnum addinttorat(ratnum a, long int x)
{
	ratnum sum;
	sum.num = a.num + x * a.den;
	sum.den = a.den;
	simplify(&sum);
	return sum;
}

static void simplify(ratnum* a)
{
	long int gcd;
	gcd = euclid(a->num, a->den);
	a->num /= gcd;
	a->den /= gcd;
}

static long int euclid(long int a, long int b)
{
	if (b == 0)	return a;
	else		return euclid(b, a % b);
}

int ratcomp(ratnum* a, ratnum* b)
{
	return  a->den * b->num - a->num * b->den;
}

ratnum modrat(ratnum a, ratnum b)
{
	ratnum re = a;
	double q = floor((a.num * b.den) / (double)(a.den * b.num));
	ratnum t2 = mkrat(-q * b.den, b.num);
	re = addrat(re, t2);
	return re;
}

ratnum multrat(ratnum a, ratnum b)
{
	ratnum prod = mkrat(a.num * b.num, a.den * b.den);
	simplify(&prod);
	return prod;
}

int rat2int(ratnum r)
{
	return floor(r.num / (double)r.den);
}

