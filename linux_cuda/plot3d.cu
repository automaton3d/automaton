#include "hip/hip_runtime.h"
/*
 * plot3d.cu
 *
 * Implements a 3d graphics pipeline.
 * This fast and simple engine is only capable of projecting isolated points.
 */
#include "plot3d.h"
#include <math.h>
#include <sys/time.h>
#include "brick.h"
#include "common.h"
#include "utils.h"
#include "text.h"
#include "params.h"
#include "automaton.h"
#include "jpeg.h"

pthread_t display;
boolean img_changed = true;

JSAMPLE *image_buffer;

// Transformation matrix
//
static double m00, m01, m02;
static double m10, m11, m12;
static double m20, m21, m22;
static double m30, m31, m32;
//
// 3d fields
//
static Vector3d center;		// center of projection
static Vector3d pc;		// transformed center of projection
static Vector3d pen;		// plotting pen

static double distance;		// view distance
static double frontDistance;	// clipping plane
static double backDistance;	// clipping plane
//
// Window
//
static double wxl, wyl;
static double wxh, wyh;
static double vxl, vyl;
static double vxh, vyh;
static double wsx, wsy;
//
static int parallel = true;
static int clipping;
static double scale = 1100;//0.8;

boolean showAxes = true, showGrid, showBox = true;

Vector3d position, _position;	// view reference point
Vector3d direction, _direction;	// camera axis
Vector3d attitude;		// view-up direction
struct timeval begin;

// Rotation

static double theta = 0;
static int rot = false;

// Colors

char colors [3 * NPREONS + 21];
static char gridcolor = 24;
static char X, Y, Z;
static char BOX;

void initPalette()
{
	X = R;
	Y = G;
	Z = B;
	//
	BOX = 18;
	gridcolor = GRAY;
	//
	colors[0] = 0;
	colors[1] = 0;
	colors[2] = 0;
	//
	colors[3] = 0xff;
	colors[4] = 0xff;
	colors[5] = 0xff;
	//
	colors[6] = 0xff;
	colors[7] = 0;
	colors[8] = 0;
	//
	colors[9] = 0;
	colors[10] = 0xff;
	colors[11] = 0;
	//
	colors[12] = 0;
	colors[13] = 0;
	colors[14] = 0xff;
	//
	colors[15] = 0x44;
	colors[16] = 0x44;
	colors[17] = 0x44;
	//
	colors[18] = 0x99;
	colors[19] = 0x99;
	colors[20] = 0x99;
	//
	// Preon colors
	//
	int i;
	for(i = 0; i < 3 * NPREONS;)
	{
		char pos = i % 3;
		switch(pos)
		{
			case 0:
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				break;
			case 1: 
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				break;
			case 2:
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				colors[21 + i++] = 127 + (127 * (long) rand()) / RAND_MAX;
				break;
		}
	}
}


void newTransform3()
{
	m00 = 1;
	m01 = 0;
	m02 = 0;
	//
	m10 = 0;
	m11 = 1;
	m12 = 0;
	//
	m20 = 0;
	m21 = 0;
	m22 = 1;
	//
	m30 = 0;
	m31 = 0;
	m32 = 0;
}

void translate(double tx, double ty, double tz)
{
	m30 += tx;
	m31 += ty;
	m32 += tz;
}

void rotateX(double s, double c)
{
	double t;
	t = m01 * c - m02 * s;
	m02 = m01 * s + m02 * c;
	m01 = t;
	//
	t = m11 * c - m12 * s;
	m12 = m11 * s + m12 * c;
	m11 = t;
	//
	t = m21 * c - m22 * s;
	m22 = m21 * s + m22 * c;
	m21 = t;
	//
	t = m31 * c - m32 * s;
	m32 = m31 * s + m32 * c;
	m31 = t;
}

void rotateY(double s, double c)
{
	double t;
	t = m00 * c + m02 * s;
	m02 = -m00 * s + m02 * c;
	m00 = t;
	//
	t = m10 * c + m12 * s;
	m12 = -m10 * s + m12 * c;
	m10 = t;
	//
	t = m20 * c + m22 * s;
	m22 = -m20 * s + m22 * c;
	m20 = t;
	//
	t = m30 * c + m32 * s;
	m32 = -m30 * s + m32 * c;
	m30 = t;
}

void rotateZ(double s, double c)
{
	double t;
	t = m00 * c - m01 * s;
	m01 = m00 * s + m01 * c;
	m00 = t;
	//
	t = m10 * c - m11 * s;
	m11 = m10 * s + m11 * c;
	m10 = t;
	//
	t = m20 * c - m21 * s;
	m21 = m20 * s + m21 * c;
	m20 = t;
	//
	t = m30 * c - m31 * s;
	m31 = m30 * s + m31 * c;
	m30 = t;
}

void viewPlaneTransform(Vector3d *p)
{
	double x = p->x * m00 + p->y * m10 + p->z * m20 + m30;
	double y = p->x * m01 + p->y * m11 + p->z * m21 + m31;
	double z = p->x * m02 + p->y * m12 + p->z * m22 + m32;
	p->x = x;
	p->y = y;
	p->z = z;
}

void makePerspectiveTransformation()
{
	pc.x = center.x;
	pc.y = center.y;
	pc.z = center.z;
	viewPlaneTransform(&pc);
	if(pc.z < 0)
		perror("Center of Projection behind View Plane.");
}

void makeViewPlaneTransformation()
{
	// Start with the identity matrix
	//
	newTransform3();
	//
	// Translate so that view plane center is new origin
	//
	translate(-(position.x + direction.x * distance),
		  -(position.y + direction.y * distance),
		  -(position.z + direction.z * distance));
	//
	// Rotate so that view plane normal is z axis
	//
	double v = sqrt(direction.y * direction.y + direction.z * direction.z);
	if(v > ROUNDOFF)
		rotateX(-direction.y / v, -direction.z / v);
	rotateY(direction.x, v);
	//
	// Determine the view-up direction in these new coordinates
	//
	double xup_vp = attitude.x * m00 + attitude.y * m10 + attitude.z * m20;
	double yup_vp = attitude.x * m01 + attitude.y * m11 + attitude.z * m21;
	//
	// Determine rotation needed to make view-up vertical
	//
	double rup = sqrt(xup_vp * xup_vp + yup_vp * yup_vp);
	if(rup < ROUNDOFF)
		perror("set-view-up along view-plane normal");
	rotateZ(xup_vp / rup, yup_vp / rup);
	//
	// Transform the center of projection
	//
	if(!parallel)
		makePerspectiveTransformation();
}

void setWindow(double _wxl, double _wxh, double _wyl, double _wyh)
{
	if(_wxl >= _wxh || _wyl >= _wyh)
		perror("bad window");
	wxl = _wxl;
	wxh = _wxh;
	wyl = _wyl;
	wyh = _wyh;
}

void setViewDepth(double _frontDistance, double _backDistance)
{
	if(frontDistance > backDistance)
		perror("Frontal Plane behind Back Plane");
		//
	frontDistance = _frontDistance;
	backDistance = _backDistance;
}

void setViewPort(double _vxl, double _vxh, double _vyl, double _vyh)
{
	if(_vxl >= _vxh || _vyl >= _vyh)
		perror("bad viewport");
	vxl = _vxl;
	vxh = _vxh;
	vyl = _vyl;
	vyh = _vyh;
}

void flipMode()
{
	parallel = !parallel;
}

void flipBox()
{
	showBox = !showBox;
}

void newView2()
{
	wsx = (vxh - vxl) / (wxh - wxl);
	wsy = (vyh - vyl) / (wyh - wyl);
}

void newView3()
{
	makeViewPlaneTransformation();
	newView2();
}

void setViewDistance(double _distance)
{
	distance = _distance;
}

void setParallel(double dx, double dy, double dz)
{
        if((fabs(dx) + fabs(dy) + fabs(dz)) < ROUNDOFF)
                perror("No direction of projection");
}

void setPerspective(double x, double y, double z)
{
	center.x = x;
	center.y = y;
	center.z = z;
}

void clearBuffer()
{
	int i;
	JSAMPLE *ptr = image_buffer;
	for(i = 0; i < 3 * WIDTH * HEIGHT; i++)
		*ptr++ = 0;
}

void parallelTransform(Vector3d *p)
{
	p->x -= pc.x;
	p->y -= pc.y;
	p->z = 0;
}

void perspectiveTransform(Vector3d *p)
{
	double d = pc.z - p->z;
	if(fabs(d) < ROUNDOFF)
	{
		p->x = (p->x - pc.x) * VERYLARGE;
		p->y = (p->y - pc.y) * VERYLARGE;
		p->z = VERYLARGE;
	}
	else
	{
		p->x = (p->x * pc.z - pc.x * p->z) / d;
		p->y = (p->y * pc.z - pc.y * p->z) / d;
		p->z /= d;
	}
}

void enter(int color)
{
	// Transform the point to camera space
	//
	viewPlaneTransform(&pen);
	//
	// Clipping against clipping planes
	//
	if(clipping && (pen.z > -frontDistance || pen.z < -backDistance))
		return;
	//
	if(parallel)
		parallelTransform(&pen);
	else
		perspectiveTransform(&pen);
	//
	// Clipping against frustum sides
	//
	if(pen.x >= wxl && pen.x <= wxh && pen.y >= wyl && pen.y <= wyh)
	{
		// Convert to screen coordinates
		//
		int xi = (int) (((pen.x - wxl) * wsx + vxl) * WIDTH + .5);
		int yi = (int) (((pen.y - wyl) * wsy + vyl) * HEIGHT + .5);
		if(xi < 0 || yi < 0 || xi >=WIDTH || yi >= HEIGHT)
			return;
		//
		// Save point in output buffer
		//
		int pos = WIDTH - xi + yi * WIDTH; 
		JSAMPLE *ptr2 = image_buffer + 3 * pos;
                *ptr2++ = colors[color];
	        *ptr2++ = colors[color + 1];
		*ptr2   = colors[color + 2];
	}
}

void plot(double x, double y, double z, int color)
{
	pen.x = x;
	pen.y = y;
	pen.z = z;
	enter(color);
}

/*
 * Initializes the projection engine.
 */
void initPlot()
{
	gettimeofday(&begin, NULL);
//	image_buffer = (JSAMPLE *) malloc(3 * WIDTH * HEIGHT * sizeof(JSAMPLE));
//	initPalette();
	double h = sqrt(225) / (1.5 * GRID);
	position.x = (int)(GD_X*BD_X*10.0 / h);
	position.y = (int)(GD_X*BD_X*5.0 / h);
	position.z = (int)(GD_X*BD_X*10.0 / h);
	//
	direction.x = -position.x;		// camera axis
	direction.y = -position.y;
	direction.z = -position.z;
	norm3d(&direction);
	//
	attitude.x = 0;				// view-up direction
	attitude.y = 0;
	attitude.z = -1;
	//
	setViewPort(0.0, 1.0, 0.0, 1.0);
	setWindow(-SIDE, SIDE, -SIDE, SIDE);
	newView2();
	setViewDistance(0);
	if(!parallel)
	{
		double xc = position.x - direction.x * scale;
		double yc = position.y - direction.y * scale;
		double zc = position.z - direction.z * scale;
		setPerspective(xc, yc, zc);
	}
	newView3();
	setViewDepth(0, 10000);
	clearBuffer();
}

void drawMarker(int x, int y, int z)
{
        double dx = (x - SIDE/2);
        double dy = (y - SIDE/2);
        double dz = (z - SIDE/2);
        plot(dx, dy, dz, WHT);
}

void drawBullet(double dx, double dy, double dz, char color)
{
	double d = 1.0 / BURST;
	plot(dx, dy, dz, color);
	plot(dx+d, dy, dz, color);
	plot(dx-d, dy, dz, color);
	plot(dx, dy+d, dz, color);
	plot(dx, dy-d, dz, color);
	plot(dx, dy, dz+d, color);
	plot(dx, dy, dz-d, color);
}

void drawVoxel(double dx, double dy, double dz, char color)
{
	int N = SIDE / 2 + 1;
	double d = 1.0 / BURST;
	int M = SIDE *d / 4;
	dx-=M; dy-=M; dz-=M;
	int x, y, z;
	for(x = 0; x < N; x++)
		for(y = 0; y < N; y++)
			plot(dx + x*d, dy + y*d, dz, color);
	for(y = 0; y < N; y++)
		for(z = 0; z < N; z++)
			plot(dx, dy + y*d, dz + z*d, color);
	for(z = 0; z < N; z++)
		for(x = 0; x < N; x++)
			plot(dx + x*d, dy, dz + z*d, color);
	//
	for(x = 0; x < N; x++)
		for(y = 0; y < N; y++)
			plot(dx + x*d, dy + y*d, dz + N*d, color);
	for(y = 0; y < N; y++)
		for(z = 0; z < N; z++)
			plot(dx+N*d, dy + y*d, dz + z*d, color);
	for(z = 0; z < N; z++)
		for(x = 0; x < N; x++)
			plot(dx + x*d, dy+N*d, dz + z*d, color);
}

/*
 * Draws the automaton voxels.
 */
void drawLattice()
{
	int x, y, z;
	char *v = voxels; 
	for(x = 0; x < SIDE; x++)
		for(y = 0; y < SIDE; y++)
			for(z = 0; z < SIDE; z++)
			{
				double dx = (x - SIDE/2);
				double dy = (y - SIDE/2);
				double dz = (z - SIDE/2);
				//
				char color = *v;
				if(color != BLK)
				{
					if(SIDE < 128)
						drawVoxel(dx, dy, dz, color);
					else
						drawBullet(dx, dy, dz, color);
				}
				else if(showGrid)
					plot(dx, dy, dz, gridcolor);
				v++;
			}
	drawMarker(0,0,0);
}

void drawBox()
{
	double L = SIDE / 2;
	double incr = L / 200;
	double dx = -L;
	while(dx < L)
	{
		plot(dx, L, L, BOX);
		plot(dx, -L, L, BOX);
		plot(dx, L, -L, BOX);
		plot(dx, -L, -L, BOX);
		dx +=incr;
	}
	double dy = -L;
	while(dy < L)
	{
		plot(L, dy, L, BOX);
		plot(-L, dy, L, BOX);
		plot(L, dy, -L, BOX);
		plot(-L, dy, -L, BOX);
		dy +=incr;
	}
	double dz = -L;
	while(dz < L)
	{
		plot(L, L, dz, BOX);
		plot(-L, L, dz, BOX);
		plot(L, -L, dz, BOX);
		plot(-L, -L, dz, BOX);
		dz +=incr;
	}
}

void drawChar(double x, double y, double z, char color, char ch)
{
	pen.x = x;
	pen.y = y;
	pen.z = z;

	// Transform the point to camera space
	//
	viewPlaneTransform(&pen);
	//
	// Clipping against clipping planes
	//
	if(clipping && (pen.z > -frontDistance || pen.z < -backDistance))
		return;
	//
	if(parallel)
		parallelTransform(&pen);
	else
		perspectiveTransform(&pen);
	//
	// Clipping against frustum sides
	//
	if(pen.x >= wxl && pen.x <= wxh && pen.y >= wyl && pen.y <= wyh)
	{
		// Convert to screen coordinates
		//
		int xi = (int) (((pen.x - wxl) * wsx + vxl) * WIDTH + .5);
		int yi = (int) (((pen.y - wyl) * wsy + vyl) * HEIGHT + .5);
		vprint(WIDTH - xi, yi, ch);
	}
}

void drawAxes()
{
	setViewPort(0.0, 0.2, 0.0, 0.2);
	setWindow(-40, 40, -40, 40);
	newView2();
	setViewDistance(0);
	if(!parallel)
	{
		double xc = position.x - direction.x * scale;
		double yc = position.y - direction.y * scale;
		double zc = position.z - direction.z * scale;
		setPerspective(xc, yc, zc);
	}
	newView3();
	for(int i = 0; i < 35; i++)
	{
		double p = i;
		if(i < 30)
		{
			plot(p, 0, 0, X);
			plot(p, +0.3, 0, X);
			plot(p, -0.3, 0, X);
			plot(p, 0, +0.3, X);
			plot(p, 0, -0.3, X);
			plot(0, p, 0, Y);
			plot(.3, p, 0, Y);
			plot(-.3, p, 0, Y);
			plot(0, p, +0.3, Y);
			plot(0, p, -0.3, Y);
			plot(0, 0, p, Z);
			plot(0, +0.3, p, Z);
			plot(0, -0.3, p, Z);
			plot(+0.3, 0, p, Z);
			plot(-0.3, 0, p, Z);
		}
		else if(i == 34)
		{
			drawChar(p, 0, 0, X, 'x');
			drawChar(0, p, 0, Y, 'y');
			drawChar(0, -0.3, p, Z, 'z');
		}
	}
	setViewPort(0.0, 1.0, 0.0, 1.0);
	setWindow(-SIDE, SIDE, -SIDE, SIDE);
	newView2();
	setViewDistance(0);
	if(!parallel)
	{
		double xc = position.x - direction.x * scale;
		double yc = position.y - direction.y * scale;
		double zc = position.z - direction.z * scale;
		setPerspective(xc, yc, zc);
	}
	newView3();
}

/*
 * Called by automaton.c Loop3d thread
 */
void updatePlot()
{
	// Camera follows a circle and points to center
	//
	if(rot)
	{
		position.x = 1.5 * GRID * cos(theta);
		position.y = 1.5 * GRID * sin(theta);
		position.z = 0;
		//
		direction.x = -cos(theta);
		direction.y = -sin(theta);
		direction.z = 0;
		//
		attitude.x = 0;
		attitude.y = 0;
		attitude.z = 1;
		//
		theta += 0.01;
	}
	//
	// Create a new 2d view
	//
	newView2();
	//
	// Move the center of projection
	//
	if(!parallel)
	{
		setPerspective(	position.x - scale * direction.x,
				position.y - scale * direction.y,
				position.z - scale * direction.z);
	}
	//
	// Create a new 3d view
	//
	newView3();
	//
	// Recreate pixels
	//
	clearBuffer();
	drawLattice();
	if(showBox)
		drawBox();
	if(showAxes)
		drawAxes();
}

void updateCamera()
{
        position.x = _position.x;
        position.y = _position.y;
        position.z = _position.z;
        //
        direction.x = _direction.x;
        direction.y = _direction.y;
        direction.z = _direction.z;
}

void visualize()
{
	updatePlot();
	//
	// Graphic text
	//
	struct timeval now;
	gettimeofday(&now, NULL);
	unsigned long long elapsed = 1000 * (now.tv_sec - begin.tv_sec) + (now.tv_usec - begin.tv_usec) / 1000;
	JSAMPLE *s;
	asprintf((char **)&s, "Automaton %dx%dx%dx%d", SIDE, SIDE, SIDE, NPREONS);
	vprints(20, 20, s);
	//
	asprintf((char **)&s, "Scenario: %s", scenarios[scenario]);
	vprints(330, 20, s);
	//
	asprintf((char **)&s, "%d threads", NTHREADS);
        vprints(20, 60, s);
	//
	asprintf((char **)&s, "Elapsed: %llu ms", elapsed);
        vprints(20, 40, s);
	//
	asprintf((char **)&s, "Views:");
	vprints(20, 90, s);
	//
	asprintf((char **)&s, "0: isometric");
	vprints(25, 110, s);
	//
	asprintf((char **)&s, "1: xy");
	vprints(25, 130, s);
	//
	asprintf((char **)&s, "2: yz");
	vprints(25, 150, s);
	//
	asprintf((char **)&s, "3: zx");
	vprints(25, 170, s);
	//
	asprintf((char **)&s, "ls=%lu  clk=%lu", timer / (2 * DIAMETER), timer);
	vprints(20, 740, s);
	//
	asprintf((char **)&s, "A: axes on/off");
	vprints(620, 680, s);
	//
	asprintf((char **)&s, "S: pause/resume");
	vprints(620, 700, s);
	//
	asprintf((char **)&s, "G: grid on/off");
	vprints(620, 720, s);
	//
	asprintf((char **)&s, "X: box on/off");
	vprints(620, 740, s);
	//
	write_JPEG_file();
}

void *DisplayLoop(void *v)
{
        pthread_detach(pthread_self());
        pthread_mutex_unlock(&cam_mutex);
        while(true)
        {
		if(splash)
		{
			image_buffer = (JSAMPLE *) malloc(3 * WIDTH * HEIGHT * sizeof(JSAMPLE));
		        initPalette();
			clearBuffer();
			//
			JSAMPLE *s;
			if(scenario == -1)
			{
				asprintf((char **)&s, "Select scenario:");
				vprints(310, 310, s);
				for(int i = 0; i < 8; i++)
				{
					asprintf((char **)&s, "%d - %s", i+1, scenarios[i]);
					vprints(330, 330 + 15*i, s);
				}
			}
			else
			{
				asprintf((char **)&s, "[WAIT...]");
				vprints(370, 395, s);
			}
			//
			write_JPEG_file();
			usleep(200000);
		}
		else
		{
	                pthread_mutex_lock(&cam_mutex);
			if(automaton_changed)
			{
				img_changed = true;
				automaton_changed = false;
			}
			pthread_mutex_unlock(&cam_mutex);
			if(input_changed)
			{
				updateCamera();
				input_changed = false;
				img_changed = true;
			}
			if(img_changed)
			{
				visualize();
				img_changed = false;
			}
			usleep(10000);
		}
        }

}

