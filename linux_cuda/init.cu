#include "hip/hip_runtime.h"
/*
 * init.cu
 */
#include <stdlib.h>
#include <assert.h>
#include "init.h"
#include "common.h"
#include "automaton.h"
#include "params.h"
#include "tuple.h"
#include "plot3d.h"
#include "brick.h"
#include "utils.h"
#include "scenarios.h"

pthread_mutex_t cam_mutex = PTHREAD_MUTEX_INITIALIZER;

__device__ int d_prime;

/*
 * Initializes sine wave parameters.
 */
void initSineWave()
{
	double wT = 2 * M_PI / SIDE;
	double k = 2 * cos(wT);
	double u1 = SIDE * sin(-2 * wT);
	double u2 = SIDE * sin(-wT);
	hipMemcpyToSymbol(HIP_SYMBOL(&K), &k, sizeof(k), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&U1), &u1, sizeof(u1), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&U2), &u2, sizeof(u2), 0, hipMemcpyHostToDevice);
}

__global__ void buildGrid(Brick *t0)
{
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        int z = blockDim.z * blockIdx.z + threadIdx.z;
	int offset = SIDE2*x + SIDE*y + z;
	if(offset >= SIDE3)
		return;
        Brick *t = t0 + offset*NPREONS;
        for(int w = 0; w < NPREONS; w++, t++)
        {
                memset(t, 0, sizeof(Brick));
		t->p0.x = x;
		t->p0.y = y;
		t->p0.z = z;
                t->p19 = w;
        }
}

/*
 * Inserts a preon in a specified address of the pri0 lattice.
 *
 * @b	brick
 * @p4	spin
 * @p6	electric charge
 * @p7	chirality
 * @p8  gravity
 * @p9	color
 * @p21 status
 * @p24 schedule
 * @p25	messenger
 */
__global__ void addPreon(Brick *t, Brick *t0, Tuple pos, int w, Tuple p4, char p5, char p6, char p7, int p8, 
unsigned char p9, int p21, unsigned p24, int p25)
{
	Tuple xyz;
        xyz.x = blockDim.x * blockIdx.x + threadIdx.x;
        xyz.y = blockDim.y * blockIdx.y + threadIdx.y;
        xyz.z = blockDim.z * blockIdx.z + threadIdx.z;
        int offset = SIDE2*xyz.x + SIDE*xyz.y + xyz.z;
	if(offset < SIDE3 && isEqual(xyz, pos))
	{
		t = t0 + offset*NPREONS + w;
		cleanBrick(t);
		t->p4 = p4;
		t->p5 = p5;
		t->p6 = p6;
		t->p7 = p7;
		t->p8 = p8;
		t->p9 = p9;
		t->p15.x = -1;
		t->p21 = p21;
		t->p24 = p24;
		t->p25 = p25;
	}
}

/*
 * Initializes the automaton program.
 */
void initAutomaton()
{
	initSineWave();
	size_t size = NCELLS * sizeof(Brick);
	//
	// Init principal lattice
	//
	hipMalloc(&d_pri0, size);
	buildGrid<<<gridDim,blockDim>>>(d_pri0);
	//
	int limit = floor(sqrt(3) * (1 << (ORDER - 1)));
	hipMemcpyToSymbol(HIP_SYMBOL(d_limit), &limit, sizeof(limit));
	//
	int prime = getPrime(SIDE);
	hipMemcpyToSymbol(HIP_SYMBOL(d_prime), &prime, sizeof(prime));
	//
	b = (Brick *)malloc(sizeof(Brick));
	//
	// Initial state of the universe
	//
	assert(scenario>=0);
	switch(scenario)
	{
		case 0:
			BurstScenario();
			break;
		case 1:
			UScenario();
			break;
		case 2:
			VacuumScenario();
			break;
		case 3:
			NoUXUScenario();
			break;
		case 4:
			UXUScenario();
			break;
		case 5:
			GRAVScenario();
			break;
		case 6:
			LonePScenario();
			break;
		case 7:
			BigBangScenario();
			break;
	}
	//
	// Init dual lattice
	//
	hipMalloc(&d_dual0, size);
	buildGrid<<<gridDim,blockDim>>>(d_dual0);
}


