#include "hip/hip_runtime.h"
/*
 * rotation.cu
 */


#include "rotation.h"
#include <stdio.h>
#include <math.h>

#include "params.h"
#include "tuple.h"


__device__ static float rotationMatrix[4][4];
__device__ static float inputMatrix[4][1] = {{0.0}, {0.0}, {0.0}, {0.0}};
__device__ static float outputMatrix[4][1] = {{0.0}, {0.0}, {0.0}, {0.0}};

__device__ void multiplyMatrix()
{
	int i;
    for(i = 0; i < 4; i++ )
    {
	int j;
        for(j = 0; j < 1; j++)
        {
            outputMatrix[i][j] = 0;
            int k;
            for(k = 0; k < 4; k++)
                outputMatrix[i][j] += rotationMatrix[i][k] * inputMatrix[k][j];
        }
    }
}

__device__ void setUpRotationMatrix(float angle, float u, float v, float w)
{
    float L = (u*u + v * v + w * w);
    angle = angle * M_PI / 180.0f; //converting to radian value
    float u2 = u * u;
    float v2 = v * v;
    float w2 = w * w;

    rotationMatrix[0][0] = (u2 + (v2 + w2) * cos(angle)) / L;
    rotationMatrix[0][1] = (u * v * (1 - cos(angle)) - w * sqrt(L) * sin(angle)) / L;
    rotationMatrix[0][2] = (u * w * (1 - cos(angle)) + v * sqrt(L) * sin(angle)) / L;
    rotationMatrix[0][3] = 0.0;

    rotationMatrix[1][0] = (u * v * (1 - cos(angle)) + w * sqrt(L) * sin(angle)) / L;
    rotationMatrix[1][1] = (v2 + (u2 + w2) * cos(angle)) / L;
    rotationMatrix[1][2] = (v * w * (1 - cos(angle)) - u * sqrt(L) * sin(angle)) / L;
    rotationMatrix[1][3] = 0.0;

    rotationMatrix[2][0] = (u * w * (1 - cos(angle)) - v * sqrt(L) * sin(angle)) / L;
    rotationMatrix[2][1] = (v * w * (1 - cos(angle)) + u * sqrt(L) * sin(angle)) / L;
    rotationMatrix[2][2] = (w2 + (u2 + v2) * cos(angle)) / L;
    rotationMatrix[2][3] = 0.0;

    rotationMatrix[3][0] = 0.0;
    rotationMatrix[3][1] = 0.0;
    rotationMatrix[3][2] = 0.0;
    rotationMatrix[3][3] = 1.0;
}

/*
 * Calculates spin rotation.
 */
__device__ void rotateSpin(Brick *t)
{
	int distance = (int) modTuple(&t->p2) % (2 * DIAMETER);
	float angle = 2 * M_PI * distance * t->p5 / (2 * DIAMETER);
    float u = (float) t->p2.x, v = (float) t->p2.y, w = (float) t->p2.z;
    inputMatrix[0][0] = t->p4.x;
    inputMatrix[1][0] = t->p4.y;
    inputMatrix[2][0] = t->p4.z;
    inputMatrix[3][0] = 1.0;
    setUpRotationMatrix(angle, u, v, w);
    multiplyMatrix();
    t->p4.x = outputMatrix[0][0];
    t->p4.y = outputMatrix[1][0];
    t->p4.z = outputMatrix[2][0];
}


