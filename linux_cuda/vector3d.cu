#include "hip/hip_runtime.h"
/*
 * vector3d.cu
 */
#include "vector3d.h"
#include <math.h>
#include <stdio.h>

char vectorBuf[4][30];

__host__ __device__ void norm3d(Vector3d *v)
{
	double h = sqrt(v->x * v->x + v->y * v->y + v->z * v->z);
	if(h == 0.0)
	{
		v->x = 0;
		v->y = 0;
		v->z = 0;
	}
	else
	{
		v->x /= h;
		v->y /= h;
		v->z /= h;
	}
}

__host__ __device__ void add3d(Vector3d *a, Vector3d b)
{
	a->x += b.x;
	a->y += b.y;
	a->z += b.z;
}

__host__ __device__ void sub3d(Vector3d *a, Vector3d b)
{
	a->x -= b.x;
	a->y -= b.y;
	a->z -= b.z;
}

__host__ __device__ double dot3d(Vector3d v1, Vector3d v2)
{
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__host__ __device__ void cross3d(Vector3d v1, Vector3d v2, Vector3d *v3)
{
	v3->x = v1.y * v2.z - v1.z * v2.y;
	v3->y = v1.z * v2.x - v1.x * v2.z;
	v3->z = v1.x * v2.y - v1.y * v2.x;
}

__host__ __device__ void rot3d(Vector3d *p, Vector3d axis, double angle)
{
	double x = p->x;
	double y = p->y;
	double z = p->z;
	double s = sin(angle);
	double c = cos(angle);
	p->x = (c + axis.x * axis.x * (1 - c)) * x + (axis.x * axis.y * (1 - c) - axis.z * s) * y + (axis.x * axis.z * (1 - c) + axis.y * s) * z;
	p->y = (axis.y * axis.x * (1 - c) + axis.z * s) * x + (c + axis.y * axis.y * (1 - c)) * y + (axis.y * axis.z * (1 - c) - axis.x * s) * z;
	p->z = (axis.z * axis.x * (1 - c) - axis.y * s) * x + (axis.z * axis.y * (1 - c) + axis.x * s) * y + (c + axis.z * axis.z * (1 - c)) * z;
}

__host__ __device__ void scale3d(Vector3d *v, double s)
{
	v->x *= s;
	v->y *= s;
	v->z *= s;
}

__host__ __device__ double module3d(Vector3d *v)
{
	return sqrt(v->x * v->x + v->y * v->y + v->z * v->z);
}

__host__ __device__ void absV3d(Vector3d *v)
{
	v->x = fabs(v->x);
	v->y = fabs(v->y);
	v->z = fabs(v->z);
}

__host__ __device__ void invert3d(Vector3d *v)
{
	v->x = -v->x;
	v->y = -v->y;
	v->z = -v->z;
}

__host__ char *vector2str(Vector3d *v)
{
        static int index = 0;
        char *ptr = vectorBuf[index];
        sprintf(ptr, "(%f,%f,%f)", v->x, v->y, v->z);
        index++;
        index &= 3;
        return ptr;
}

