#include "hip/hip_runtime.h"
/*
 * tuple.cu
 */
#include "tuple.h"
#include "params.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

__constant__ Tuple V0 = { 1.732*SIDE, 1.732*SIDE, 1.732*SIDE};

__host__ __device__ void resetTuple(Tuple *t)
{
        t->x = 0;
        t->y = 0;
        t->z = 0;
}

__device__ void rectify(Tuple *v)
{
	if(v->x >= SIDE)
		v->x -= SIDE;
	if(v->y >= SIDE)
		v->y -= SIDE;
	if(v->z >= SIDE)
		v->z -= SIDE;
	//
	if(v->x < 0)
		v->x += SIDE;
	if(v->y < 0)
		v->y += SIDE;
	if(v->z < 0)
		v->z += SIDE;
}

__device__ int isNull(Tuple t)
{
	return t.x == 0 && t.y == 0 && t.z == 0;
}

__device__ int isEqual(Tuple t1, Tuple t2)
{
	return t1.x == t2.x && t1.y == t2.y && t1.z == t2.z;
}

__device__ int isOpposite(Tuple t1, Tuple t2)
{
	return t1.x == -t2.x && t1.y == -t2.y && t1.z == -t2.z;
}

__device__ void invertTuple(Tuple *t)
{
	t->x = -t->x;
	t->y = -t->y;
	t->z = -t->z;
}

__device__ void addTuples(Tuple *a, Tuple b)
{
	a->x += b.x;
	a->y += b.y;
	a->z += b.z;
}

__device__ void addRectify(Tuple *a, Tuple b)
{
	a->x += b.x;
	a->y += b.y;
	a->z += b.z;
	rectify(a);
}

__device__ void subTuples(Tuple *a, Tuple b)
{
	a->x -= b.x;
	a->y -= b.y;
	a->z -= b.z;
}

__device__ void subRectify(Tuple *a, Tuple b)
{
	a->x -= b.x;
	a->y -= b.y;
	a->z -= b.z;
	rectify(a);
}

__device__ void subTuples3(Tuple *r, Tuple a, Tuple b)
{
	r->x = a.x - b.x;
	r->y = a.y - b.y;
	r->z = a.z - b.z;
}

/*
 * Module.
 */
__device__ double modTuple(Tuple *v)
{
	return sqrt((double)(v->x * v->x + v->y * v->y + v->z * v->z));
}

/*
 * Module squared.
 */
__device__ double mod2Tuple(Tuple *v)
{
	return v->x * v->x + v->y * v->y + v->z * v->z;
}

__device__ void normalizeTuple(Tuple *t)
{
	double h = sqrt((double)(t->x * t->x + t->y * t->y + t->z * t->z));
	t->x = (int)(t->x * SIDE / h);
	t->y = (int)(t->y * SIDE / h);
	t->z = (int)(t->z * SIDE / h);
}

__device__ void tupleCross(Tuple v1, Tuple v2, Tuple *v3)
{
	v3->x = v1.y * v2.z - v1.z * v2.y;
	v3->y = v1.z * v2.x - v1.x * v2.z;
	v3->z = v1.x * v2.y - v1.y * v2.x;
}

__device__ int compareTuples(Tuple *a, Tuple *b)
{
	int la = a->x * a->x + a->y * a->y + a->z * a->z;
	int lb = b->x * b->x + b->y * b->y + b->z * b->z;
	if(la == lb)
		return 0;
	return la > lb ? 1 : -1;
}

__device__ int tupleDot(Tuple *a, Tuple *b)
{
	return a->x * b->x + a->y * b->y + a->z * b->z;
}

__device__ void tupleAbs(Tuple *t)
{
	t->x = abs(t->x);
	t->y = abs(t->x);
	t->z = abs(t->x);
}

__device__ void scaleTuple(Tuple *t, int s)
{
	t->x *= s;
	t->y *= s;
	t->z *= s;
}

char *tuple2str(Tuple *t)
{
	char *s;
	asprintf((char **)&s, "[%d,%d,%d]", t->x, t->y, t->z);
	return s;
}


