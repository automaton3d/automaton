#include "hip/hip_runtime.h"
/*
 * Interaction routines
 * 
 * The code is intentionally left not optimized to enhance
 * where the rules were supressed.
*/
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

/*
 * Both re-emmited from CP.
 */
__device__ void cpcp(Cell* draft1, Cell* draft2, bool collapse)
{
	// Copy momentum information for flash
	//
	if (ALIGNED(draft1->o, draft1->p))
	{
		COPY(draft1->pole, draft1->p);
		COPY(draft2->pole, draft1->p);
	}
	else
	{
		COPY(draft1->pole, draft2->p);
		COPY(draft2->pole, draft2->p);
	}
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
	//
	if (collapse)
	{
		// Disintegrate the packet
		//
		draft1->code = COLLAPSE;
		draft2->code = COLLAPSE;
	}
}

/*
 * Both re-emmited from respective pole.
 */
__device__ void polepole(Cell* draft1, Cell* draft2)
{
	// Copy momentum information for flash
	//
	COPY(draft1->pole, draft1->p);
	COPY(draft2->pole, draft2->p);
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
}

/*
 * Inertia mechanism.
 */
__device__ void inertia(Cell* draft1, Cell* draft2)
{
	// Copy momentum information for flash
	//
	if (ALIGNED(draft1->o, draft1->p))
	{
		COPY(draft1->pole, draft2->p);
		COPY(draft2->pole, draft2->p);
	}
	else
	{
		COPY(draft1->pole, draft1->p);
		COPY(draft2->pole, draft1->p);
	}
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
}

__device__ void bosonxboson(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->charge & C_MASK) & 7;
	unsigned q1 = ((stable1->charge & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->charge & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->charge & D_MASK) >> 5;
	unsigned c2 = (stable2->charge & C_MASK) & 7;
	unsigned q2 = ((stable2->charge & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->charge & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->charge & D_MASK) >> 5;
	//
	if (d1 != d2)
	{
		// SUPRESSED
		// Interactions between pairs in different sectors are not allowed
		//
		return;
	}
	//
	// Non-trivial colors?
	//
	if (c1 != NEUTRAL && c1 != NEUTRAL_BAR && c2 != NEUTRAL && c2 != NEUTRAL_BAR)
	{
		// Cohesion of gluons?
		//
		if (c1 == c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		//
		// Complementary colors?
		//
		else if (c1 == ~c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		//
		// Colors are diverse
		//
		else
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		//
		// Swap colors
		// Blindly exchange colors, ignoring all other charges
		//
		int c1 = stable1->charge & C_MASK;
		int c2 = stable2->charge & C_MASK;
		draft1->charge &= ~C_MASK;
		draft2->charge &= ~C_MASK;
		draft1->charge |= c2;
		draft2->charge |= c1;
		//
		polepole(draft1, draft2);
	}
	//
	// Gluon x [photon, Z, W]
	//
	else if (c1 != NEUTRAL && c1 != NEUTRAL_BAR)
	{
		if (c1 == c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		else if (c1 == ~c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		else
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			// SUPRESSED
		}
		// SUPRESSED
		// There is no evidence for gluon interacting with other bosons
	}
	//
	// [photon,Z,W] x [gluon]
	//
	else if (c2 != NEUTRAL && c2 != NEUTRAL_BAR)
	{
		if (c1 == c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		else if (c1 == ~c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		else
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
		}
		// SUPRESSED
		// There is no evidence for gluon interacting with other bosons
	}
	//
	// [photon,Z,W] x [photon,Z,W]
	//
	else
	{
		if (q1 == q2)
		{
			if (w1 == w2)
			{
				// SUPRESSED
			}
			else
			{
				// SUPRESSED
			}
			// SUPRESSED
		}
		else
		{
			if (w1 == w2)
			{
				// SUPRESSED
			}
			else
			{
				// SUPRESSED
			}
			// SUPRESSED
		}
		// SUPRESSED
		// There is no evidence for interaction between these bosons
	}
}

__device__ void fermionxboson(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->charge & C_MASK) & 7;
	unsigned q1 = ((stable1->charge & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->charge & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->charge & D_MASK) >> 5;
	unsigned c2 = (stable2->charge & C_MASK) & 7;
	unsigned q2 = ((stable2->charge & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->charge & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->charge & D_MASK) >> 5;
	//
	if (d1 != d2)
	{
		// SUPRESSED
		// Only bubbles in the same sector are allowed to interact in this way
		//
		return;
	}
	//
	// Quark x gluon?
	//
	if (c1 != NEUTRAL && c1 != NEUTRAL_BAR && c2 != NEUTRAL && c2 != NEUTRAL_BAR)
	{
		if (c1 == c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
				}
				else
				{
				}
			}
			else
			{
				if (w1 == w2)
				{
				}
				else
				{
				}
			}
		}
		else if (c1 == ~c2)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
				}
				else
				{
				}
			}
			else
			{
				if (w1 == w2)
				{
				}
				else
				{
				}
			}
		}
		//
		// Electron x photon
		//
		else
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			//
			// Reissue both from CP
			//
			cpcp(draft1, draft2, true);
		}
	}
	//
	// Quark x [photon, Z, W]?
	//
	else if (c1 != NEUTRAL && c1 != NEUTRAL_BAR)
	{
		if (q1 == q2)
		{
			if (w1 == w2)
			{
			}
			else
			{
			}
		}
		else
		{
			if (w1 == w2)
			{
			}
			else
			{
			}
		}
	}
	//
	// Electron x gluon?
	//
	else if (c2 != NEUTRAL && c2 != NEUTRAL_BAR)
	{
		if (q1 == q2)
		{
			if (w1 == w2)
			{
				// SUPRESSED
			}
			else
			{
				// SUPRESSED
			}
			// SUPRESSED
		}
		else
		{
			if (w1 == w2)
			{
				// SUPRESSED
			}
			else
			{
				// SUPRESSED
			}
			// SUPRESSED
		}
		// SUPRESSED
	}
	//
	// Electron x [photon, Z, W]?
	//
	else
	{
		if (q1 == q2)
		{
			if (w1 == w2)
			{
				if (c1 == c2 == 0 && w1 == 1)
				{
					polepole(draft1, draft2);
				}
				else if (c1 == c2 == 7 && w1 == 0)
				{
					polepole(draft1, draft2);
				}
			}
			else
			{
				// SUPRESSED
			}
		}
		else
		{
			if (w1 == w2)
			{
				if (c1 == c2 == 0 && w1 == 1)
				{
					polepole(draft1, draft2);
				}
				else if (c1 == c2 == 7 && w1 == 0)
				{
					polepole(draft1, draft2);
				}
			}
			else
			{
				// SUPRESSED
			}
		}
		//
		// Is it a propeller?
		//
		if (stable1->b == stable2->b)
		{
			// Inertia
			//
			inertia(draft1, draft2);
		}
	}
}

__device__ void fermionxfermion(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->charge & C_MASK) & 7;
	unsigned q1 = ((stable1->charge & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->charge & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->charge & D_MASK) >> 5;
	unsigned c2 = (stable2->charge & C_MASK) & 7;
	unsigned q2 = ((stable2->charge & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->charge & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->charge & D_MASK) >> 5;
	//
	//
	// Matter/antimatter flags
	//
	bool matter1 = c1 == 0 || c1 == 1 || c1 == 2 || c1 == 4;
	bool matter2 = c2 == 0 || c2 == 1 || c2 == 2 || c2 == 4;
	//
	// Same sector?
	//
	if (d1 == d2)
	{
		// quark x quark?
		//
		if (c1 != NEUTRAL && c1 != NEUTRAL_BAR && c2 != NEUTRAL && c2 != NEUTRAL_BAR)
		{
			if (c1 == c2)
			{
				// Quark cohesion?
				//
				if (q1 == q2)
				{
					if (w1 == w2)
					{
						// SUPRESSED
					}
					else
					{
						// SUPRESSED
					}
					polepole(draft1, draft2);
				}
				//
				// Different electric charge
				//
				else
				{
					if (w1 == w2)
					{
						// SUPRESSED
					}
					else
					{
						// Quark annihilation?
						//
						if (c1 == ~c2)
						{
							cpcp(draft1, draft2, true);
						}
						else
						{
							// SUPRESSED
						}
					}
				}
			}
			//
			// Complementary colors?
			//
			else if (c1 == ~c2)
			{
				if (q1 == q2)
				{
					if (w1 == w2)
					{
						// SUPRESSED
					}
					else
					{
						// SUPRESSED
					}
				}
				//
				// Opposite electric charges
				//
				else
				{
					if (w1 == w2)
					{
						// SUPPRESSED
					}
					//
					// Quark annihilation
					//
					else
					{
						cpcp(draft1, draft2, true);
					}
				}
			}
			//
			// Diverse colors
			//
			else
			{
				if (q1 == q2)
				{
					if (w1 == w2)
					{
						// SUPPRESSED
					}
					else
					{
						// SUPPRESSED
					}
					// SUPPRESSED
				}
				else
				{
					if (w1 == w2)
					{
						// SUPPRESSED
					}
					else
					{
						// SUPPRESSED
					}
					// SUPPRESSED
				}
				int c1 = stable1->charge & C_MASK;
				int c2 = stable2->charge & C_MASK;
				draft1->charge &= ~C_MASK;
				draft2->charge &= ~C_MASK;
				draft1->charge |= c2;
				draft2->charge |= c1;
				cpcp(draft1, draft2, false);
			}
		}
		//
		// quark x electron?
		//
		else if (c1 != NEUTRAL && c1 != NEUTRAL_BAR)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
					// SUPRESSED
				}
				else
				{
					// SUPRESSED
				}
				// SUPRESSED
			}
			else
			{
				if (w1 == w2)
				{
					if (matter1 != matter2)
					{
						polepole(draft1, draft2);
					}
					else
					{
						// SUPRESSED
					}
				}
				else
				{
					if (matter1 == matter2)
					{
						polepole(draft1, draft2);
					}
					else
					{
						// SUPRESSED
					}
				}
			}
		}
		//
		// electron x quark
		//
		else if (c2 != NEUTRAL && c2 != NEUTRAL_BAR)
		{
			if (q1 == q2)
			{
				if (w1 == w2)
				{
				}
				else
				{
				}
			}
			else
			{
				if (w1 == w2)
				{
					if (matter1 == matter2)
					{
						polepole(draft1, draft2);
					}
					else
					{
						// SUPRESSED
					}
				}
				else
				{
					if (matter1 != matter2)
					{
						polepole(draft1, draft2);
					}
					else
					{
						// SUPRESSED
					}
				}
			}
		}
	}
	//
	// Different sectors
	//
	else
	{
		bool s1 = (c1 == c2 == 0 && q1 == q2 == 1 && d1 == d2 == 0);
		bool s2 = (c1 == c2 == 7 && q1 == q2 == 0 && d1 == d2 == 1);
		bool s3 = (c1 == c2 != 0 != 7 && q1 == q2);
		//
		bool c1 = (q1 == q2 && w1 == w2 && s1 == s2);
		bool c2 = (q1 != q2 && w1 != w2 && s1 != s2);
		bool c3 = (d1 == 0 && w1 == 0 && w1 != w2);
		bool c4 = (d1 == 1 && w1 == 1 && w1 != w2);
		//
		if ((d1 == 0 && s1 == s2) || (d1 == 1 && s1 == s3 && s1 != s2))
		{
			// Swap colors
			//
			int c1 = stable1->charge & C_MASK;
			int c2 = stable2->charge & C_MASK;
			draft1->charge &= ~C_MASK;
			draft2->charge &= ~C_MASK;
			draft1->charge |= c2;
			draft2->charge |= c1;
			//
			cpcp(draft1, draft2, true);
		}
		//
		// Chiral?
		//
		else if (c1  ||  c2  ||  c3  ||  c4) 
		{
			// Change hands
			//
			int w1 = stable1->charge & W_MASK;
			int w2 = stable2->charge & W_MASK;
			draft1->charge &= ~W_MASK;
			draft2->charge &= ~W_MASK;
			draft1->charge |= w2;
			draft2->charge |= w1;
			//
			polepole(draft1, draft2);
		}
	}
}


/*
 * Compares two cells in adjacent columns. 
 */
__device__ void compareCols(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Play pseudo dices
	//
	if (stable1->noise > abs(stable1->phi) &&
		stable2->noise > abs(stable2->phi) &&
		(!ISNULL(stable1->p) || !ISNULL(stable2->p)))
	{
		// Preserve momentum for parallel transport
		//
		COPY(draft1->pole, stable2->p);
		COPY(draft2->pole, stable1->p);
		//
		if (stable1->code == BOSON && stable2->code == BOSON)
			bosonxboson(stable1, stable2, draft1, draft2);
		else if (stable1->code == BOSON && stable2->code == FERMION)
			fermionxboson(stable2, stable1, draft2, draft1);
		else if (stable1->code == FERMION && stable2->code == BOSON)
			fermionxboson(stable1, stable2, draft1, draft2);
		else
			fermionxfermion(stable1, stable2, draft1, draft2);
	}
}

/*
 * Confronts cells for interactions. 
 */
__global__ void interact(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Interactions only allowed at the last tick of a light step
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		// Compare columns for interaction match
		//
		Cell* stable1 = stable;
		Cell* draft1 = draft;
		for (int i = 0; i < SIDE2; i++)
		{
			// If the re-emission cell was reached, reset the pole vector to the free bubble
			//
			if (stable1->flash)
			{
				RESET(draft1->o);
				draft1->t = 0;
			}
			else
			{
				Cell* stable2 = stable;
				Cell* draft2 = draft;
				for (int j = 0; j < SIDE2; j++)
				{
					if (i != j && stable1->f > 0 && stable2->f > 0 &&
						stable1->b != stable2->b && !ISEQUAL(stable1->o, stable2->o))
						compareCols(stable1, stable2, draft1, draft2);
					//
					stable2 = nextV(stable2);
					draft2 = nextV(draft2);
				}
			}
			//
			stable1 = nextV(stable1);
			draft1 = nextV(draft1);
		}
	}
}
