#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

__device__ void compareColumns(Cell *stable1, Cell *stable2, Cell *draft1, Cell *draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->charge & C_MASK) & 7;
	unsigned q1 = ((stable1->charge & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->charge & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->charge & D_MASK) >> 5;
	unsigned c2 = (stable2->charge & C_MASK) & 7;
	unsigned q2 = ((stable2->charge & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->charge & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->charge & D_MASK) >> 5;
	//
	// Play pseudo dices
	//
	if (stable1->noise > abs(stable1->phi) &&
		stable2->noise > abs(stable2->phi) &&
		(!ISNULL(stable1->p) || !ISNULL(stable2->p)))
	{
		bool sig1 = (c1 == NEUTRAL && c2 == NEUTRAL && q1 == 1 && q2 == 1 && d1 == 0);
		bool sig2 = (c1 == N_BAR && c2 == N_BAR && q1 == 1 && q2 == 1 && d1 == 1);
		bool sig3 = (c2 != NEUTRAL && c2 != N_BAR && q1 == q2);
		//
		int c1 = (q1 == q2 && w1 == w2 && sig1 == sig2);
		int c2 = (q1 != q2 && w1 != w2 && sig1 != sig2);
		int c3 = (d1 == 0 && w1 == 0 && w1 != w2);
		int c4 = (d1 != d2 && w1 == 1 && w1 != w2);
		//
		// Same sector?
		//
		if (d1 == d2)
		{
			// Non-overlapping?
			//
			if (!ISEQUAL(stable1->o, stable2->o))
			{
				// Fermionic x Fermionic case
				//
				if (q1 != q2)
				{
					// Annihilation
					//
					draft1->b = (stable1->b * stable2->b) % SIDE2;
					draft2->b = draft1->b;
					//
					// Reissue R1 and R2 from this
					//
					RESET(draft1->o);
					RESET(draft2->o);
				}
				//
				// Are the two cells similar?
				//
				else if (q1 == q2 && w1 == w2 && c1 == c2)
				{
					// Cohesion
					//
					if (stable1->b != stable2->b)
					{
						// Calculate the new unique bonding value
						//
						draft1->b = (stable1->b * stable2->b) % SIDE2;
						draft2->b = draft1->b;
					}
					//
					// Exchange spins: s1 <-> s2
					//
					draft1->s[0] = stable2->s[0];
					draft2->s[0] = stable1->s[0];
					draft1->s[1] = stable2->s[1];
					draft2->s[1] = stable1->s[1];
					draft1->s[2] = stable2->s[2];
					draft2->s[2] = stable1->s[2];
					//
					// Reissue R1 from pole(R1) and R2 from pole(R2)
					//
					RESET(draft1->o);
					RESET(draft2->o);
					COPY(draft1->pole, stable1->p);
					COPY(draft2->pole, stable2->p);
				}
			}
			//
			// Bosonic x Bosonic case
			//
			else if (stable1->f > 1 && stable2->f > 1)
			{
				// gluon-gluon?
				//
				if (c1 == c2 && stable1->code == GLUON && stable2->code == GLUON)
				{
					// Swap colors
					//
					int color1 = stable1->charge & C_MASK;
					int color2 = stable2->charge & C_MASK;
					draft1->charge &= ~C_MASK;
					draft2->charge &= ~C_MASK;
					draft1->charge |= color2;
					draft2->charge |= color1;
					//
					// Reissue R1 from pole(R1)
					//
					RESET(draft1->o);
					draft1->dir = 0;
					draft1->t = 0;
					draft2->dir = 0;
					draft2->t = 0;
				}
				else if (!ISNULL(stable1->p) && !ISNULL(stable2->p))
				{
					// Chiral?
					//
					if (c1 || c2 || c3 || c4)
					{
						// Reissue R1 and R2 from cstable1
						//
						draft2->b = draft1->b;
						RESET(draft1->o);
						RESET(draft2->o);
					}
					else
					{
						// TODO
					}
				}
				else if (sig1 != 0 && sig1 != 3 && sig2 != 0 && sig2 != 3)
				{
					// Swap colors
					//
					int c1 = stable1->charge & C_MASK;
					int c2 = stable2->charge & C_MASK;
					draft1->charge &= ~C_MASK;
					draft2->charge &= ~C_MASK;
					draft1->charge |= c2;
					draft2->charge |= c1;
					//
					draft2->b = draft1->b;
					//
					// Reissue R1 and R2 from cstable1
					//
					draft2->b = draft1->b;
					RESET(draft1->o);
					RESET(draft2->o);
				}
			}
			//
			// F x B
			//
			if (stable1->f == 1 && stable2->f > 1)
			{
				// F x B
				//
				if ((stable1->charge & C_MASK) != 0 && (stable1->charge & C_MASK) != C_MASK && (stable2->charge & C_MASK) != 0 &&
					(stable2->charge & C_MASK) != C_MASK)
				{
					// Swap colors
					//
					int c1 = stable1->charge & C_MASK;
					int c2 = stable2->charge & C_MASK;
					draft1->charge &= ~C_MASK;
					draft2->charge &= ~C_MASK;
					draft1->charge |= c2;
					draft2->charge |= c1;
					draft2->b = draft2->b;
					//
					// Reissue R1 from pole(R1) and R2 from pole(R2)
					//
					RESET(draft1->o);
					RESET(draft2->o);
				}
				else
				{
					draft2->b = draft1->b;
					//
					// Reissue R1 and R2 from this
					//
					RESET(draft1->pole);
					RESET(draft1->o);
					RESET(draft2->pole);
					RESET(draft2->o);
				}
			}
			else if (stable1->f > 1 && stable2->f == 1)
			{
				// B x F
				//
				if ((stable1->charge & C_MASK) != 0 && (stable1->charge & C_MASK) != C_MASK && (stable2->charge & C_MASK) != 0 &&
					(stable2->charge & C_MASK) != C_MASK)
				{
					// Swap colors
					//
					int c1 = stable1->charge & C_MASK;
					int c2 = stable2->charge & C_MASK;
					draft1->charge &= ~C_MASK;
					draft2->charge &= ~C_MASK;
					draft1->charge |= c2;
					draft2->charge |= c1;
					//
					// Reissue R1 from pole(R1) and R2 from pole(R2)
					//
					RESET(draft1->o);
					RESET(draft2->o);
				}
				else
				{
					draft2->b = draft1->b;
					//
					// Reissue R1 and R2 from this
					//
					RESET(draft1->pole);
					RESET(draft2->pole);
				}
			}
			else if (stable1->b == stable2->b)
			{
				// Messenger interactions
				//
				if (!ISNULL(stable1->p))
				{
					// REISSUE(stable, POLE(stable))
					//
					RESET(draft1->pole);
					//
					// REISSUE(draft, TRANSPORT(draft, stable));
					//
					draft2->pole[0] = draft1->o[0] - stable2->o[0];
					draft2->pole[1] = draft1->o[1] - stable2->o[1];
					draft2->pole[2] = draft1->o[2] - stable2->o[2];
				}
				else
				{
					// REISSUE(draft, POLE(draft));
					//
					RESET(draft2->pole);
					//
					// REISSUE(stable, TRANSPORT(stable, draft));
					//
					draft1->pole[0] = stable2->o[0] - stable1->o[0];
					draft1->pole[1] = stable2->o[1] - stable1->o[1];
					draft1->pole[2] = stable2->o[2] - stable1->o[2];
				}
			}
		}
		else
		{
			// Inter-sector
			//
			if ((d1 == 0 && sig1 == sig2) || (d1 == 1 && sig1 == sig3))
			{
				// Swap colors
				//
				int c1 = stable1->charge & C_MASK;
				int c2 = stable2->charge & C_MASK;
				draft1->charge &= ~C_MASK;
				draft2->charge &= ~C_MASK;
				draft1->charge |= c2;
				draft2->charge |= c1;
				//
				// Reissue R1 and R2 from this
				//
				RESET(draft1->pole);
				RESET(draft2->pole);
			}
			//
			// Chiral?
			//
			else if (q1 == q2 && w1 == w2)//c1 || c2 || c3 || c4)
			{
				int c1 = stable1->charge & W_MASK;
				int c2 = stable2->charge & W_MASK;
				draft1->charge &= ~W_MASK;
				draft2->charge &= ~W_MASK;
				draft1->charge |= c2;
				draft2->charge |= c1;
				//
				// Reissue R1 and R2 from this
				//
				RESET(draft1->pole);
				RESET(draft2->pole);
			}
		}
	}
}

__global__ void interact(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Interactions only allowed at the last tick of a light step
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		// Compare columns for interaction match
		//
		Cell* stable1 = stable;
		Cell* draft1 = draft;
		for (int i = 0; i < SIDE2; i++)
		{
			// If the re-emission cell was reached, reset the pole vector to the free bubble
			//
			if (ISNULL(stable1->pole))
			{
				COPY(draft1->pole, stable1->p);
				RESET(draft1->o);
				draft1->t = 0;
			}
			else
			{
				Cell* stable2 = stable;
				Cell* draft2 = draft;
				for (int j = 0; j < SIDE2; j++)
				{
					if (i != j && stable1->f > 0 && stable2->f > 0 &&
						stable1->b != stable2->b && !ISEQUAL(stable1->o, stable2->o))
						compareColumns(stable1, stable2, draft1, draft2);
					//
					stable2 = nextV(stable2);
					draft2 = nextV(draft2);
				}
			}
			//
			stable1 = nextV(stable1);
			draft1 = nextV(draft1);
		}
	}
}
