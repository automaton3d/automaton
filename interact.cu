#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void interact(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell *active_stack, *passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int v = 0; v < 1/*SIDE2*/; v++)
		{
			int sig1 = ((active_stack->charge ^ passive_stack->charge) & C_MASK) == 0 && (passive_stack->charge & C_MASK) == 0 &&
				((active_stack->charge ^ passive_stack->charge) & Q_MASK) == 0 && (passive_stack->charge & Q_MASK) == Q_MASK && 
				((active_stack->charge ^ passive_stack->charge) & D_MASK) == 0 && (passive_stack->charge & D_MASK) == 0;
			int sig2 = ((active_stack->charge ^ passive_stack->charge) & C_MASK) == 0 &&
				(passive_stack->charge & C_MASK) == C_MASK && ((active_stack->charge ^ passive_stack->charge) & Q_MASK) == 0 && 
				(passive_stack->charge & Q_MASK) == 0 && ((active_stack->charge ^ passive_stack->charge) & D_MASK) == 0 && (passive_stack->charge & D_MASK) == D_MASK;
			int sig3 = ((active_stack->charge ^ passive_stack->charge) & C_MASK) == 0 && (passive_stack->charge & C_MASK) != 0 && (passive_stack->charge & C_MASK) != C_MASK &&
				((active_stack->charge ^ passive_stack->charge) & Q_MASK) == 0;
			//
			int c1 = ((active_stack->charge ^ passive_stack->charge) & Q_MASK) == 0 && ((active_stack->charge ^ passive_stack->charge) & W_MASK) == 0 && sig1 == sig2;
			int c2 = ((active_stack->charge ^ passive_stack->charge) & Q_MASK) != 0 && ((active_stack->charge ^ passive_stack->charge) & W_MASK) != 0 && sig1 != sig2;
			int c3 = (active_stack->charge & D_MASK) == 0 && (active_stack->charge & W_MASK) == 0 && ((active_stack->charge ^ passive_stack->charge) & W_MASK) != 0;
			int c4 = (active_stack->charge & D_MASK) == D_MASK && (active_stack->charge & W_MASK) == W_MASK && ((active_stack->charge ^ passive_stack->charge) & W_MASK) != 0;
			//
			if(ISNULL(active_stack->pole))
				COPY(passive_stack->pole, active_stack->p);
			//
			// Play pseudo dices
			//
			if (false)//active_stack->noise > abs(active_stack->phi) && passive_stack->noise > abs(passive_stack->phi) && (!ISNULL(active_stack->p) || !ISNULL(passive_stack->p)))
			{
				if (((active_stack->charge ^ passive_stack->charge) & D_MASK) == 0)
				{
					// Same sector?
					//
					if (active_stack->f == 1 && passive_stack->f == 1)
					{
						// F x F
						//
						if (((active_stack->charge ^ passive_stack->charge) & Q_MASK) != 0)
						{
							// Annihilation?
							//
							active_stack->b = (active_stack->b * passive_stack->b) % SIDE2; // ??? erro ???
							passive_stack->b = active_stack->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(active_stack->pole);// ???
							COPY(passive_stack->pole, active_stack->p);
						}
						else if (sig1 || sig2 || sig3)
						{
							// Similar?
							//
							// Cohesion
							//
							if (active_stack->b != passive_stack->b)
							{
								active_stack->b = (active_stack->b * passive_stack->b) % SIDE2;
								passive_stack->b = active_stack->b;
							}
							//
							// s1 <-> s2
							//
							int temp;
							temp = active_stack->s[0];
							passive_stack->s[0] = active_stack->s[0];
							active_stack->s[0] = temp;
							temp = active_stack->s[1];
							passive_stack->s[1] = active_stack->s[1];
							active_stack->s[1] = temp;
							temp = active_stack->s[2];
							passive_stack->s[2] = active_stack->s[2];
							active_stack->s[2] = temp;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(active_stack->o);
							RESET(passive_stack->o);	//???
							COPY(passive_stack->pole, active_stack->p); // ???
						}
					}
					else if (active_stack->f > 1 && passive_stack->f > 1)
					{
						// B x B
						//
						if (((active_stack->charge ^ ~passive_stack->charge) & C_MASK) == 0 && active_stack->code == passive_stack->code && passive_stack->code == GLUON)
						{
							// gluon-gluon?
							//
							// Swap colors
							//
							int temp = active_stack->charge & C_MASK;
							active_stack->charge &= ~C_MASK;
							active_stack->charge |= (passive_stack->charge & C_MASK);
							passive_stack->charge &= ~C_MASK;
							passive_stack->charge |= temp;
							//
							// Reissue R1 from pole(R1)
							//
							RESET(active_stack->o);
							passive_stack->d0 = 0;	// replicar !!!
							passive_stack->t = 0;	// replicar !!!
						}
						else if (!ISNULL(active_stack->p) && !ISNULL(passive_stack->p))
						{
							if (c1 || c2 || c3 || c4)
							{
								// chiral?
								//
								// Reissue R1 and R2 from cp1
								//
								passive_stack->b = active_stack->b;
								RESET(active_stack->o);
								//
								// Reissue R2 from cp1
								//
								RESET(passive_stack->o);
							}
							else
							{
								// TODO
							}
						}
						else if (sig1 != 0 && sig1 != 3 && sig2 != 0 && sig2 != 3)
						{
							int temp = active_stack->charge & C_MASK;
							active_stack->charge &= ~C_MASK;
							active_stack->charge |= (passive_stack->charge & C_MASK);
							passive_stack->charge &= ~C_MASK;
							passive_stack->charge |= temp;
							passive_stack->b = active_stack->b;
							//
							// Reissue R1 and R2 from cp1
							//
							passive_stack->b = active_stack->b;
							RESET(active_stack->o);
							RESET(passive_stack->o);
						}
					}
					if (active_stack->f == 1 && passive_stack->f > 1)
					{
						// F x B
						//
						if ((active_stack->charge & C_MASK) != 0 && (active_stack->charge & C_MASK) != C_MASK && (passive_stack->charge & C_MASK) != 0 && 
							(passive_stack->charge & C_MASK) != C_MASK)
						{
							int temp = active_stack->charge & C_MASK;
							active_stack->charge &= ~C_MASK;
							active_stack->charge |= (passive_stack->charge & C_MASK);
							passive_stack->charge &= ~C_MASK;
							passive_stack->charge |= temp;
							passive_stack->b = active_stack->b;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(active_stack->o);
							RESET(passive_stack->o);
						}
						else
						{
							passive_stack->b = active_stack->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(active_stack->pole);
							RESET(active_stack->o);
							RESET(passive_stack->pole);
							RESET(passive_stack->o);
						}
					}
					else if (active_stack->f > 1 && passive_stack->f == 1)
					{
						// B x F
						//
						if ((active_stack->charge & C_MASK) != 0 && (active_stack->charge & C_MASK) != C_MASK && (passive_stack->charge & C_MASK) != 0 && 
							(passive_stack->charge & C_MASK) != C_MASK)
						{
							int temp = active_stack->charge & C_MASK;
							active_stack->charge &= ~C_MASK;
							active_stack->charge |= (passive_stack->charge & C_MASK);
							passive_stack->charge &= ~C_MASK;
							passive_stack->charge |= temp;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(active_stack->o);
							RESET(passive_stack->o);
						}
						else
						{
							passive_stack->b = active_stack->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(active_stack->pole);
							RESET(passive_stack->pole);
						}
					}
					else if (active_stack->b == passive_stack->b)
					{
						// Messenger interactions
						//
						if (!ISNULL(active_stack->p))
						{
							// REISSUE(active_stack, POLE(active_stack))
							//
							RESET(active_stack->pole);
							//
							// REISSUE(passive_stack, TRANSPORT(passive_stack, active_stack));
							//
							passive_stack->pole[0] = active_stack->o[0] - passive_stack->o[0];
							passive_stack->pole[1] = active_stack->o[1] - passive_stack->o[1];
							passive_stack->pole[2] = active_stack->o[2] - passive_stack->o[2];
						}
						else
						{
							// REISSUE(passive_stack, POLE(passive_stack));
							//
							RESET(passive_stack->pole);
							//
							// REISSUE(active_stack, TRANSPORT(active_stack, passive_stack));
							//
							active_stack->pole[0] = passive_stack->o[0] - active_stack->o[0];
							active_stack->pole[1] = passive_stack->o[1] - active_stack->o[1];
							active_stack->pole[2] = passive_stack->o[2] - active_stack->o[2];
						}
					}
				}
				else
				{
					// Inter-sector
					//
					if ((((active_stack->charge & D_MASK) == 0 && sig1 == 2) || ((active_stack->charge & D_MASK) == 1 && sig1 == 3)))
					{
						int temp = active_stack->charge & C_MASK;
						active_stack->charge &= ~C_MASK;
						active_stack->charge |= (passive_stack->charge & C_MASK);
						passive_stack->charge &= ~C_MASK;
						passive_stack->charge |= temp;
						//
						// Reissue R1 and R2 from this
						//
						RESET(active_stack->pole);
						RESET(passive_stack->pole);
					}
					else if (c1 || c2 || c3 || c4)
					{
						// Chiral?
						//
						int temp = active_stack->charge & W_MASK;
						active_stack->charge &= ~W_MASK;
						active_stack->charge |= (passive_stack->charge & W_MASK);
						passive_stack->charge &= ~W_MASK;
						passive_stack->charge |= temp;
						//
						// Reissue R1 and R2 from this
						//
						RESET(active_stack->pole);
						RESET(passive_stack->pole);
					}
				}
			}
			active_stack->t = 0;
			passive_stack->t = 0;
			active_stack->synch = -1;
			passive_stack->synch = -1;
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	}
}
