#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void interact(struct Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		struct Cell* cell = lattice + id;
		int sig1 = cell->h->c == cell->c && cell->c == 0 && cell->h->q == cell->q && cell->q == 1 && cell->h->d == cell->d && cell->d == 0;
		int sig2 = cell->h->c == cell->c && cell->c == 7 && cell->h->q == cell->q && cell->q == 0 && cell->h->d == cell->d && cell->d == 1;
		int sig3 = cell->h->c == cell->c && cell->c != 0 && cell->c != 7 && cell->h->q == cell->q;
		//
		int c1 = cell->h->q == cell->q && cell->h->w == cell->w && sig1 == sig2;
		int c2 = cell->h->q != cell->q && cell->h->w != cell->w && sig1 != sig2;
		int c3 = cell->h->d == 0 && cell->h->w == 0 && cell->h->w != cell->w;
		int c4 = cell->h->d == 1 && cell->h->w == 1 && cell->h->w != cell->w;
		//
		// Play pseudo dices
		//
		if (cell->h->noise > abs(cell->h->phi) && cell->noise > abs(cell->phi) && (!ISNULL(cell->h->p) || !ISNULL(cell->p)))
		{
			if (cell->h->d == cell->d)
			{
				// Same sector?
				//
				if (cell->h->f == 1 && cell->f == 1)
				{
					// F x F
					//
					if (cell->h->q != cell->q)
					{
						// Annihilation?
						//
						cell->h->b = (cell->h->b * cell->b) % SIDE2;
						cell->b = cell->h->b;
						//
						// Reissue R1 and R2 from this
						//
						RESET(cell->h->pole);
						RESET(cell->pole);
					}
					else if (sig1 || sig2 || sig3)
					{
						// Similar?
						//
						// Cohesion
						//
						if (cell->h->b != cell->b)
						{
							cell->h->b = (cell->h->b * cell->b) % SIDE2;
							cell->b = cell->h->b;
						}
						//
						// s1 <-> s2
						//
						int temp;
						temp = cell->h->s[0];
						cell->s[0] = cell->h->s[0];
						cell->h->s[0] = temp;
						temp = cell->h->s[1];
						cell->s[1] = cell->h->s[1];
						cell->h->s[1] = temp;
						temp = cell->h->s[2];
						cell->s[2] = cell->h->s[2];
						cell->h->s[2] = temp;
						//
						// Reissue R1 from pole(R1) and R2 from pole(R2)
						//
						RESET(cell->h->o);
						RESET(cell->o);
					}
				}
				else if (cell->h->f > 1 && cell->f > 1)
				{
					// B x B
					//
					if (cell->h->c == ~cell->c && cell->h->code == cell->code && cell->code == GLUON)
					{
						// gluon-gluon?
						//
						// Swap colors
						//
						int temp = cell->h->c;
						cell->h->c = cell->c;
						cell->c = temp;
						//
						// Reissue R1 from pole(R1)
						//
						RESET(cell->h->o);
					}
					else if (!ISNULL(cell->h->p) && !ISNULL(cell->p))
					{
						if (c1 || c2 || c3 || c4)
						{
							// chiral?
							//
							// Reissue R1 and R2 from cp1
							//
							cell->b = cell->h->b;
							RESET(cell->h->o);
							//
							// Reissue R2 from cp1
							//
							RESET(cell->o);
						}
						else
						{
							// TODO
						}
					}
					else if (sig1 != 0 && sig1 != 3 && sig2 != 0 && sig2 != 3)
					{
						int temp = cell->h->c;
						cell->h->c = cell->c;
						cell->c = temp;
						cell->b = cell->h->b;
						//
						// Reissue R1 and R2 from cp1
						//
						cell->b = cell->h->b;
						RESET(cell->h->o);
						RESET(cell->o);
					}
				}
				if (cell->h->f == 1 && cell->f > 1)
				{
					// F x B
					//
					if (cell->h->c != 0 && cell->h->c != 7 && cell->c != 0 && cell->c != 7)
					{
						int temp = cell->h->c;
						cell->h->c = cell->c;
						cell->c = temp;
						cell->b = cell->h->b;
						//
						// Reissue R1 from pole(R1) and R2 from pole(R2)
						//
						RESET(cell->h->o);
						RESET(cell->o);
					}
					else
					{
						cell->b = cell->h->b;
						//
						// Reissue R1 and R2 from this
						//
						RESET(cell->h->pole);
						RESET(cell->h->o);
						RESET(cell->pole);
						RESET(cell->o);
					}
				}
				else if (cell->h->f > 1 && cell->f == 1)
				{
					//
					// B x F
					//
					if (cell->h->c != 0 && cell->h->c != 7 && cell->c != 0 && cell->c != 7)
					{
						int temp = cell->h->c;
						cell->h->c = cell->c;
						cell->c = temp;
						cell->b = cell->h->b;
						//
						// Reissue R1 from pole(R1) and R2 from pole(R2)
						//
						RESET(cell->h->o);
						RESET(cell->o);
					}
					else
					{
						cell->b = cell->h->b;
						//
						// Reissue R1 and R2 from this
						//
						RESET(cell->h->pole);
						RESET(cell->pole);
					}
				}
				else if (cell->h->b == cell->b)
				{
					// Messenger interactions
					//
					if (!ISNULL(cell->h->p))
					{
						// REISSUE(cell->h, POLE(cell->h))
						//
						RESET(cell->h->pole);
						//
						// REISSUE(cell, TRANSPORT(cell, cell->h));
						//
						cell->pole[0] = cell->h->o[0] - cell->o[0];
						cell->pole[1] = cell->h->o[1] - cell->o[1];
						cell->pole[2] = cell->h->o[2] - cell->o[2];
					}
					else
					{
						// REISSUE(cell, POLE(cell));
						//
						RESET(cell->pole);
						//
						// REISSUE(cell->h, TRANSPORT(cell->h, cell));
						//
						cell->h->pole[0] = cell->o[0] - cell->h->o[0];
						cell->h->pole[1] = cell->o[1] - cell->h->o[1];
						cell->h->pole[2] = cell->o[2] - cell->h->o[2];
					}
				}
			}
			else
			{
				// Inter-sector
				//
				if (((cell->h->d == 0 && sig1 == 2) || (cell->h->d == 1 && sig1 == 3)))
				{
					int temp = cell->h->c;
					cell->h->c = cell->c;
					cell->c = temp;
					//
					// Reissue R1 and R2 from this
					//
					RESET(cell->h->pole);
					RESET(cell->pole);
				}
				else if (c1 || c2 || c3 || c4)
				{
					// Chiral?
					//
					int temp = cell->h->w;
					cell->h->w = cell->w;
					cell->w = temp;
					//
					// Reissue R1 and R2 from this
					//
					RESET(cell->h->pole);
					RESET(cell->pole);
				}
			}
		}
	}
	/*
	else {
		if ((cell->t / LIGHT) % 2 == 1 && cell->t > 1 && cell->t < TMAX) {
			//
			// Shift 'vertically'
			//
			cell->h->h = cell->h->v->h;
			cell->h->v = cell->h->v->v;
		}
	}
	//
	// Update time counter
	//
	if (cell->t <= LIGHT)
		cell->t--;
	else
		cell->t -= LIGHT;
	if (cell->t == 0) {
		cell->t = TMAX;
		cell->active = !cell->active;
	}
	*/
}
