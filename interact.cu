#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

__device__ unsigned int rnd;

__global__ void interact(Cell* lattice)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell *stable, *draft;
		if (cell->active)
		{
			stable = cell;
			draft = cell + SIDE3 * SIDE2;
		}
		else
		{
			draft = cell;
			stable = cell + SIDE3 * SIDE2;
		}
		//
		// Interactions only allowed at the last tick of a light step
		//
		if (draft->t % LIGHT != 0)
		{
			return;
		}
		hiprandState state;
		// hiprand_init(0, id, 0, &state);

		//
		// Compare the two columns for interaction matches
		//
		for (int v = 0; v < SIDE2; v++)
		{
			#ifdef INTERACT
			int sig1 = ((stable->charge ^ draft->charge) & C_MASK) == 0 && (draft->charge & C_MASK) == 0 &&
				((stable->charge ^ draft->charge) & Q_MASK) == 0 && (draft->charge & Q_MASK) == Q_MASK && 
				((stable->charge ^ draft->charge) & D_MASK) == 0 && (draft->charge & D_MASK) == 0;
			int sig2 = ((stable->charge ^ draft->charge) & C_MASK) == 0 &&
				(draft->charge & C_MASK) == C_MASK && ((stable->charge ^ draft->charge) & Q_MASK) == 0 && 
				(draft->charge & Q_MASK) == 0 && ((stable->charge ^ draft->charge) & D_MASK) == 0 && (draft->charge & D_MASK) == D_MASK;
			int sig3 = ((stable->charge ^ draft->charge) & C_MASK) == 0 && (draft->charge & C_MASK) != 0 && (draft->charge & C_MASK) != C_MASK &&
				((stable->charge ^ draft->charge) & Q_MASK) == 0;
			//
			int c1 = ((stable->charge ^ draft->charge) & Q_MASK) == 0 && ((stable->charge ^ draft->charge) & W_MASK) == 0 && sig1 == sig2;
			int c2 = ((stable->charge ^ draft->charge) & Q_MASK) != 0 && ((stable->charge ^ draft->charge) & W_MASK) != 0 && sig1 != sig2;
			int c3 = (stable->charge & D_MASK) == 0 && (stable->charge & W_MASK) == 0 && ((stable->charge ^ draft->charge) & W_MASK) != 0;
			int c4 = (stable->charge & D_MASK) == D_MASK && (stable->charge & W_MASK) == W_MASK && ((stable->charge ^ draft->charge) & W_MASK) != 0;
			//
			if(ISNULL(stable->pole))
				COPY(draft->pole, stable->p);
			//
			// Play pseudo dices
			//
			if (stable->noise > abs(stable->phi) && draft->noise > abs(draft->phi) && (!ISNULL(stable->p) || !ISNULL(draft->p)))
			{
				if (((stable->charge ^ draft->charge) & D_MASK) == 0)
				{
					// Same sector?
					//
					if (stable->f == 1 && draft->f == 1)
					{
						// F x F
						//
						if (((stable->charge ^ draft->charge) & Q_MASK) != 0)
						{
							// Annihilation?
							//
							stable->b = (stable->b * draft->b) % SIDE2; // ??? erro ???
							draft->b = stable->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(stable->pole);// ???
							COPY(draft->pole, stable->p);
						}
						else if (sig1 || sig2 || sig3)
						{
							// Similar?
							//
							// Cohesion
							//
							if (stable->b != draft->b)
							{
								stable->b = (stable->b * draft->b) % SIDE2;
								draft->b = stable->b;
							}
							//
							// s1 <-> s2
							//
							int temp;
							temp = stable->s[0];
							draft->s[0] = stable->s[0];
							stable->s[0] = temp;
							temp = stable->s[1];
							draft->s[1] = stable->s[1];
							stable->s[1] = temp;
							temp = stable->s[2];
							draft->s[2] = stable->s[2];
							stable->s[2] = temp;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(stable->o);
							RESET(draft->o);	//???
							COPY(draft->pole, stable->p); // ???
						}
					}
					else if (stable->f > 1 && draft->f > 1)
					{
						// B x B
						//
						if (((stable->charge ^ ~draft->charge) & C_MASK) == 0 && stable->code == draft->code && draft->code == GLUON)
						{
							// gluon-gluon?
							//
							// Swap colors
							//
							int temp = stable->charge & C_MASK;
							stable->charge &= ~C_MASK;
							stable->charge |= (draft->charge & C_MASK);
							draft->charge &= ~C_MASK;
							draft->charge |= temp;
							//
							// Reissue R1 from pole(R1)
							//
							RESET(stable->o);
							draft->dir = 0;	// replicar !!!
							draft->t = 0;	// replicar !!!
						}
						else if (!ISNULL(stable->p) && !ISNULL(draft->p))
						{
							if (c1 || c2 || c3 || c4)
							{
								// chiral?
								//
								// Reissue R1 and R2 from cp1
								//
								draft->b = stable->b;
								RESET(stable->o);
								//
								// Reissue R2 from cp1
								//
								RESET(draft->o);
							}
							else
							{
								// TODO
							}
						}
						else if (sig1 != 0 && sig1 != 3 && sig2 != 0 && sig2 != 3)
						{
							int temp = stable->charge & C_MASK;
							stable->charge &= ~C_MASK;
							stable->charge |= (draft->charge & C_MASK);
							draft->charge &= ~C_MASK;
							draft->charge |= temp;
							draft->b = stable->b;
							//
							// Reissue R1 and R2 from cp1
							//
							draft->b = stable->b;
							RESET(stable->o);
							RESET(draft->o);
						}
					}
					if (stable->f == 1 && draft->f > 1)
					{
						// F x B
						//
						if ((stable->charge & C_MASK) != 0 && (stable->charge & C_MASK) != C_MASK && (draft->charge & C_MASK) != 0 && 
							(draft->charge & C_MASK) != C_MASK)
						{
							int temp = stable->charge & C_MASK;
							stable->charge &= ~C_MASK;
							stable->charge |= (draft->charge & C_MASK);
							draft->charge &= ~C_MASK;
							draft->charge |= temp;
							draft->b = stable->b;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(stable->o);
							RESET(draft->o);
						}
						else
						{
							draft->b = stable->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(stable->pole);
							RESET(stable->o);
							RESET(draft->pole);
							RESET(draft->o);
						}
					}
					else if (stable->f > 1 && draft->f == 1)
					{
						// B x F
						//
						if ((stable->charge & C_MASK) != 0 && (stable->charge & C_MASK) != C_MASK && (draft->charge & C_MASK) != 0 && 
							(draft->charge & C_MASK) != C_MASK)
						{
							int temp = stable->charge & C_MASK;
							stable->charge &= ~C_MASK;
							stable->charge |= (draft->charge & C_MASK);
							draft->charge &= ~C_MASK;
							draft->charge |= temp;
							//
							// Reissue R1 from pole(R1) and R2 from pole(R2)
							//
							RESET(stable->o);
							RESET(draft->o);
						}
						else
						{
							draft->b = stable->b;
							//
							// Reissue R1 and R2 from this
							//
							RESET(stable->pole);
							RESET(draft->pole);
						}
					}
					else if (stable->b == draft->b)
					{
						// Messenger interactions
						//
						if (!ISNULL(stable->p))
						{
							// REISSUE(stable, POLE(stable))
							//
							RESET(stable->pole);
							//
							// REISSUE(draft, TRANSPORT(draft, stable));
							//
							draft->pole[0] = stable->o[0] - draft->o[0];
							draft->pole[1] = stable->o[1] - draft->o[1];
							draft->pole[2] = stable->o[2] - draft->o[2];
						}
						else
						{
							// REISSUE(draft, POLE(draft));
							//
							RESET(draft->pole);
							//
							// REISSUE(stable, TRANSPORT(stable, draft));
							//
							stable->pole[0] = draft->o[0] - stable->o[0];
							stable->pole[1] = draft->o[1] - stable->o[1];
							stable->pole[2] = draft->o[2] - stable->o[2];
						}
					}
				}
				else
				{
					// Inter-sector
					//
					if ((((stable->charge & D_MASK) == 0 && sig1 == 2) || ((stable->charge & D_MASK) == 1 && sig1 == 3)))
					{
						int temp = stable->charge & C_MASK;
						stable->charge &= ~C_MASK;
						stable->charge |= (draft->charge & C_MASK);
						draft->charge &= ~C_MASK;
						draft->charge |= temp;
						//
						// Reissue R1 and R2 from this
						//
						RESET(stable->pole);
						RESET(draft->pole);
					}
					else if (c1 || c2 || c3 || c4)
					{
						// Chiral?
						//
						int temp = stable->charge & W_MASK;
						stable->charge &= ~W_MASK;
						stable->charge |= (draft->charge & W_MASK);
						draft->charge &= ~W_MASK;
						draft->charge |= temp;
						//
						// Reissue R1 and R2 from this
						//
						RESET(stable->pole);
						RESET(draft->pole);
					}
				}
			}
			#endif
			if (!ISNULL(stable->p) && !ISNULL(stable->o))
			{
				rnd = hiprand(&state);
				if (rnd % 100 < 50 && draft->t > 0)
				{
					rnd = hiprand(&state);
					//printf("t=%d LIGHT=%d: %d\n", draft->t , draft->t / LIGHT, rnd);
				}
			}
			//
			// Next register
			//
			stable = nextV(stable);
			draft = nextV(draft);
		}
	}
}
