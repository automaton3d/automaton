#include "hip/hip_runtime.h"
/*
 * Interaction routines
 * 
 */
#include "hip/hip_runtime.h"
#include ""
#include "automaton.cuh"

/*
 * Both re-emmited from CP.
 */
__device__ void cpcp(Cell* draft1, Cell* draft2, bool collapse)
{
	// Copy momentum information for flash
	//
	if (ALIGNED(draft1->o, draft1->p))
	{
		COPY(draft1->pole, draft1->p);
		COPY(draft2->pole, draft1->p);
	}
	else
	{
		COPY(draft1->pole, draft2->p);
		COPY(draft2->pole, draft2->p);
	}
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
	//
	if (collapse)
	{
		// Disintegrate the packet
		//
		draft1->code = COLLAPSE;
		draft2->code = COLLAPSE;
	}
}

/*
 * Both re-emmited from respective pole.
 */
__device__ void polepole(Cell* draft1, Cell* draft2)
{
	// Copy momentum information for flash
	//
	COPY(draft1->pole, draft1->p);
	COPY(draft2->pole, draft2->p);
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
}

/*
 * Inertia mechanism.
 */
__device__ void inertia(Cell* draft1, Cell* draft2)
{
	// Copy momentum information for flash
	//
	if (ALIGNED(draft1->o, draft1->p))
	{
		COPY(draft1->pole, draft2->p);
		COPY(draft2->pole, draft2->p);
	}
	else
	{
		COPY(draft1->pole, draft1->p);
		COPY(draft2->pole, draft1->p);
	}
	//
	// Start flash flooding
	//
	draft1->flash = SIDE;
	draft2->flash = SIDE;
}

__device__ void bosonxboson(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Same sector?
	//
	if (((stable1->chrg ^ stable2->chrg) & D_MASK) == 0)
	{
		// Gluon x gluon?
		//
		if ((stable1->chrg & C_MASK) != 0 && (stable1->chrg & C_MASK) != C_MASK &&
			(stable2->chrg & C_MASK) != 0 && (stable2->chrg & C_MASK) != C_MASK)
		{
			// Exchange colors, ignoring other charges
			//
			draft1->chrg &= ~C_MASK;
			draft2->chrg &= ~C_MASK;
			draft1->chrg |= stable2->chrg & C_MASK;
			draft2->chrg |= stable1->chrg & C_MASK;
			//
			polepole(draft1, draft2);
		}
		//
		// Neutral 1?
		//
		else if ((stable1->chrg & C_MASK) == 0)
		{
			if((stable1->code & Q_MASK) == 1)
			{
				// Photon 1 or Z 1
				//
				if ((stable1->code & W_MASK) == 1)
				{
					// Photon 1
					//
					if ((stable2->code & W_MASK) == 0 && (stable2->code & Q_MASK) == 0 && (((stable2->chrg>>1) ^ stable2->chrg) & 1) == 0)
					{
						// Photon 1 x W 2
						//
						polepole(draft1, draft2);
						// TODO
					}
				}
				else
				{
					// Z 1
					//
					if ((stable2->code & W_MASK) == 0 && (stable2->code & Q_MASK) == 0 && (((stable2->chrg >> 1) ^ stable2->chrg) & 1) == 0)
					{
						// Z 1 x W 2
						//
						polepole(draft1, draft2);
						// TODO
					}
				}
			}
		}
		else if ((stable2->chrg & C_MASK) == 0)
		{
			if ((stable2->code & Q_MASK) == 1)
			{
				// Photon 2 or Z 2
				//
				if ((stable2->code & W_MASK) == 1)
				{
					// Photon 2
					//
					if ((stable1->code & W_MASK) == 0 && (stable1->code & Q_MASK) == 0 && (((stable1->chrg >> 1) ^ stable1->chrg) & 1) == 0)
					{
						// Photon 2 x W 1
						//
						polepole(draft1, draft2);
						// TODO
					}
				}
				else
				{
					// Z 2
					//
					if ((stable1->code & W_MASK) == 0 && (stable1->code & Q_MASK) == 0 && (((stable1->chrg >> 1) ^ stable1->chrg) & 1) == 0)
					{
						// Z 2 x W 1
						//
						polepole(draft1, draft2);
						// TODO
					}
				}
			}
		}
	}
}

__device__ void fermionxboson(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->chrg & C_MASK) & C_MASK;
	unsigned q1 = ((stable1->chrg & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->chrg & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->chrg & D_MASK) >> 5;
	unsigned c2 = (stable2->chrg & C_MASK) & C_MASK;
	unsigned q2 = ((stable2->chrg & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->chrg & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->chrg & D_MASK) >> 5;
	//
	if (d1 != d2)
	{
		// SUPRESSED
		// (Only same sector are allowed to interact in this way)
		//
		return;
	}
	//
	// Quark x gluon?
	//
	if (c1 != NEUTRAL && c1 != ~NEUTRAL && c2 != NEUTRAL && c2 != ~NEUTRAL)
	{
		if (c1 == ~c2)		// TODO why??
		{
			// Blindly exchange colors, ignoring all other charges
			//
			draft1->chrg &= ~C_MASK;
			draft2->chrg &= ~C_MASK;
			draft1->chrg |= c2;
			draft2->chrg |= c1;
			//
			polepole(draft1, draft2);
		}
	}
	//
	// Quark x [photon, Z, W]?
	//
	else if (c1 != NEUTRAL && c1 != ~NEUTRAL)
	{
		// Is it a propeller?
		//
		if (stable1->a == stable2->a)
		{
			// Inertia
			//
			inertia(draft1, draft2);
		}
		else if (q1 == q2)
		{
			polepole(draft1, draft2);
		}
		else
		{
			polepole(draft1, draft2);
		}
	}
	//
	// Electron x [photon, Z, W]?
	//
	else
	{
		// Is it a propeller?
		//
		if (stable1->a == stable2->a)
		{
			// Inertia
			//
			inertia(draft1, draft2);
		}
		else if (q1 == q2)
		{
			if (w1 == w2)
			{
				if (c1 == c2 == 0 && w1 == 1)
				{
					polepole(draft1, draft2);
				}
				else if (c1 == c2 == C_MASK && w1 == 0)
				{
					polepole(draft1, draft2);
				}
			}
		}
		else
		{
			if (w1 == w2)
			{
				if (c1 == c2 == 0 && w1 == 1)
				{
					polepole(draft1, draft2);
				}
				else if (c1 == c2 == C_MASK && w1 == 0)
				{
					polepole(draft1, draft2);
				}
			}
		}
	}
}

__device__ void fermionxfermion(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Isolate charge bits
	//
	unsigned c1 = (stable1->chrg & C_MASK) & 7;
	unsigned q1 = ((stable1->chrg & Q_MASK) >> 3) & 1;
	unsigned w1 = ((stable1->chrg & W_MASK) >> 4) & 1;
	unsigned d1 = (stable1->chrg & D_MASK) >> 5;
	unsigned c2 = (stable2->chrg & C_MASK) & 7;
	unsigned q2 = ((stable2->chrg & Q_MASK) >> 3) & 1;
	unsigned w2 = ((stable2->chrg & W_MASK) >> 4) & 1;
	unsigned d2 = (stable2->chrg & D_MASK) >> 5;
	//
	//
	// Matter/antimatter flags
	//
	#ifdef SOL1
	bool matter1 = ((stable1->chrg >> 2) | ((stable1->chrg >> 1) & stable1->chrg)) & 1;
	bool matter1 = ((stable2->chrg >> 2) | ((stable2->chrg >> 1) & stable2->chrg)) & 1;
	#else
	bool matter1 = c1 == 0 || c1 == 1 || c1 == 2 || c1 == 4;
	bool matter2 = c2 == 0 || c2 == 1 || c2 == 2 || c2 == 4;
	#endif
	//
	// Same sector?
	//
	if (d1 == d2)
	{
		// quark x quark?
		//
		if (c1 != NEUTRAL && c1 != ~NEUTRAL && c2 != NEUTRAL && c2 != ~NEUTRAL)
		{
			// Same color and electric charge?
			//
			if (c1 == c2 && q1 == q2)	// TODO: include affinity?? weak charge??
			{
				// Quark cohesion
				//
				polepole(draft1, draft2);
			}
			//
			// Complementary charges?
			//
			else if (c1 == ~c2 && w1 == ~w2 && q1 == ~q2)
			{
				// Quark annihilation?
				//
				cpcp(draft1, draft2, true);
			}
		}
		//
		// quark x electron?
		//
		else if (c1 != NEUTRAL && c1 != ~NEUTRAL)
		{
			if (q1 == q2 && matter1 == matter2)
			{
				// Implement repulsion
			}
			else if (q1 != q2 && matter1 == matter2)
			{
				// Implement attraction
			}
		}
		//
		// Electron x electron
		//
		else if ((c1 == NEUTRAL || c1 == ~NEUTRAL) && c1 == c2)
		{
			if (q1 == q2 && matter1 == matter2)
			{
				// Implement cohesion
				//
				polepole(draft1, draft2);
			}
			else if (q1 != q2 && w1 != w2 && matter1 != matter2)
			{
				// Electron annihilation?
				//
				cpcp(draft1, draft2, true);
			}
		}
	}
	//
	// Different sectors
	//
	else
	{
		bool s1 = (c1 == c2 == 0 && q1 == q2 == 1 && d1 == d2 == 0);
		bool s2 = (c1 == c2 == 7 && q1 == q2 == 0 && d1 == d2 == 1);
		bool s3 = (c1 == c2 != 0 != 7 && q1 == q2);
		//
		bool c1 = (q1 == q2 && w1 == w2 && s1 == s2);
		bool c2 = (q1 != q2 && w1 != w2 && s1 != s2);
		bool c3 = (d1 == 0 && w1 == 0 && w1 != w2);
		bool c4 = (d1 == 1 && w1 == 1 && w1 != w2);
		//
		if ((d1 == 0 && s1 == s2) || (d1 == 1 && s1 == s3 && s1 != s2))
		{
			// Swap colors
			//
			int c1 = stable1->chrg & C_MASK;
			int c2 = stable2->chrg & C_MASK;
			draft1->chrg &= ~C_MASK;
			draft2->chrg &= ~C_MASK;
			draft1->chrg |= c2;
			draft2->chrg |= c1;
			//
			cpcp(draft1, draft2, true);
		}
		//
		// Chiral?
		//
		else if (c1  ||  c2  ||  c3  ||  c4) 
		{
			// Change hands
			//
			int w1 = stable1->chrg & W_MASK;
			int w2 = stable2->chrg & W_MASK;
			draft1->chrg &= ~W_MASK;
			draft2->chrg &= ~W_MASK;
			draft1->chrg |= w2;
			draft2->chrg |= w1;
			//
			polepole(draft1, draft2);
		}
	}
}

/*
 * Compares two cells in adjacent columns. 
 */
__device__ void compareCols(Cell* stable1, Cell* stable2, Cell* draft1, Cell* draft2)
{
	// Play pseudo dices against the sine phase
	//
	if (stable1->noise > abs(stable1->v) &&
		stable2->noise > abs(stable1->v) &&
		(!ISNULL(stable1->p) || !ISNULL(stable2->p)))
	{
		// Preserve momentum for parallel transport
		//
		COPY(draft1->pole, stable2->p);
		COPY(draft2->pole, stable1->p);
		//
		if (stable1->code == BOSON && stable2->code == BOSON)
			bosonxboson(stable1, stable2, draft1, draft2);
		else if (stable1->code == BOSON && stable2->code == FERMION)
			fermionxboson(stable2, stable1, draft2, draft1);
		else if (stable1->code == FERMION && stable2->code == BOSON)
			fermionxboson(stable1, stable2, draft1, draft2);
		else
			fermionxfermion(stable1, stable2, draft1, draft2);
	}
}

/*
 * Confronts cells for interactions. 
 */
__global__ void interact(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		//
		// Interactions only allowed at the last tick of a light step
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		// Compare columns for interaction match
		//
		Cell* stable1 = stable;
		Cell* draft1 = draft;
		for (int i = 0; i < SIDE2; i++)
		{
			// If the re-emission cell was reached, reset the pole vector to the free bubble
			//
		//	if (stable1->flash)
			//{
				//RESET(draft1->o);	// ????
				//draft1->t = 0;		// ????
		//	}
			//else
			//{
				Cell* stable2 = stable;
				Cell* draft2 = draft;
				for (int j = 0; j < SIDE2; j++)
				{
					if (i != j && stable1->f > 0 && stable2->f > 0 &&
						stable1->a != stable2->a && !ISEQUAL(stable1->o, stable2->o))
						compareCols(stable1, stable2, draft1, draft2);
					//
					stable2 = nextV(stable2);
					draft2 = nextV(draft2);
				}
			//}
			//
			stable1 = nextV(stable1);
			draft1 = nextV(draft1);
		}
	}
}
