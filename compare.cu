#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void compare(struct Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		struct Cell* root = lattice + id;
		struct Cell *active_stack, *passive_stack;
		if (root->active)
		{
			active_stack = root;
			passive_stack = root->h;
		}
		else
		{
			active_stack = root->h;
			passive_stack = root;
		}
		struct Cell* active_cell = active_stack;
		struct Cell* passive_cell = passive_stack;;
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically'
			//
			passive_cell = passive_stack;
			Cell t1;
			Cell t2 = *passive_cell;
			for (int j = 0; j < SIDE2; j++)
			{
				t1 = *passive_cell->v;
				passive_cell->f = t2.f;
				passive_cell->b = t2.b;
				passive_cell->q = t2.q;
				passive_cell->w = t2.w;
				passive_cell->c = t2.c;
				passive_cell->d = t2.d;
				COPY(passive_cell->o, t2.o);
				COPY(passive_cell->p, t2.p);
				COPY(passive_cell->s, t2.s);
				passive_cell->phi = t2.phi;
				passive_cell->code = t2.code;
				t2 = t1;
				passive_cell = passive_cell->v;
			}
			//
			// Compare 'columns'
			//
			passive_cell = passive_stack;
			for (int j = 0; j < SIDE2; j++)
			{
				if (active_cell->b == passive_cell->b && ISEQUAL(active_cell->o, passive_cell->o))
				{
					if (passive_cell->code == 0)
					{
						if (passive_cell->c == ~active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = PHOTON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = GLUON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = NEUTRINO;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = Z;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == active_cell->q)
							passive_cell->code = W;
						//
						if (passive_cell->code != 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code)
					{
						passive_cell->f++;
					}
				}
				active_cell = active_cell->v;
				passive_cell = passive_cell->v;
			}
		}
	}
}
