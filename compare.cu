#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void compare(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* active_stack, * passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically'
			//
			cell = passive_stack + (SIDE2-2)*SIDE3;
			Cell* nextv = cell + SIDE3;
			Cell temp = *nextv;
			for (int j = 0; j < SIDE2; j++)
			{
				if (j == SIDE2 - 1)
					cell = &temp;
				nextv->f = cell->f;
				nextv->b = cell->b;
				nextv->charge = cell->charge;
				COPY(nextv->o, cell->o);
				COPY(nextv->p, cell->p);
				COPY(nextv->s, cell->s);
				nextv->phi = cell->phi;
				nextv->code = cell->code;
				//
				cell -= SIDE3;
				nextv -= SIDE3;
			}
			//
			// Compare 'columns'
			//
			Cell* active_cell = active_stack;
			Cell *passive_cell = passive_stack;
			for (int j = 0; j < SIDE2; j++)
			{
				if (active_cell->b == passive_cell->b && ISEQUAL(active_cell->o, passive_cell->o))
				{
					if (passive_cell->code == 0)
					{
						/*
						if (passive_cell->c == ~active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = PHOTON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = GLUON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = NEUTRINO;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = Z;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == active_cell->q)
							passive_cell->code = W;
						*/
						//
						if (passive_cell->code != 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code)
					{
						passive_cell->f++;
					}
				}
				active_cell = nextV(active_cell);
				passive_cell = nextV(passive_cell);
			}
		}
	}
}
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void compare(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* active_stack, * passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically'
			//
			cell = passive_stack + (SIDE2-2)*SIDE3;
			Cell* nextv = cell + SIDE3;
			Cell temp = *nextv;
			for (int j = 0; j < SIDE2; j++)
			{
				/*
				if (j == SIDE2 - 1)
					cell = &temp;
				nextv->f = cell->f;
				nextv->b = cell->b;
				nextv->charge = cell->charge;
				COPY(nextv->o, cell->o);
				COPY(nextv->p, cell->p);
				COPY(nextv->s, cell->s);
				nextv->phi = cell->phi;
				nextv->code = cell->code;
				//
				cell -= SIDE3;
				nextv -= SIDE3;
				*/
			}
			//
			// Compare 'columns'
			//
			Cell* active_cell = active_stack;
			Cell *passive_cell = passive_stack;
			for (int j = 0; j < SIDE2; j++)
			{
				/*
				if (active_cell->b == passive_cell->b && ISEQUAL(active_cell->o, passive_cell->o))
				{
					if (passive_cell->code == 0)
					{
						if (passive_cell->c == ~active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = PHOTON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = GLUON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = NEUTRINO;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = Z;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == active_cell->q)
							passive_cell->code = W;
						//
						if (passive_cell->code != 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code)
					{
						passive_cell->f++;
					}
				}
				*/
				active_cell = nextV(active_cell);
				passive_cell = nextV(passive_cell);
			}
		}
	}
}
