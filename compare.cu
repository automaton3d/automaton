#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void compare(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* active_stack, * passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically'
			//
			Cell temp = *cell;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(cell);
				if (j == SIDE2 - 1)
					next = &temp;
				cell->f = next->f;
				cell->b = next->b;
				cell->charge = next->charge;
				COPY(cell->o, next->o);
				COPY(cell->p, next->p);
				COPY(cell->s, next->s);
				cell->phi = next->phi;
				cell->code = next->code;
				cell = nextV(cell);
			}
			//
			// Compare 'columns'
			//
			Cell* active_cell = active_stack;
			Cell *passive_cell = passive_stack;
			for (int j = 0; j < SIDE2; j++)
			{
				/*
				if (active_cell->b == passive_cell->b && ISEQUAL(active_cell->o, passive_cell->o))
				{
					if (passive_cell->code == 0)
					{
						if (passive_cell->c == ~active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = PHOTON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == ~active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = GLUON;
						else if (passive_cell->c == active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = NEUTRINO;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == ~active_cell->q)
							passive_cell->code = Z;
						else if (passive_cell->c == ~active_cell->c && passive_cell->w == active_cell->w &&
							passive_cell->q == active_cell->q)
							passive_cell->code = W;
						//
						if (passive_cell->code != 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code)
					{
						passive_cell->f++;
					}
				}
				*/
		//		active_cell = nextV(active_cell);
			//	passive_cell = nextV(passive_cell);
			}
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	}
}
