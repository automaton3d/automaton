#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Compares two columns to update variables b, f and code.
 */
__global__ void compare(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		//
		// Not the last tick?
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		Cell* ptr1, * ptr2;
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically' the passive column
			//
			ptr2 = draft;
			Cell temp = *ptr2;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(ptr2);
				if (j == SIDE2 - 1)
					next = &temp;
				ptr2->f = next->f;
				ptr2->a = next->a;
				ptr2->chrg = next->chrg;
				ptr2->code = next->code;
				//
				// Next pointer value
				//
				ptr2 = next;
			}
			//
			// Compare 'columns'
			//
			ptr1 = stable;
			ptr2 = draft;
			for (int j = 0; j < SIDE2; j++)
			{
				assert(stable->u == draft->u);
				assert(stable->v == draft->v);
				// Same sector?
				//
				if (((ptr1->chrg ^ ptr2->chrg) & D_MASK) == 0)
				{
					// Do they have same affinity?
					//
					if (ptr1->a == ptr2->a)
					{
						if (ptr1->code == COLLAPSE)
						{
							ptr2->code = 0;
							ptr2->f = 1;
							ptr2->a = ptr2->floor;
						}
						//
						// Are bubbles superposing
						//
						else if(ISEQUAL(ptr1->o, ptr2->o))
						{
							// Virgin?
							//
							if (ptr2->code == 0)
							{
								// Pair formation
								//
								ptr2->code = 
									((ptr2->chrg & C_MASK) ^ (ptr1->chrg & C_MASK)) |
									((ptr2->chrg & W_MASK) ^ (ptr1->chrg & W_MASK)) |
									((ptr2->chrg & Q_MASK) ^ (ptr1->chrg & Q_MASK));
								//
								// Affinity is the same now
								//
								ptr2->a = ptr1->a;
								//
								// Adjust frequency
								//
								ptr2->f++;
							}
						}
					}
				}
				//
				// Next pointer values
				//
				ptr1 = nextV(ptr1);
				ptr2 = nextV(ptr2);
			}
		}
	}
}
