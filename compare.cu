#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Compares two columns to update variables b, f and code.
 */
__global__ void compare(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Not the last tick?
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		Cell* ptr1, * ptr2;
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically' the passive column
			//
			ptr2 = draft;
			Cell temp = *ptr2;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(ptr2);
				if (j == SIDE2 - 1)
					next = &temp;
				ptr2->f = next->f;
				ptr2->a = next->a;
				ptr2->charge = next->charge;
				ptr2->code = next->code;
				//
				// Next pointer value
				//
				ptr2 = next;
			}
			//
			// Compare 'columns'
			//
			ptr1 = stable;
			ptr2 = draft;
			for (int j = 0; j < SIDE2; j++)
			{
				assert(stable->u == draft->u);
				assert(stable->v == draft->v);
				// Same sector?
				//
				if (((ptr1->charge ^ ptr2->charge) & D_MASK) == 0)
				{
					// Do they have same affinity?
					//
					if (ptr1->a == ptr2->a)
					{
						if (ptr1->code == COLLAPSE)
						{
							ptr2->code = 0;
							ptr2->f = 1;
							ptr2->a = ptr2->floor;
						}
						//
						// Are bubbles superposing
						//
						else if(ISEQUAL(ptr1->o, ptr2->o))
						{
							// Virgin?
							//
							if (ptr2->code == 0)
							{
								// Pair formation
								//
								unsigned char cc =
									(ptr2->charge & C_MASK) ^ (ptr1->charge & C_MASK);
								unsigned char ww =
									(ptr2->charge & W_MASK) ^ (ptr1->charge & W_MASK);
								unsigned char qq =
									(ptr2->charge & Q_MASK) ^ (ptr1->charge & Q_MASK);
								//
								if (cc == 0 && ww == 0 && qq == Q_MASK)
								{
									ptr2->a = ptr1->a;
									ptr2->code = NEUTRINO;
									ptr2->f++;
								}
								else if (cc == 0 && ww == W_MASK && qq == Q_MASK)
								{
									ptr2->a = ptr1->a;
									ptr2->code = GLUON;
									ptr2->f++;
								}
								else if (cc == C_MASK && ww == 0 && qq == 0)
								{
									ptr2->a = ptr1->a;
									ptr2->code = W;
									ptr2->f++;
								}
								else if (cc == C_MASK && ww == 0 && qq == Q_MASK)
								{
									ptr2->a = ptr1->a;
									ptr2->code = Z;
									ptr2->f++;
								}
								else if (cc == C_MASK && ww == W_MASK && qq == Q_MASK)
								{
									ptr2->a = ptr1->a;
									ptr2->code = PHOTON;
									ptr2->f++;
								}
							}
						}
					}
				}
				//
				// Next pointer values
				//
				ptr1 = nextV(ptr1);
				ptr2 = nextV(ptr2);
			}
		}
	}
}
