#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

/*
 * Compares two columns to update variables f and code.
 */
__global__ void compare(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Not the last tick?
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		#define COMPARE
		#if defined(COMPARE)
		Cell* active_cell, * passive_cell;
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically' the passive column
			//
			passive_cell = draft;
			Cell temp = *passive_cell;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(passive_cell);
				if (j == SIDE2 - 1)
					next = &temp;
				passive_cell->f = next->f;
				passive_cell->b = next->b;
				passive_cell->charge = next->charge;
				COPY(passive_cell->o, next->o);
				COPY(passive_cell->p, next->p);
				COPY(passive_cell->s, next->s);
				passive_cell->phi = next->phi;
				passive_cell->code = next->code;
				//
				// Next pointer value
				//
				passive_cell = next;
			}
			//
			// Compare 'columns'
			//
			active_cell = stable;
			passive_cell = draft;
			for (int j = 0; j < SIDE2; j++)
			{
				// Test if the bubbles are superposing
				//
				if (active_cell->b == passive_cell->b &&
					ISEQUAL(active_cell->o, passive_cell->o))
				{
					// Virgin?
					//
					if (passive_cell->code == 0)
					{
						unsigned char cc = 
							(passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK);
						unsigned char ww = 
							(passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK);
						unsigned char qq = 
							(passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK);
						//
						if (cc == 0 && ww == 0 && qq == Q_MASK)
							passive_cell->code = NEUTRINO;
						else if (cc == 0 && ww == W_MASK && qq == Q_MASK)
							passive_cell->code = GLUON;
						else if (cc == C_MASK && ww == 0 && qq == 0)
							passive_cell->code = W;
						else if (cc == C_MASK && ww == 0 && qq == Q_MASK)
							passive_cell->code = Z;
						else if (cc == C_MASK && ww == W_MASK && qq == Q_MASK)
							passive_cell->code = PHOTON;
						//
						if (passive_cell->code != 0 && passive_cell->f > 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code && 
						passive_cell->f > 0)
					{
						passive_cell->f++;
					}
				}
				//
				// Next pointer values
				//
				active_cell = nextV(active_cell);
				passive_cell = nextV(passive_cell);
			}
		}
		#endif
	}
}
