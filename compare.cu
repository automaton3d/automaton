#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "automaton.h"

/*
 * Compares two columns to update variables f and code.
 */
__global__ void compare(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Not the last tick?
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		#define COMPARE
		#if defined(COMPARE)
		Cell* ptr1, * ptr2;
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically' the passive column
			//
			ptr2 = draft;
			Cell temp = *ptr2;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(ptr2);
				if (j == SIDE2 - 1)
					next = &temp;
				ptr2->f = next->f;
				ptr2->b = next->b;
				ptr2->charge = next->charge;
				COPY(ptr2->o, next->o);
				COPY(ptr2->p, next->p);
				COPY(ptr2->s, next->s);
				ptr2->phi = next->phi;
				ptr2->code = next->code;
				//
				// Next pointer value
				//
				ptr2 = next;
			}
			//
			// Compare 'columns'
			//
			ptr1 = stable;
			ptr2 = draft;
			for (int j = 0; j < SIDE2; j++)
			{
				// Same sector?
				//
				if (((ptr1->charge ^ ptr2->charge) & C_MASK) == 0)
				{
					// Update frequency
					//
					if (ptr1->f > 0 && ptr2->f > 0 && ptr1->b == ptr2->b)
						ptr2->f++;
					//
					// Test if the bubbles are superposing
					//
					if (ptr1->b == ptr2->b &&
						ISEQUAL(ptr1->o, ptr2->o))
					{
						// Virgin?
						//
						if (ptr2->code == 0)
						{
							unsigned char cc =
								(ptr2->charge & C_MASK) ^ (ptr1->charge & C_MASK);
							unsigned char ww =
								(ptr2->charge & W_MASK) ^ (ptr1->charge & W_MASK);
							unsigned char qq =
								(ptr2->charge & Q_MASK) ^ (ptr1->charge & Q_MASK);
							//
							if (cc == 0 && ww == 0 && qq == Q_MASK)
								ptr2->code = NEUTRINO;
							else if (cc == 0 && ww == W_MASK && qq == Q_MASK)
								ptr2->code = GLUON;
							else if (cc == C_MASK && ww == 0 && qq == 0)
								ptr2->code = W;
							else if (cc == C_MASK && ww == 0 && qq == Q_MASK)
								ptr2->code = Z;
							else if (cc == C_MASK && ww == W_MASK && qq == Q_MASK)
								ptr2->code = PHOTON;
						}
					}
				}
				//
				// Next pointer values
				//
				ptr1 = nextV(ptr1);
				ptr2 = nextV(ptr2);
			}
		}
		#endif
	}
}
