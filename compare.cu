#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void compare(Cell* lattice)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* active_stack, * passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		if (passive_stack->t % LIGHT != 0)
		{
			return;
		}
		#ifdef COMPARE
		for (int i = 0; i < SIDE2; i++)
		{
			// Shift 'vertically'
			//
			Cell temp = *cell;
			for (int j = 0; j < SIDE2; j++)
			{
				Cell* next = nextV(cell);
				if (j == SIDE2 - 1)
					next = &temp;
				cell->f = next->f;
				cell->b = next->b;
				cell->charge = next->charge;
				COPY(cell->o, next->o);
				COPY(cell->p, next->p);
				COPY(cell->s, next->s);
				cell->phi = next->phi;
				cell->code = next->code;
				cell = nextV(cell);
			}
			//
			// Compare 'columns'
			//
			Cell* active_cell = active_stack;
			Cell *passive_cell = passive_stack;
			for (int j = 0; j < SIDE2; j++)
			{
				if (active_cell->b == passive_cell->b && ISEQUAL(active_cell->o, passive_cell->o))
				{
					if (passive_cell->code == 0)
					{
						if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == C_MASK && 
							  ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == W_MASK &&
							  ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == Q_MASK)
							passive_cell->code = PHOTON;
						else if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == 0 &&
							    ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == W_MASK &&
							    ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == Q_MASK)
							passive_cell->code = PHOTON;
						else if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == 0 &&
							     ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == 0 &&
							     ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == Q_MASK)
							passive_cell->code = PHOTON;
						else if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == 0 &&
							     ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == 0 &&
							     ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == Q_MASK)
							passive_cell->code = PHOTON;
						else if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == C_MASK &&
							     ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == 0 &&
							     ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == Q_MASK)
							passive_cell->code = PHOTON;
						else if (((passive_cell->charge & C_MASK) ^ (active_cell->charge & C_MASK)) == C_MASK &&
							     ((passive_cell->charge & W_MASK) ^ (active_cell->charge & W_MASK)) == 0 &&
							     ((passive_cell->charge & Q_MASK) ^ (active_cell->charge & Q_MASK)) == 0)
							passive_cell->code = PHOTON;
						//
						if (passive_cell->code != 0)
							passive_cell->f++;
					}
					else if (passive_cell->code == active_cell->code)
					{
						passive_cell->f++;
					}
				}
				active_cell = nextV(active_cell);
				passive_cell = nextV(passive_cell);
			}
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	#endif
	}
}
