#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Reverses roles. 
 */
__global__ void commute(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		//
		// Scan an entire column
		//
		for (int v = 0; v < SIDE2; v++)
		{
			// On the last tick, disassemble all pairs
			//
			if (draft->t % LIGHT == 0 && draft->f > 0)
				draft->f = 1;
			//
			// Copy all variables
			//
			stable->t = draft->t;
			stable->dir = draft->dir;
			stable->charge = draft->charge;
			stable->code = draft->code;
			stable->noise = draft->noise;
			stable->a = draft->a;
			stable->sync = draft->sync;
			stable->f = draft->f;
			stable->flash = draft->flash;
			stable->u = draft->u;
			stable->v = draft->v;
			COPY(stable->p, draft->p);
			COPY(stable->s, draft->s);
			COPY(stable->o, draft->o);
			//
			// Next register
			//
			draft = nextV(draft);
			stable = nextV(stable);
		}
	}
}

