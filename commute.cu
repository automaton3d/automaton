#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void commute(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* passive_stack, * active_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int v = 0; v < SIDE2; v++)
		{
			active_stack->active = false;
			passive_stack->active = true;
			active_stack->charge = passive_stack->charge;
			COPY(active_stack->o, passive_stack->o);
			COPY(active_stack->p, passive_stack->p);
			COPY(active_stack->s, passive_stack->s);
			COPY(active_stack->pole, passive_stack->pole);
			active_stack->code = 0;
			active_stack->noise ^= passive_stack->noise;
			active_stack->f = passive_stack->f;
			active_stack->b = passive_stack->b;
			if (ISNULL(passive_stack->o) && !ISNULL(passive_stack->p))
			{
				active_stack->f = 1;
				active_stack->b = 0;
			}
			passive_stack = nextV(passive_stack);
			active_stack = nextV(active_stack);
		}
	}
}

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void commute(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell* cell = lattice + id;
		Cell* passive_stack, * active_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		for (int v = 0; v < SIDE2; v++)
		{
			active_stack->active = false;
			passive_stack->active = true;
			active_stack->charge = passive_stack->charge;
			COPY(active_stack->o, passive_stack->o);
			COPY(active_stack->p, passive_stack->p);
			COPY(active_stack->s, passive_stack->s);
			COPY(active_stack->pole, passive_stack->pole);
			active_stack->code = 0;
			active_stack->noise ^= passive_stack->noise;
			active_stack->f = passive_stack->f;
			active_stack->b = passive_stack->b;
			active_stack->synch = passive_stack->synch;
			if (ISNULL(passive_stack->o) && !ISNULL(passive_stack->p))
			{
				active_stack->f = 1;
				active_stack->b = 0;
			}
			passive_stack = nextV(passive_stack);
			active_stack = nextV(active_stack);
		}
	}
}

