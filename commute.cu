#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.h"

__global__ void commute(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		for (int v = 0; v < SIDE2; v++)
		{
			// On the last tick, disassemble all pairs
			//
			if (draft->t % LIGHT == 0 && draft->f > 0)
				draft->f = 1;
			//
			// Copy all variables
			//
			stable->t = draft->t;
			stable->dir = draft->dir;
			stable->charge = draft->charge;
			stable->code = draft->code;
			stable->noise = draft->noise;
			stable->b = draft->b;
			stable->synch = draft->synch;
			stable->f = draft->f;
			COPY(stable->p, draft->p);
			COPY(stable->s, draft->s);
			COPY(stable->o, draft->o);
			stable->flash = draft->flash;
			//
			// Commute roles
			//
			stable->active = false;
			draft->active = true;
			//
			// Next register
			//
			draft = nextV(draft);
			stable = nextV(stable);
		}
	}
}

