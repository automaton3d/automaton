#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Reverses roles. 
 */
__global__ void commute(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		//
		// Scan an entire column
		//
		for (int v = 0; v < SIDE2; v++)
		{
			// On the last tick, disassemble all pairs
			//
			if (draft->t % LIGHT == 0 && draft->f > 0)
				draft->f = 1;
			//
			// Copy all variables
			//
			stable->t = draft->t;
			stable->dir = draft->dir;
			stable->charge = draft->charge;
			stable->code = draft->code;
			stable->noise = draft->noise;
			stable->a = draft->a;
			stable->sync = draft->sync;
			stable->f = draft->f;
			stable->flash = draft->flash;
			stable->u = draft->u;
			stable->v = draft->v;
			COPY(stable->p, draft->p);
			COPY(stable->s, draft->s);
			COPY(stable->o, draft->o);

			if (v == 173 && stable->t > 61 && stable->f > 0 && MOD2(stable->o) < 2)
				printf("mod2(o)=%d\n", stable->t);
			//
			// Next register
			//
			draft = nextV(draft);
			stable = nextV(stable);
		}
	}
}

