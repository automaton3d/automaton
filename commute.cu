#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.h"

__global__ void commute(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		for (int v = 0; v < SIDE2; v++)
		{
			// Copy all variables
			//
			stable->t = draft->t;
			stable->dir = draft->dir;
			stable->charge = draft->charge;
			stable->code = draft->code;
			stable->noise = draft->noise;
			stable->b = draft->b;
			stable->synch = draft->synch;
			stable->f = draft->f;
			COPY(stable->p, draft->p);
			COPY(stable->s, draft->s);
			COPY(stable->o, draft->o);
			COPY(stable->pole, draft->pole);
			//
			// Commute roles
			//
			stable->active = false;
			draft->active = true;
			//
			// Next register
			//
			draft = nextV(draft);
			stable = nextV(stable);
		}
	}
}

