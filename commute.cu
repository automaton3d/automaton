#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Reverses roles. 
 */
__global__ void commute(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		if (draft->active)
		{
			Cell* temp = draft;
			draft = stable;
			stable = temp;
		}
		//
		// Scan an entire column
		//
		for (int v = 0; v < SIDE2; v++)
		{
			// On the last tick, disassemble all pairs
			//
			if (draft->t % LIGHT == 0 && draft->f > 0)
				draft->f = 1;
			//
			// Copy all variables
			//
			stable->t = draft->t;
			stable->dir = draft->dir;
			stable->charge = draft->charge;
			stable->code = draft->code;
			stable->noise = draft->noise;
			stable->b = draft->b;
			stable->synch = draft->synch;
			stable->f = draft->f;
			COPY(stable->p, draft->p);
			COPY(stable->s, draft->s);
			COPY(stable->o, draft->o);
			stable->flash = draft->flash;



			if (v == 173 && stable->t > 61 && stable->f > 0 && MOD2(stable->o) < 2)
				printf("mod2(o)=%d\n", stable->t);
			//
			// Commute roles
			//
			stable->active = false;
			draft->active = true;
			//
			// Next register
			//
			draft = nextV(draft);
			stable = nextV(stable);
		}
	}
}

