#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void commute(struct Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		struct Cell* cell = lattice + id;
		if (!cell->active)
			cell = cell->h;
		//
		for (int v = 0; v < SIDE2; v++)
		{
			cell->active = false;
			cell->h->active = true;
			cell->h->q = cell->q;
			cell->h->w = cell->w;
			cell->h->c = cell->c;
			cell->h->d = cell->d;
			COPY(cell->h->o, cell->o);
			COPY(cell->h->p, cell->p);
			COPY(cell->h->s, cell->s);
			COPY(cell->h->pole, cell->pole);
			cell->h->code = 0;
			cell->h->noise ^= cell->noise;
			cell->h->f = cell->f;
			cell->h->b = cell->b;
			if (ISNULL(cell->o) && !ISNULL(cell->p))
			{
				cell->h->f = 1;
				cell->h->b = 0;
			}
			cell = cell->v;
		}
	}
}

