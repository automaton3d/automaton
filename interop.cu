#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "automaton.cuh"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "cglm/vec3.h"

__global__ void interop(Cell* lattice, vec3 *dev_color, int floor)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		hiprandState state;
		hiprand_init(0, id, 0, &state);
		bool p = false, flash = false;
		Cell* cell = lattice + id;
		//
		// Calculate voxel color
		//
		if (floor < 0)
		{
			for (int i = 0; i < SIDE2; i++)
			{
				int rnd = hiprand(&state)& (SIDE2 - 1);
				if (rnd < SIDE/4)
				{
					if (!ISNULL(cell->p))
					{
						floor = i;
						p = true;
						break;
					}
					else if (cell->f > 0)
					{
						floor = i;
						flash = true;
					}
				}
				cell = nextV(cell);
			}
		}
		else
		{
			for (int i = 0; i < floor; i++)
				cell = nextV(cell);
			//
			if (!ISNULL(cell->p))
				p = true;
			else if (cell->flash)
				flash = true;
		}
		//
		// Update voxel color
		//
		float* ptr = (float*)(dev_color + id);
		if (p)
		{
			*ptr++ = 1;
			*ptr++ = 0;
			*ptr = 0;
		}
		else if(flash)
		{
			*ptr++ = 0;
			*ptr++ = 1;
			*ptr = 0;
		}
		else
		{
			*ptr++ = 0.6;
			*ptr++ = 0.6;
			*ptr = 0.8;
		}
	}
}

