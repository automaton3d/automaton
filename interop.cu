#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "automaton.h"
#include "cglm/vec3.h"

__global__ void interop(Cell* lattice, vec3 *dev_color, int all)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		bool p = false, f = false;
		Cell* cell = lattice + id;
		//
		// Calculate voxel color
		//
		if (all)
		{
			for (int i = 0; i < SIDE2; i++)
			{
				if (!ISNULL(cell->p))
				{
					p = true;
					break;
				}
				else if (cell->f > 0)
				{
					f = true;
				}
				cell = nextV(cell);
			}
		}
		else
		{
			for (int i = 0; i < 150; i++)
				cell = nextV(cell);
			//
			if (!ISNULL(cell->p))
				p = true;
			else if (cell->f > 0)
				f = true;
		}

		//
		// Update voxel color
		//
		float* ptr = (float*)(dev_color + id);
		if (p)
		{
			*ptr++ = 1;
			*ptr++ = 0;
			*ptr++ = 0;
		}
		else if(f)
		{
			*ptr++ = 0;
			*ptr++ = 0;
			*ptr++ = 1;
		}
		else
		{
			*ptr++ = 0.6;
			*ptr++ = 0.6;
			*ptr++ = 0.6;
		}
	}
}

