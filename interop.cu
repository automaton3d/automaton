#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "automaton.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "cglm/vec3.h"

__global__ void interop(Cell* lattice, vec3 *dev_color, int floor)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		hiprandState state;
		hiprand_init(0, id, 0, &state);
		bool p = false, f = false;
		Cell* cell = lattice + id;
		//
		// Calculate voxel color
		//
		if (floor < 0)
		{
			for (int i = 0; i < SIDE2; i++)
			{
				int rnd = hiprand(&state)& (SIDE2 - 1);
				if (rnd < SIDE/4)
				{
					if (!ISNULL(cell->p))
					{
						floor = i;
						p = true;
						break;
					}
					else if (cell->f > 0)
					{
						floor = i;
						f = true;
					}
				}
				cell = nextV(cell);
			}
		}
		else
		{
			for (int i = 0; i < floor; i++)
				cell = nextV(cell);
			//
			if (!ISNULL(cell->p))
				p = true;
			else if (cell->f > 0)
				f = true;
		}
		//
		// Update voxel color
		//
		float* ptr = (float*)(dev_color + id);
		if (p)
		{
			*ptr++ = 1;
			*ptr++ = 0;
			*ptr = 0;
		}
		else if(f)
		{
			/*
			switch (floor % 3)
			{
				case 0:
					*ptr++ = 0;
					*ptr++ = 1;
					*ptr = 0;
					break;
				case 1:
					*ptr++ = 0;
					*ptr++ = 0;
					*ptr = 1;
					break;
				case 2:
					*ptr++ = 1;
					*ptr++ = 1;
					*ptr = 0;
					break;
			}
			*/
			*ptr++ = (MOD2(cell->o) & (SIDE-1))/ (float)SIDE;
			*ptr++ = ((MOD2(cell->o) >> 8) & (SIDE - 1)) / (float)SIDE;
			*ptr = ((MOD2(cell->o) >> 16) & (SIDE - 1)) / (float)SIDE;
		}
		else
		{
			*ptr++ = 0.6;
			*ptr++ = 0.6;
			*ptr = 0.8;
		}
	}
}

