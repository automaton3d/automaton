#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "automaton.h"
#include "cglm/vec3.h"

__global__ void interop(struct Cell* lattice, vec3 *dev_color)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < SIDE3)
	{
		struct Cell* cell = lattice + id;
		if (cell->active)
		{
			cell = cell->h;
		}
		bool p = false, f = false;
		//
		// Calculate voxel color
		//



		cell = cell->v;
		cell = cell->v;
		cell = cell->v;
		cell = cell->v;
		cell = cell->v;

		if (!ISNULL(cell->p))
		{
			p = true;
		}
		else if (cell->f > 0)
		{
			f = true;
		}

		/*
		for (int i = 0; i < SIDE2; i++)
		{
			if (!ISNULL(cell->p))
			{
				p = true;
				break;
			}
			else if (cell->f > 0)
			{
				f = true;
			}
			cell = cell->v;
		}
		*/
		//
		// Update voxel color
		//
		float* ptr = (float*)(dev_color + id);
		if (p)
		{
			*ptr++ = 1;
			*ptr++ = 0;
			*ptr++ = 0;
		}
		else if(f)
		{
			*ptr++ = 0;
			*ptr++ = 0;
			*ptr++ = 1;
		}
		else
		{
			*ptr++ = 0;
			*ptr++ = 1;
			*ptr++ = 0;
		}
	}
}

