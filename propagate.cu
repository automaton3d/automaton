#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__device__ int signum(int x)
{
    if (x > 0) return 1;
    if (x < 0) return -1;
    return 0;
}

__device__ void calculate(struct Cell* cell)
{
    if (cell->f > 0 || !ISNULL(cell->p))
    {
        if (cell->ctrl > 0)
        {
            // Track decay
            //
            cell->phi *= (1 - 1 / (2 * cell->t));
            //
            // Minsky circle algorithm
            //
            int xNew = cell->cosine - (cell->sine >> SHIFT);
            int yNew = cell->sine + (cell->cosine >> SHIFT);
            cell->cosine = xNew;
            cell->sine = yNew;
            //
            cell->ctrl--;
        }
//        if ((cell->t*cell->t > cell->synch || ISNULL(cell->o)) && !ISNULL(cell->pole))
//        {
            int o2 = cell->o[0] * cell->o[0] + cell->o[1] * cell->o[1] + cell->o[2] * cell->o[2];
            for (int dir = 0; dir < 6; dir++)
            {
                // von Neumann
                //
                int a = 0, b = 0, c = 0;
                struct Cell* neighbor;
                switch (dir)
                {
                case 0: a = +1; neighbor = cell->px; break;
                case 1: a = -1; neighbor = cell->nx; break;
                case 2:	b = +1; neighbor = cell->py; break;
                case 3: b = -1; neighbor = cell->nx; break;
                case 4: c = +1; neighbor = cell->pz; break;
                case 5: c = -1; neighbor = cell->nz; break;
                }
                int mod2 = 
                    (cell->o[0] + a) * (cell->o[0] + a) + 
                    (cell->o[1] + b) * (cell->o[1] + b) + 
                    (cell->o[2] + c) * (cell->o[2] + c);
                if (mod2 >= o2)
                {
                    neighbor->active = false;
                    neighbor->f = cell->f;
                    neighbor->b = cell->b;
                    neighbor->q = cell->q;
                    neighbor->w = cell->w;
                    neighbor->c = cell->c;
                    neighbor->d = cell->d;
                    //
                    neighbor->o[0] = cell->o[0] + a;
                    neighbor->o[1] = cell->o[1] + b;
                    neighbor->o[2] = cell->o[2] + c;
                    //
//                    RESET(neighbor->p);
                    COPY(neighbor->p, cell->p);
                    COPY(neighbor->s, cell->s);
                    //
                    neighbor->synch = LIGHT2 * mod2;
                    //

                    /*
                    if (!ISNULL(cell->p))
                    {
                        bool found = 0;
                        if (abs(cell->p[0]) > abs(cell->p[1]))
                        {
                            if (abs(cell->p[0]) > abs(cell->p[2]))
                            {
                                if (a == signum(cell->p[0])) found = true;
                            }
                            else
                            {
                                if (c == signum(cell->p[2])) found = true;
                            }
                        }
                        else if (abs(cell->p[0]) < abs(cell->p[1]))
                        {
                            if (abs(cell->p[0]) > abs(cell->p[2]))
                            {
                                if (b == signum(cell->p[1])) found = true;
                            }
                            else
                            {
                                if (c == signum(cell->p[2])) found = true;
                            }
                        }
                        else
                        {
                            if (abs(cell->p[0]) > abs(cell->p[2]))
                            {
                                if (a == signum(cell->p[0])) found = true;
                            }
                            else
                            {
                                if (c == signum(cell->p[2])) found = true;
                            }
                        }
                        if (found)
                        {
                            neighbor->pole[0] = cell->pole[0] - a;
                            neighbor->pole[1] = cell->pole[1] - b;
                            neighbor->pole[2] = cell->pole[2] - c;
                            COPY(neighbor->p, cell->p);
                        }
                    }
                    */
                }
  //          }
            //
            // Erase origin cell
            //
            cell->f = 0;
            RESET(cell->p);
        }
    }
    cell->t++;
}

__global__ void expand(struct Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < SIDE2)
    {
        for (int step = 0; step < LIGHT; step++)
        {
            struct Cell* cell = lattice + idx * (long long)SIDE3;
            if (cell->active)
                cell = cell->h;
            for (int z = 0; z < SIDE; z++)
            {
                for (int y = 0; y < SIDE; y++)
                {
                    for (int x = 0; x < SIDE; x++)
                        calculate(cell++);
                }
            }
        }
    }
}
