#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

#define S   (SIDE/2)

/*
 * Tests whether the direction dir is a valid path in the visit-once-tree.
 */
__device__ bool isAllowed(int dir, char vdir[3], char o[3], unsigned char d0)
{
    // Calculate new origin vector
    //
    int d1 = o[0] * o[0] + o[1] * o[1] + o[2] * o[2];
    int x = o[0] + vdir[0];
    int y = o[1] + vdir[1];
    int z = o[2] + vdir[2];
    //
    // Test for expansion
    //
    int d2 = x * x + y * y + z * z;
    if (d2 <= d1)
        return false;
    //
    // Wrapping test
    //
    if (x == S + 1 || x == -S || y == S + 1 || y == -S || z == S + 1 || z == -S)
        return false;
    //
    // Root allows all six directions
    //
    int level = abs(x) + abs(y) + abs(z);
    if (level == 1)
        return true;
    //
    // x axis
    //
    if (x > 0 && y == 0 && z == 0 && dir == 0)
        return true;
    else if (x < 0 && y == 0 && z == 0 && dir == 1)
        return true;
    //
    // y axis
    //
    else if (x == 0 && y > 0 && z == 0 && dir == 2)
        return true;
    else if (x == 0 && y < 0 && z == 0 && dir == 3)
        return true;
    //
    // z axis
    //
    else if (x == 0 && y == 0 && z > 0 && dir == 4)
        return true;
    else if (x == 0 && y == 0 && z < 0 && dir == 5)
        return true;
    //
    // xy plane
    //
    else if (x > 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 2);
        else
            return (dir == 2 && d0 == 0);
    }
    else if (x < 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 2);
        else
            return (dir == 2 && d0 == 1);
    }
    else if (x > 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 3);
        else
            return (dir == 3 && d0 == 0);
    }
    else if (x < 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 3);
        else
            return (dir == 3 && d0 == 1);
    }
    //
    // yz plane
    //
    else if (x == 0 && y > 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 2);
        else
            return (dir == 2 && d0 == 4);
    }
    else if (x == 0 && y < 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 3);
        else
            return (dir == 3 && d0 == 4);
    }
    else if (x == 0 && y > 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 2);
        else
            return (dir == 2 && d0 == 5);
    }
    else if (x == 0 && y < 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 3);
        else
            return (dir == 3 && d0 == 5);
    }
    //
    // zx plane
    //
    else if (x > 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 0);
        else
            return (dir == 0 && d0 == 4);
    }
    else if (x < 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 1);
        else
            return (dir == 1 && d0 == 4);
    }
    else if (x > 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 0);
        else
            return (dir == 0 && d0 == 5);
    }
    else if (x < 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 1);
        else
            return (dir == 1 && d0 == 5);
    }
    else
    {
        // Spirals
        //
        int x0 = x + S;
        int y0 = y + S;
        int z0 = z + S;
        //
        switch (level % 3)
        {
        case 0:
            if (x0 != S && y0 != S)
                return (z0 > S && dir == 4) || (z0 < S && dir == 5);
            break;
        case 1:
            if (y0 != S && z0 != S)
                return (x0 > S && dir == 0) || (x0 < S && dir == 1);
            break;
        case 2:
            if (x0 != S && z0 != S)
                return (y0 > S && dir == 2) || (y0 < S && dir == 3);
            break;
        }
    }
    return false;
}

__device__ void branch(Cell* active_cell, Cell* passive_cell)
{
    if (passive_cell->ctrl > 0)
    {
        // Track decay
        //
        passive_cell->phi *= (1 - 1 / (2 * passive_cell->t));
        //
        // Minsky circle algorithm
        //
        int xNew = passive_cell->cosine - (passive_cell->sine >> SHIFT);
        int yNew = passive_cell->sine + (passive_cell->cosine >> SHIFT);
        passive_cell->cosine = xNew;
        passive_cell->sine = yNew;
        //
        passive_cell->ctrl--;
    }
    if (active_cell->f > 0 || !ISNULL(active_cell->p))
    {
        Cell* neighbor;
        if (active_cell->t * active_cell->t > active_cell->synch)
        {
            // Explore von Neumann directions
            //
            Cell* neighbor;
            for (int dir = 0; dir < 6; dir++)
            {
                char vdir[3] = { 0, 0, 0 };
                switch (dir)
                {
                    case 0:
                        vdir[0] = +1;
                        if (active_cell->type & 0x40)
                            neighbor = passive_cell - (SIDE - 1);
                        else
                            neighbor = passive_cell + 1;
                        break;
                    case 1:
                        vdir[0] = -1;
                        if (active_cell->type & 0x80)
                            neighbor = passive_cell + (SIDE - 1);
                        else
                            neighbor = passive_cell - 1;
                        break;
                    case 2:
                        vdir[1] = +1;
                        if (active_cell->type & 0x10)
                            neighbor = passive_cell - (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell + SIDE;
                        break;
                    case 3:
                        vdir[1] = -1;
                        if (active_cell->type & 0x20)
                            neighbor = passive_cell + (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell - SIDE;
                        break;
                    case 4:
                        vdir[2] = +1;
                        if (active_cell->type & 0x04)
                            neighbor = passive_cell - (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell + SIDE2;
                        break;
                    case 5:
                        vdir[2] = -1;
                        if (active_cell->type & 0x08)
                            neighbor = passive_cell + (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell - SIDE2;
                        break;
                }
                //
                // Test if branch is legal
                //
                if(isAllowed(dir, vdir, active_cell->o, active_cell->dir))
                {
                    neighbor->dir = dir;
                    neighbor->f = active_cell->f;
                    neighbor->b = active_cell->b;
                    neighbor->charge = active_cell->charge;
                    //
                    neighbor->o[0] = active_cell->o[0] + vdir[0];
                    neighbor->o[1] = active_cell->o[1] + vdir[1];
                    neighbor->o[2] = active_cell->o[2] + vdir[2];
                    //
                    int mod2 = neighbor->o[0] * neighbor->o[0] + neighbor->o[1] * neighbor->o[1] + neighbor->o[2] * neighbor->o[2];
                    RESET(neighbor->p);
                    COPY(neighbor->s, active_cell->s);
                    neighbor->synch = LIGHT2 * mod2;
                    //
                    if (!ISNULL(active_cell->p) && !ISNULL(active_cell->pole))
                    {
                        int d1 = active_cell->pole[0] * active_cell->pole[0] + active_cell->pole[1] * active_cell->pole[1] + active_cell->pole[2] * active_cell->pole[2];
                        neighbor->pole[0] = active_cell->pole[0] - vdir[0];
                        neighbor->pole[1] = active_cell->pole[1] - vdir[1];
                        neighbor->pole[2] = active_cell->pole[2] - vdir[2];
                        int d2 = neighbor->pole[0] * neighbor->pole[0] + neighbor->pole[1] * neighbor->pole[1] + neighbor->pole[2] * neighbor->pole[2];
                        if (d2 < d1)
                        {
                            COPY(neighbor->p, active_cell->p);
                        }
                    }
                }
            }
            //
            passive_cell->f = 0;
            RESET(passive_cell->p);
        }
    }
    active_cell->t++;
    passive_cell->t++;
}

__global__ void expand(Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < SIDE2)
    {
        lattice->b = 0;


        for (int step = 0; step < LIGHT; step++)
        {
            Cell* cell = lattice + idx * (long long)SIDE3;
            Cell* active_cube, * passive_cube;
            if (cell->active)
            {
                active_cube = cell;
                passive_cube = cell + SIDE3 * SIDE2;
            }
            else
            {
                passive_cube = cell;
                active_cube = cell + SIDE3 * SIDE2;
            }
            //
            for (int z = 0; z < SIDE; z++)
            {
                for (int y = 0; y < SIDE; y++)
                {
                    for (int x = 0; x < SIDE; x++)
                    {
                        branch(active_cube, passive_cube);
                        active_cube++;
                        passive_cube++;
                    }
                }
            }
        }
    }
}
