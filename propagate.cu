#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__device__ int signum(int x)
{
    if (x > 0) return 1;
    if (x < 0) return -1;
    return 0;
}

__device__ char dirs[6][3] = { { +1, 0, 0 }, { -1, 0, 0 }, { 0, +1, 0 }, { 0, -1, 0}, { 0, 0, +1 }, { 0, 0, -1 } };

/*
 * Tests whether the direction dir is a valid path in the visit-once-tree.
 */
__device__ bool isAllowed(int dir, char p[3], unsigned char d0)
{
    int d1 = p[0] * p[0] + p[1] * p[1] + p[2] * p[2];
    int x = p[0] + dirs[dir][0];
    int y = p[1] + dirs[dir][1];
    int z = p[2] + dirs[dir][2];
    int d2 = x * x + y * y + z * z;
    if (d2 <= d1)
        return false;
    //
    // Wrapping test
    //
    if (x == SIDE / 2 + 1 || x == -SIDE / 2 || y == SIDE / 2 + 1 || y == -SIDE / 2 || z == SIDE / 2 + 1 || z == -SIDE / 2)
        return false;
    //
    // Root
    //
    int level = abs(x) + abs(y) + abs(z);
    if (level == 1)
        return true;
    //
    // x axis
    //
    if (x > 0 && y == 0 && z == 0 && dir == 0)
        return true;
    else if (x < 0 && y == 0 && z == 0 && dir == 1)
        return true;
    //
    // y axis
    //
    else if (x == 0 && y > 0 && z == 0 && dir == 2)
        return true;
    else if (x == 0 && y < 0 && z == 0 && dir == 3)
        return true;
    //
    // z axis
    //
    else if (x == 0 && y == 0 && z > 0 && dir == 4)
        return true;
    else if (x == 0 && y == 0 && z < 0 && dir == 5)
        return true;
    //
    // xy plane
    //
    else if (x > 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 2);
        else
            return (dir == 2 && d0 == 0);
    }
    else if (x < 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 2);
        else
            return (dir == 2 && d0 == 1);
    }
    else if (x > 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 3);
        else
            return (dir == 3 && d0 == 0);
    }
    else if (x < 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 3);
        else
            return (dir == 3 && d0 == 1);
    }
    //
    // yz plane
    //
    else if (x == 0 && y > 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 2);
        else
            return (dir == 2 && d0 == 4);
    }
    else if (x == 0 && y < 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 3);
        else
            return (dir == 3 && d0 == 4);
    }
    else if (x == 0 && y > 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 2);
        else
            return (dir == 2 && d0 == 5);
    }
    else if (x == 0 && y < 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 3);
        else
            return (dir == 3 && d0 == 5);
    }
    //
    // zx plane
    //
    else if (x > 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 0);
        else
            return (dir == 0 && d0 == 4);
    }
    else if (x < 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 1);
        else
            return (dir == 1 && d0 == 4);
    }
    else if (x > 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 0);
        else
            return (dir == 0 && d0 == 5);
    }
    else if (x < 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 1);
        else
            return (dir == 1 && d0 == 5);
    }
    else
    {
        // Spirals
        //
        int x0 = x + SIDE / 2;
        int y0 = y + SIDE / 2;
        int z0 = z + SIDE / 2;
        //
        switch (level % 3)
        {
        case 0:
            if (x0 != SIDE / 2 && y0 != SIDE / 2)
                return (z0 > SIDE / 2 && dir == 4) || (z0 < SIDE / 2 && dir == 5);
            break;
        case 1:
            if (y0 != SIDE / 2 && z0 != SIDE / 2)
                return (x0 > SIDE / 2 && dir == 0) || (x0 < SIDE / 2 && dir == 1);
            break;
        case 2:
            if (x0 != SIDE / 2 && z0 != SIDE / 2)
                return (y0 > SIDE / 2 && dir == 2) || (y0 < SIDE / 2 && dir == 3);
            break;
        }
    }
    return false;
}

#define S   (SIDE/2)

__device__ void branch(Cell* active_cell, Cell* passive_cell)
{
    if (passive_cell->ctrl > 0)
    {
        // Track decay
        //
        passive_cell->phi *= (1 - 1 / (2 * passive_cell->t));
        //
        // Minsky circle algorithm
        //
        int xNew = passive_cell->cosine - (passive_cell->sine >> SHIFT);
        int yNew = passive_cell->sine + (passive_cell->cosine >> SHIFT);
        passive_cell->cosine = xNew;
        passive_cell->sine = yNew;
        //
        passive_cell->ctrl--;
    }
    if (active_cell->f > 0 || !ISNULL(active_cell->p))
    {
        if (active_cell->t * active_cell->t > active_cell->synch)
        {
            if (ISNULL(active_cell->pole) && !ISNULL(active_cell->p))
                return;
            int dx = 0, dy = 0, dz = 0;
            Cell* neighbor;
            int difs[6] = { -S, -S, -S, -S, -S, -S, };
            Cell* neighbors[6];
            for (int dir = 0; dir < 6; dir++)
            {
                switch (dir)
                {
                    case 0:
                        if (active_cell->o[0] == S)
                            continue;
                        dx = +1;
                        if (passive_cell->type & 0x40)
                            neighbor = passive_cell - (SIDE - 1);
                        else
                            neighbor = passive_cell + 1;
                        break;
                    case 1:
                        if (active_cell->o[0] == -S)
                            continue;
                        dx = -1;
                        if (passive_cell->type & 0x80)
                            neighbor = passive_cell + (SIDE - 1);
                        else
                            neighbor = passive_cell - 1;
                        break;
                    case 2:
                        if (active_cell->o[1] == S)
                            continue;
                        continue;
                        dy = +1;
                        if (passive_cell->type & 0x10)
                            neighbor = passive_cell - (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell + SIDE;
                        break;
                    case 3:
                        if (active_cell->o[1] == -S)
                            continue;
                        dy = -1;
                        if (passive_cell->type & 0x20)
                            neighbor = passive_cell + (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell - SIDE;
                        break;
                    case 4:
                        if (active_cell->o[2] == S)
                            continue;
                        dz = +1;
                        if (passive_cell->type & 0x04)
                            neighbor = passive_cell - (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell + SIDE2;
                        break;
                    case 5:
                        if (active_cell->o[2] == -S)
                            continue;
                        dz = -1;
                        if (passive_cell->type & 0x08)
                            neighbor = passive_cell + (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell - SIDE2;
                        break;
                }
                neighbors[dir] = neighbor;
                //
                // Test if neighbor is virgin
                //
//                unsigned char d0
                if(isAllowed(dir, active_cell->o, 0))
//                if (ISNULL(neighbor->p) && neighbor->f == 0)
                {
                    neighbor->d0 = dir;
                    neighbor->f = active_cell->f;
                    neighbor->b = active_cell->b;
                    neighbor->charge = active_cell->charge;
                    //
                    neighbor->o[0] = active_cell->o[0] + dx;
                    neighbor->o[1] = active_cell->o[1] + dy;
                    neighbor->o[2] = active_cell->o[2] + dz;
                    //
                    int mod2 = neighbor->o[0] * neighbor->o[0] + neighbor->o[1] * neighbor->o[1] + neighbor->o[2] * neighbor->o[2];
                    RESET(neighbor->p);
                    COPY(neighbor->s, active_cell->s);
                    neighbor->synch = LIGHT2 * mod2;
                    //
                    if (!ISNULL(active_cell->p))
                    {
                        int d1 = active_cell->pole[0] * active_cell->pole[0] + active_cell->pole[1] * active_cell->pole[1] + active_cell->pole[2] * active_cell->pole[2];
                        neighbor->pole[0] = active_cell->pole[0] - dx;
                        neighbor->pole[1] = active_cell->pole[1] - dy;
                        neighbor->pole[2] = active_cell->pole[2] - dz;
                        int d2 = neighbor->pole[0] * neighbor->pole[0] + neighbor->pole[1] * neighbor->pole[1] + neighbor->pole[2] * neighbor->pole[2];
                        difs[dir] = d1 - d2;
                    }
                }
            }
            //
            // Transfer momentum
            //
            if (!ISNULL(active_cell->p))
            {
                int max = -S;
                Cell* choice;
                for (int dir = 0; dir < 6; dir++)
                {
                    if (difs[dir] > max)
                    {
                        max = difs[dir];
                        choice = neighbors[dir];
                    }
                }
                COPY(choice->p, active_cell->p);
            }
            //
            passive_cell->f = 0;
            RESET(passive_cell->p);
        }
        active_cell->t++;
        passive_cell->t++;
    }
}

__global__ void expand(Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < SIDE2)
    {
        for (int step = 0; step < 1; step++)
        {
            Cell* cell = lattice + idx * (long long)SIDE3;
            Cell* active_cube, * passive_cube;
            if (cell->active)
            {
                active_cube = cell;
                passive_cube = cell + SIDE3 * SIDE2;
            }
            else
            {
                passive_cube = cell;
                active_cube = cell + SIDE3 * SIDE2;
            }
            //
            for (int z = 0; z < SIDE; z++)
            {
                for (int y = 0; y < SIDE; y++)
                {
                    for (int x = 0; x < SIDE; x++)
                    {
                        branch(active_cube, passive_cube);
                        active_cube++;
                        passive_cube++;
                    }
                }
            }
        }
    }
}
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <math.h>
#include "automaton.h"

#define S   (SIDE/2)

__device__ int signum(int x)
{
    if (x > 0) return 1;
    if (x < 0) return -1;
    return 0;
}

/*
 * Tests whether the direction dir is a valid path in the visit-once-tree.
 */
__device__ bool isAllowed(int dir, char vdir[3], short o[3], unsigned char d0)
{
    // Calculate new origin vector
    //
    int d1 = o[0] * o[0] + o[1] * o[1] + o[2] * o[2];
    int x = o[0] + vdir[0];
    int y = o[1] + vdir[1];
    int z = o[2] + vdir[2];
    //
    // Test for expansion
    //
    int d2 = x * x + y * y + z * z;
    if (d2 <= d1)
        return false;
    //
    // Wrapping test
    //
    if (x == S + 1 || x == -S || y == S + 1 || y == -S || z == S + 1 || z == -S)
        return false;
    //
    // Root allows all six directions
    //
    int level = abs(x) + abs(y) + abs(z);
    if (level == 1)
        return true;
    //
    // x axis
    //
    if (x > 0 && y == 0 && z == 0 && dir == 0)
        return true;
    else if (x < 0 && y == 0 && z == 0 && dir == 1)
        return true;
    //
    // y axis
    //
    else if (x == 0 && y > 0 && z == 0 && dir == 2)
        return true;
    else if (x == 0 && y < 0 && z == 0 && dir == 3)
        return true;
    //
    // z axis
    //
    else if (x == 0 && y == 0 && z > 0 && dir == 4)
        return true;
    else if (x == 0 && y == 0 && z < 0 && dir == 5)
        return true;
    //
    // xy plane
    //
    else if (x > 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 2);
        else
            return (dir == 2 && d0 == 0);
    }
    else if (x < 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 2);
        else
            return (dir == 2 && d0 == 1);
    }
    else if (x > 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 3);
        else
            return (dir == 3 && d0 == 0);
    }
    else if (x < 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 3);
        else
            return (dir == 3 && d0 == 1);
    }
    //
    // yz plane
    //
    else if (x == 0 && y > 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 2);
        else
            return (dir == 2 && d0 == 4);
    }
    else if (x == 0 && y < 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 3);
        else
            return (dir == 3 && d0 == 4);
    }
    else if (x == 0 && y > 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 2);
        else
            return (dir == 2 && d0 == 5);
    }
    else if (x == 0 && y < 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 3);
        else
            return (dir == 3 && d0 == 5);
    }
    //
    // zx plane
    //
    else if (x > 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 0);
        else
            return (dir == 0 && d0 == 4);
    }
    else if (x < 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 1);
        else
            return (dir == 1 && d0 == 4);
    }
    else if (x > 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 0);
        else
            return (dir == 0 && d0 == 5);
    }
    else if (x < 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 1);
        else
            return (dir == 1 && d0 == 5);
    }
    else
    {
        // Spirals
        //
        int x0 = x + S;
        int y0 = y + S;
        int z0 = z + S;
        //
        switch (level % 3)
        {
        case 0:
            if (x0 != S && y0 != S)
                return (z0 > S && dir == 4) || (z0 < S && dir == 5);
            break;
        case 1:
            if (y0 != S && z0 != S)
                return (x0 > S && dir == 0) || (x0 < S && dir == 1);
            break;
        case 2:
            if (x0 != S && z0 != S)
                return (y0 > S && dir == 2) || (y0 < S && dir == 3);
            break;
        }
    }
    return false;
}

__device__ void branch(Cell* active_cell, Cell* passive_cell)
{
    if (passive_cell->ctrl > 0)
    {
        // Track decay
        //
        passive_cell->phi *= (1 - 1 / (2 * passive_cell->t));
        //
        // Minsky circle algorithm
        //
        int xNew = passive_cell->cosine - (passive_cell->sine >> SHIFT);
        int yNew = passive_cell->sine + (passive_cell->cosine >> SHIFT);
        passive_cell->cosine = xNew;
        passive_cell->sine = yNew;
        //
        passive_cell->ctrl--;
    }
    if (active_cell->f > 0 || !ISNULL(active_cell->p))
    {
        Cell* neighbor;
        int difs[6] = { -S, -S, -S, -S, -S, -S, };
        Cell* neighbors[6];
        if (active_cell->t > active_cell->synch)
        {
            if (ISNULL(active_cell->pole) && !ISNULL(active_cell->p))
              return;
            //
            // Last branch
            //
            unsigned char d0 = active_cell->d0;
            //
            // Explore von Neumann directions
            //
            int difs[6] = { -S, -S, -S, -S, -S, -S, };
            Cell* neighbor;
            Cell* neighbors[6];
            for (int dir = 0; dir < 6; dir++)
            {
                char vdir[3] = { 0, 0, 0 };
                switch (dir)
                {
                    case 0:
                        vdir[0] = +1;
                        if (active_cell->type & 0x40)
                            neighbor = passive_cell - (SIDE - 1);
                        else
                            neighbor = passive_cell + 1;
                        break;
                    case 1:
                        vdir[0] = -1;
                        if (active_cell->type & 0x80)
                            neighbor = passive_cell + (SIDE - 1);
                        else
                            neighbor = passive_cell - 1;
                        break;
                    case 2:
                        vdir[1] = +1;
                        if (active_cell->type & 0x10)
                            neighbor = passive_cell - (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell + SIDE;
                        break;
                    case 3:
                        vdir[1] = -1;
                        if (active_cell->type & 0x20)
                            neighbor = passive_cell + (SIDE2 - SIDE);
                        else
                            neighbor = passive_cell - SIDE;
                        break;
                    case 4:
                        vdir[2] = +1;
                        if (active_cell->type & 0x04)
                            neighbor = passive_cell - (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell + SIDE2;
                        break;
                    case 5:
                        vdir[2] = -1;
                        if (active_cell->type & 0x08)
                            neighbor = passive_cell + (SIDE3 - SIDE2);
                        else
                            neighbor = passive_cell - SIDE2;
                        break;
                }
                neighbors[dir] = neighbor;
                //
                // Test if neighbor is virgin
                //
                if(isAllowed(dir, vdir, active_cell->o, d0))
                {
                    neighbor->d0 = dir;
                    neighbor->f = active_cell->f;
                    neighbor->b = active_cell->b;
                    neighbor->charge = active_cell->charge;
                    //
                    neighbor->o[0] = active_cell->o[0] + vdir[0];
                    neighbor->o[1] = active_cell->o[1] + vdir[1];
                    neighbor->o[2] = active_cell->o[2] + vdir[2];
                    //
                    int mod2 = neighbor->o[0] * neighbor->o[0] + neighbor->o[1] * neighbor->o[1] + neighbor->o[2] * neighbor->o[2];
                    RESET(neighbor->p);
                    COPY(neighbor->s, active_cell->s);
                    neighbor->synch = LIGHT * sqrt((float)mod2);
                    //
                    if (!ISNULL(active_cell->p))
                    {
                        int d1 = active_cell->pole[0] * active_cell->pole[0] + active_cell->pole[1] * active_cell->pole[1] + active_cell->pole[2] * active_cell->pole[2];
                        neighbor->pole[0] = active_cell->pole[0] - vdir[0];
                        neighbor->pole[1] = active_cell->pole[1] - vdir[1];
                        neighbor->pole[2] = active_cell->pole[2] - vdir[2];
                        int d2 = neighbor->pole[0] * neighbor->pole[0] + neighbor->pole[1] * neighbor->pole[1] + neighbor->pole[2] * neighbor->pole[2];
                        difs[dir] = d1 - d2;
                    }
                }
            }
            //
            // Transfer momentum
            //
            if (!ISNULL(active_cell->p))
            {
                int max = -S;
                Cell* choice;
                for (int dir = 0; dir < 6; dir++)
                {
                    if (difs[dir] > max)
                    {
                        max = difs[dir];
                        choice = neighbors[dir];
                    }
                }
                COPY(choice->p, active_cell->p);
            }
            //
            passive_cell->f = 0;
            RESET(passive_cell->p);
        }
    }
    active_cell->t++;
    passive_cell->t++;
}

__global__ void expand(Cell* lattice)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < SIDE2)
    {
        for (int step = 0; step < LIGHT; step++)
        {
            Cell* cell = lattice + idx * (long long)SIDE3;
            Cell* active_cube, * passive_cube;
            if (cell->active)
            {
                active_cube = cell;
                passive_cube = cell + SIDE3 * SIDE2;
            }
            else
            {
                passive_cube = cell;
                active_cube = cell + SIDE3 * SIDE2;
            }
            //
            for (int z = 0; z < SIDE; z++)
            {
                for (int y = 0; y < SIDE; y++)
                {
                    for (int x = 0; x < SIDE; x++)
                    {
                        branch(active_cube, passive_cube);
                        active_cube++;
                        passive_cube++;
                    }
                }
            }
        }
    }
}
