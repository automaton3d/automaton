#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void replicate(struct Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		struct Cell* cell = lattice + id;
		if (!cell->active)
			cell = cell->h;
		for (int v = 0; v < SIDE2; v++)
		{
			cell->f = cell->h->f;
			cell->b = cell->h->b;
			COPY(cell->pole, cell->h->pole);
			cell->code = cell->h->code;
			cell->noise |= cell->h->noise;
			cell = cell->v;
		}
	}
}
