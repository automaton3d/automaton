#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <assert.h>
#include "automaton.cuh"

/*
 * Lets the two columns equal. 
 */
__global__ void replicate(Cell* lattice)
{
	long xyz = blockDim.x * blockIdx.x + threadIdx.x;
	if (xyz < SIDE3)
	{
		// Calculate pointers
		//
		Cell* draft = lattice + xyz;
		Cell* stable = draft + SIDE2 * SIDE3;
		//
		// Not last tick?
		//
		if (draft->t % LIGHT != 0)
			return;
		//
		// Scan the two columns
		//
		for (int v = 0; v < SIDE2; v++)
		{
			// Copy only variables that changed in compare()
			//
			stable->a = draft->a;
			stable->f = draft->f;
			stable->code = draft->code;
			assert(stable->u == draft->u);
			assert(stable->v == draft->v);
			//
			// Next register
			//
			stable = nextV(stable);
			draft = nextV(draft);
		}
	}
}
