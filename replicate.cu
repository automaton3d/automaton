#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void replicate(struct Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		struct Cell* cell = lattice + id;
		if (!cell->active)
			cell = cell->h;
		for (int v = 0; v < SIDE2; v++)
		{
			cell->f = cell->h->f;
			cell->b = cell->h->b;
			COPY(cell->pole, cell->h->pole);
			cell->code = cell->h->code;
			cell->noise |= cell->h->noise;
			cell = cell->v;
		}
	}
}
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void replicate(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell *cell = lattice + id;
		Cell* active_stack, *passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		//
		for (int v = 0; v < SIDE2; v++)
		{
			active_stack->f = passive_stack->f;
			active_stack->b = passive_stack->b;
			active_stack->code = passive_stack->code;
			active_stack->noise |= passive_stack->noise;
			COPY(active_stack->pole, passive_stack->pole);
			//
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	}
}
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void replicate(Cell* lattice)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell *cell = lattice + id;
		Cell* active_stack, *passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		//
		for (int v = 0; v < SIDE2; v++)
		{
			active_stack->f = passive_stack->f;
			active_stack->b = passive_stack->b;
			active_stack->code = passive_stack->code;
			active_stack->noise |= passive_stack->noise;
			COPY(active_stack->pole, passive_stack->pole);
			COPY(active_stack->p, passive_stack->p);	// pacth não é isso....
			//
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	}
}
