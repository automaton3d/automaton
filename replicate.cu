#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include "automaton.h"

__global__ void replicate(Cell* lattice)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < SIDE3)
	{
		Cell *cell = lattice + id;
		Cell* active_stack, *passive_stack;
		if (cell->active)
		{
			active_stack = cell;
			passive_stack = cell + SIDE3 * SIDE2;
		}
		else
		{
			passive_stack = cell;
			active_stack = cell + SIDE3 * SIDE2;
		}
		//
		for (int v = 0; v < SIDE2; v++)
		{
			active_stack->f = passive_stack->f;
			active_stack->b = passive_stack->b;
			active_stack->code = passive_stack->code;
			active_stack->noise = passive_stack->noise;
			COPY(active_stack->pole, passive_stack->pole);
			COPY(active_stack->p, passive_stack->p);
			//
			active_stack = nextV(active_stack);
			passive_stack = nextV(passive_stack);
		}
	}
}
