#include "hip/hip_runtime.h"
#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#define GLEW_STATIC

#include <stdio.h>
#include <stdlib.h>
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "callbacks.h"
#include "automaton.h"
#include "cglm/mat4.h"
#include "cglm/affine.h"
#include "cglm/cglm.h"
#include "cglm/call.h"
#include "cglm/cam.h"
#include "cglm/vec3.h"

#include "automaton.h"

const char* vertexShaderSource = "#version 460 core\n"
"layout(location = 0) in vec3 aPos;\n"
"layout(location = 1) in vec3 aOffset;\n"
"layout(location = 2) in vec3 aColor;\n"
"out vec3 fColor;\n"
"uniform mat4 projection;\n"
"uniform mat4 view;\n"
"uniform mat4 model;\n"
"void main()\n"
"{\n"
"	gl_Position = projection * view * model * vec4(aPos + aOffset, 1.0);\n"
"	fColor = aColor;\n"
"}\0";

unsigned int vertexShader;

const char* fragmentShaderSource = "#version 460 core\n"
"in vec3 fColor;\n"
"out vec4 FragColor;\n"
"void main()\n"
"{\n"
"	if(fColor==vec3(0.6,0.6,0.7))"
"		FragColor = vec4(0.6,0.6,0.7,0.05);\n"
"	else\n"
"		FragColor = vec4(fColor, 1);\n"
"}\0";

unsigned int shaderProgram;
unsigned int vao;

mat4 model = GLM_MAT4_IDENTITY_INIT;
mat4 view = GLM_MAT4_IDENTITY_INIT;
mat4 projection;

// Camera

float yaw = -90;
float pitch = 0;

vec3 cameraPos;
vec3 cameraFront;
vec3 cameraUp = { 0.0f, 1.0f, 0.0f };

Cell* host_lattice;
Cell* dev_lattice;

struct hipGraphicsResource* cuda_resource;
unsigned int colorVBO;
vec3 colors[SIDE3];
size_t num_bytes;

hipError_t cudaStatus;
DWORD start;

void initCuda()
{
	size_t heapsize = 2 * SIDE2 * SIDE3 * sizeof(Cell);
	printf("Program launched: SIDE=%d, sizeof=%zd, heap=%zd\n", SIDE, sizeof(Cell), heapsize); fflush(stdout);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("global memory: \n\ttotal=%zd\n\tfree=%zd", total, free);
	//
	cudaStatus = hipMalloc((void**)&dev_lattice, heapsize);
	if (cudaStatus != hipSuccess)
	{
		perror("hipMalloc failed");
		exit(1);
	}
	hipMemGetInfo(&free, &total);
	printf("\n\tused=%zd\n", total - free);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("ck=%d\n", prop.concurrentKernels);
	fflush(stdout);
	//
	// Host memory allocation
	//
	host_lattice = (Cell*)malloc(heapsize);
	if (host_lattice == NULL)
	{
		perror("host ram unavailable");
		exit(1);
	}
	hologram << <GRID1, BLOCK1 >> > (dev_lattice);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		puts("KERNEL error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
}

void closeApp()
{
	cudaStatus = hipGraphicsUnregisterResource(cuda_resource);
	if (cudaStatus != hipSuccess)
	{
		puts("unregister error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
	hipFree(dev_lattice);
	hipDeviceReset();
	free(host_lattice);
	printf("finished.\n");
}

void updateCamera()
{
	cameraFront[0] = cos(glm_rad(yaw)) * cos(glm_rad(pitch));
	cameraFront[1] = sin(glm_rad(pitch));
	cameraFront[2] = sin(glm_rad(yaw)) * cos(glm_rad(pitch));
	glm_normalize(cameraFront);
	cameraPos[0] = -SIDE * cameraFront[0];
	cameraPos[1] = -SIDE * cameraFront[1];
	cameraPos[2] = -SIDE * cameraFront[2];
	//
	// Assemble the view matrix
	//
	vec3 sum;
	glm_vec3_add(cameraPos, cameraFront, sum);
	glm_lookat(cameraPos, sum, cameraUp, view);
	//
	int loc = glGetUniformLocation(shaderProgram, "view");
	glUniformMatrix4fv(loc, 1, GL_FALSE, view[0]);
}

int initOpenGL(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitWindowSize(800, 600);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
	char s[] = "                              ";
	sprintf(s, "Automaton %dx%dx%dx%dx%d", SIDE, SIDE, SIDE, SIDE2, 2);
	glutCreateWindow(s);
	printf("\tGPU: %s\n", glGetString(GL_VERSION));
	GLenum err = glewInit();
	if (GLEW_OK != err)
	{
		printf("glew init %s\n", glewGetErrorString(err)); fflush(stdout);
		return -1;
	}
	//
	// Create shaders
	//
	int success;
	char infoLog[512];
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		perror("Vertex shader failed.\n");
		return -1;
	}
	unsigned int fragmentShader;
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		perror("Vertex shader failed.\n");
		return -1;
	}
	//
	// Link shaders
	//
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);
	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if (!success)
	{
		glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
		perror("Linking error.\n");
		return -1;
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);
	//
	// Initialize common point
	//
	vec3 pointVertices = { 0, 0, 0 };
	//
	// Initialize instance translations
	//
	vec3 translations[SIDE3];
	int index = 0;
	for (int z = 0; z < SIDE; z++)
	{
		for (int y = 0; y < SIDE; y++)
		{
			for (int x = 0; x < SIDE; x++)
			{
				translations[index][0] = x / (float)SIDE - 0.5f;
				translations[index][1] = y / (float)SIDE - 0.5f;
				translations[index][2] = z / (float)SIDE - 0.5f;
				index++;
			}
		}
	}
	//
	// Initialize instance colors
	//
	index = 0;
	for (int z = 0; z < SIDE; z++)
	{
		for (int y = 0; y < SIDE; y++)
		{
			for (int x = 0; x < SIDE; x++)
			{
				colors[index][0] = 0.5f;
				colors[index][1] = 0.5f;
				colors[index][2] = 0.8f;
				index++;
			}
		}
	}
	glUseProgram(shaderProgram);
	//
	// Create vbos
	//
	unsigned int pointVBO, positionVBO, colorVBO;
	glGenBuffers(1, &pointVBO);
	glBindBuffer(GL_ARRAY_BUFFER, pointVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vec3), pointVertices, GL_STATIC_DRAW);
	glGenBuffers(1, &positionVBO);
	glBindBuffer(GL_ARRAY_BUFFER, positionVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vec3) * SIDE3, &translations[0], GL_STATIC_DRAW);
	glGenBuffers(1, &colorVBO);
	glBindBuffer(GL_ARRAY_BUFFER, colorVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vec3) * SIDE3, &colors[0], GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	// Create vao
	//
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);
	//
	// Add vbos
	//
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, pointVBO);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3), (void*)0);
	//
	glEnableVertexAttribArray(1);
	glBindBuffer(GL_ARRAY_BUFFER, positionVBO);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(vec3), (void*)0);
	glVertexAttribDivisor(1, 1);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	glEnableVertexAttribArray(2);
	glBindBuffer(GL_ARRAY_BUFFER, colorVBO);
	glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, sizeof(vec3), (void*)0);
	glVertexAttribDivisor(2, 1);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	// Voxel size
	//
	glPointSize(4);	
	//
	// Connect color vbo to cuda
	//
	cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_resource, colorVBO, cudaGraphicsMapFlagsNone);
	if (cudaStatus != hipSuccess)
	{
		puts("connect error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
	//
	// Create the projection matrix
	//
	glUseProgram(shaderProgram);
	glm_ortho(-1, 1, -1, 1, -1.0f, 200, projection);
	//
	// Depth and transparency
	//
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND); 
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	//
	// Upadte uniforms
	//
	int loc = glGetUniformLocation(shaderProgram, "projection");
	glUniformMatrix4fv(loc, 1, GL_FALSE, projection[0]);
	vec3 v = { 0, 0, 0 };
	glm_translate(model, v);
	loc = glGetUniformLocation(shaderProgram, "model");
	glUniformMatrix4fv(loc, 1, GL_FALSE, model[0]);
	updateCamera();
	//
	glClearColor(0.1f, 0.2f, 0.2f, 1.0f);
	glutKeyboardFunc(&keyboard);
	glutIdleFunc(&animation);
	//
	// Draw first frame 
	//
	glUseProgram(shaderProgram);
	//
	// Set display function
	//
	glutDisplayFunc(&display);
	//
	return 0;
}

/* 
 * Program entry point.
 */
int main(int argc, char** argv)
{
	initCuda();
	//printResults(true);
	initOpenGL(argc, argv);
	start = GetTickCount();
	glutMainLoop();
	return EXIT_SUCCESS;
}
