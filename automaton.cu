#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "automaton.h"

struct Cell* lattice;

__global__ void initCA(struct Cell* lattice)
{
    // Create lattice
    //
    size_t idx = blockIdx.x* blockDim.x + threadIdx.x;
    int h = idx % 2;
    int v = idx >> 1;
    size_t offset = SIDE3*v + SIDE2*SIDE3*h;
    struct Cell* cell = lattice + offset;
    int floor = idx >> 1;
    int i = 0;
    bool active = h == 0;
    for (int z = 0; z < SIDE; z++)
        for (int y = 0; y < SIDE; y++)
            for (int x = 0; x < SIDE; x++)
            {
                cell->active = active;
                cell->t = 0;
                cell->noise = i;
                cell->f = 0;
                RESET(cell->p);
                RESET(cell->s);
                if (z == 0 && (x + SIDE*y) == floor)
                {
                    cell->f = 1;
                    if (x < SIDE / 2)
                    {
                        cell->d = true;
                    }
                    else
                    {
                        cell->d = false;
                    }
                    //
                    unsigned char tiling = (x % 2) ^ (y % 2);
                    if (tiling)
                    {
                        cell->c = 0;
                        cell->w = false;
                        cell->q = true;
                    }
                    else
                    {
                        cell->c = 7;
                        cell->w = true;
                        cell->q = false;
                    }
                    //
                    // Initialize spin
                    //
                    if (x == SIDE - 1)
                    {
                        cell->s[2] = (cell->d) ? -SIDE / 2 : +SIDE / 2;
                        cell->p[2] = (i % 2) ? +SIDE/2 : -SIDE/2;
                    }
                    else
                    {
                        switch (i % 6)
                        {
                            case 0:
                                cell->s[0] = +SIDE / 2;
                                cell->p[1] = +SIDE / 2;
                                break;
                            case 1:
                                cell->s[0] = -SIDE / 2;
                                cell->p[1] = -SIDE / 2;
                                break;
                            case 2:
                                cell->s[1] = +SIDE / 2;
                                cell->p[2] = +SIDE / 2;
                                break;
                            case 3:
                                cell->s[1] = -SIDE / 2;
                                cell->p[2] = -SIDE / 2;
                                break;
                            case 4:
                                cell->s[2] = +SIDE / 2;
                                cell->p[0] = +SIDE / 2;
                                break;
                            case 5:
                                cell->s[2] = -SIDE / 2;
                                cell->p[0] = -SIDE / 2;
                                break;
                        }
                    }
                }
                //
                cell->sine = 0;
                cell->cosine = SIDE / 2;
                //
                if(x == SIDE-1)
                    cell->px = cell - (SIDE - 1) * CELL;
                else
                    cell->px = cell + CELL;
                //
                if(x == 0)
                    cell->nx = cell + (SIDE - 1) * CELL;
                else
                    cell->nx = cell - CELL;
                if(y == SIDE-1)
                    cell->py = cell - (SIDE - 1) * SIDE * CELL;
                else
                    cell->py = cell + SIDE * CELL;
                if(y == 0)
                    cell->ny = cell + (SIDE - 1) * CELL;
                else
                    cell->ny = cell - SIDE * CELL;
                if(z == SIDE-1)
                    cell->pz = cell -(SIDE - 1) * SIDE2 * CELL;
                else
                    cell->pz = cell + SIDE2 * CELL;
                if(z == 0)
                    cell->nz = cell + (SIDE - 1) * SIDE2 * CELL;
                else
                    cell->nz = cell - SIDE2 * CELL;
                if(floor == SIDE2-1)
                    cell->v = cell - (SIDE2 - 1) * SIDE3 * CELL;
                else
                    cell->v = cell + SIDE3 * CELL;
                //
                // Neighbor
                //
                if(cell->active)
                    cell->h = cell + SIDE3 * SIDE2 * CELL;
                else
                    cell->h = cell - SIDE3 * SIDE2 * CELL;
                //
                // Elevator
                //
                if(floor == SIDE2-1)
                    cell->v = cell - (SIDE2 - 1) * SIDE3 * CELL;
                else
                    cell->v = cell + SIDE3 * CELL;
                //
                i++;
                cell++;
            }
}

struct Cell* dev_lattice;

void initApp()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("free=%lld, total=%lld\n", free, total);
    //
    size_t heapsize = 2 * SIDE2 * SIDE3 * sizeof(struct Cell);
    printf("Program launched: %d, %d, %zd, %zd\n", SIDE2, SIDE3, sizeof(struct Cell), heapsize); fflush(stdout);
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&dev_lattice, heapsize);
    if (cudaStatus != hipSuccess)
    {
        perror("hipMalloc failed");
        exit(1);
    }

    hipMemGetInfo(&free, &total);
    printf("free=%lld, total=%lld\n", free, total);



    printf("device lattice allocated\n"); fflush(stdout);
    initCA << <GRIDDIM, BLOCKDIM >> > (dev_lattice);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        puts("KERNEL error");
        perror(hipGetErrorString(cudaStatus));
        exit(1);
    }
    //
    lattice = (struct Cell*)malloc(heapsize);
    if (lattice == NULL)
    {
        perror("host ram unavailable");
        exit(1);
    }
    printf("host lattice allocated\n"); fflush(stdout);
    hipMemcpy(lattice, dev_lattice, 2 * SIDE2 * SIDE3 * sizeof(struct Cell), hipMemcpyDeviceToHost);
 }

void closeApp()
{
    hipFree(dev_lattice);
    hipDeviceReset();
    free(lattice);
    printf("finished.\n");
}


__global__ void interact()
{
    if (threadIdx.x < SIDE && threadIdx.y < SIDE && blockIdx.x < SIDE)
    {
        // Execute
    }
}

int main()
{
    initApp();
    for (int i = 0; i < 100; i++)
    {
        expand<<<GRIDDIM, BLOCKDIM >>>(lattice);
    }
    struct Cell* cell = lattice;
    for (int h = 0; h < 2; h++)
        for (int v = 0; v < SIDE2; v++)
            for (int z = 0; z < SIDE; z++)
                for (int y = 0; y < SIDE; y++)
                    for (int x = 0; x < SIDE; x++)
                    {
                        if (cell->active && !ISNULL(cell->p))
                        {
                            printf("%d, %d, %d, v=%d, h=%d noise=%d p=(%d,%d,%d)\n", x, y, z, v, h, cell->noise, cell->p[0], cell->p[1], cell->p[2]);
                            fflush(stdout);
                        }
                        cell++;
                    }
    closeApp();
    return 0;
}
