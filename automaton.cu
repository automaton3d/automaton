#include "hip/hip_runtime.h"
#pragma comment(lib, "C:\\GL\\GLUT\\lib\\x64\\freeglut.lib")

#define GLEW_STATIC

#include <stdio.h>
#include <stdlib.h>
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "callbacks.h"
#include "automaton.h"
#include "cglm/mat4.h"
#include "cglm/affine.h"
#include "cglm/cglm.h"
#include "cglm/call.h"
#include "cglm/cam.h"
#include "cglm/vec3.h"

#include "automaton.h"

const char* vertexShaderSource = "#version 460 core\n"
"layout(location = 0) in vec3 aPos;\n"
"layout(location = 1) in vec3 aColor;\n"
"out vec4 fColor;\n"
"uniform mat4 projection;\n"
"uniform mat4 view;\n"
"uniform mat4 model;\n"

"void main()\n"
"{\n"
"	vec3 aOffset;"
#if ORDER==4
"	aOffset[0] = (gl_InstanceID & 15) * 8 - 64;\n"
"	aOffset[1] = ((gl_InstanceID >> 4) & 15) * 8 - 64;\n"
"	aOffset[2] = (gl_InstanceID >> 8) * 8 - 64;\n"
#else  if ORDER==5
"	aOffset[0] = (gl_InstanceID & 31) * 16 - 256;\n"
"	aOffset[1] = ((gl_InstanceID >> 5) & 31) * 16 - 256;\n"
"	aOffset[2] = (gl_InstanceID >> 10) * 16 - 256;\n"
#endif
"	gl_Position = projection * view * model * vec4(aPos + aOffset, 1.0);\n"
"	if(aColor.x==0.6 && aColor.y==0.6 && aColor.z==0.8)\n"
"		fColor = vec4(aColor, 0.2);\n"
"	else\n"
"		fColor = vec4(aColor, 1.0);\n"
"}\0";

unsigned int vertexShader;

const char* fragmentShaderSource = "#version 460 core\n"
"in vec4 fColor;\n"
"out vec4 FragColor;\n"
"void main()\n"
"{\n"
"	FragColor = fColor;\n"
"}\0";

unsigned int shaderProgram;
unsigned int vao;

mat4 model = GLM_MAT4_IDENTITY_INIT;
mat4 view = GLM_MAT4_IDENTITY_INIT;
mat4 projection;

// Camera

float yaw = -120;// -90;
float pitch = 45;// 0;

vec3 cameraPos;
vec3 cameraFront;
vec3 cameraUp = { 0.0f, 1.0f, 0.0f };

Cell* host_lattice;
Cell* dev_lattice;

struct hipGraphicsResource* cuda_resource;
unsigned int colorVBO;
size_t num_bytes;

hipError_t cudaStatus;
DWORD start;

vec3 colors[SIDE3];

GLfloat cubeVertices[] = 
{
	-1.0f,-1.0f,-1.0f, // triangle 1 : begin
	-1.0f,-1.0f, 1.0f,
	-1.0f, 1.0f, 1.0f, // triangle 1 : end
	1.0f, 1.0f,-1.0f, // triangle 2 : begin
	-1.0f,-1.0f,-1.0f,
	-1.0f, 1.0f,-1.0f, // triangle 2 : end
	1.0f,-1.0f, 1.0f,
	-1.0f,-1.0f,-1.0f,
	1.0f,-1.0f,-1.0f,
	1.0f, 1.0f,-1.0f,
	1.0f,-1.0f,-1.0f,
	-1.0f,-1.0f,-1.0f,
	-1.0f,-1.0f,-1.0f,
	-1.0f, 1.0f, 1.0f,
	-1.0f, 1.0f,-1.0f,
	1.0f,-1.0f, 1.0f,
	-1.0f,-1.0f, 1.0f,
	-1.0f,-1.0f,-1.0f,
	-1.0f, 1.0f, 1.0f,
	-1.0f,-1.0f, 1.0f,
	1.0f,-1.0f, 1.0f,
	1.0f, 1.0f, 1.0f,
	1.0f,-1.0f,-1.0f,
	1.0f, 1.0f,-1.0f,
	1.0f,-1.0f,-1.0f,
	1.0f, 1.0f, 1.0f,
	1.0f,-1.0f, 1.0f,
	1.0f, 1.0f, 1.0f,
	1.0f, 1.0f,-1.0f,
	-1.0f, 1.0f,-1.0f,
	1.0f, 1.0f, 1.0f,
	-1.0f, 1.0f,-1.0f,
	-1.0f, 1.0f, 1.0f,
	1.0f, 1.0f, 1.0f,
	-1.0f, 1.0f, 1.0f,
	1.0f,-1.0f, 1.0f
};

void initCuda()
{
	size_t heapsize = 2 * SIDE2 * SIDE3 * sizeof(Cell);
	printf("Program launched: SIDE=%d, sizeof=%zd, heap=%zd\n", SIDE, sizeof(Cell), heapsize); fflush(stdout);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("global memory: \n\ttotal=%zd\n\tfree=%zd", total, free);
	//
	cudaStatus = hipMalloc((void**)&dev_lattice, heapsize);
	if (cudaStatus != hipSuccess)
	{
		perror("hipMalloc failed");
		exit(1);
	}
	hipMemGetInfo(&free, &total);
	printf("\n\tused=%zd\n", total - free);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("ck=%d\n", prop.concurrentKernels);
	fflush(stdout);
	//
	// Host memory allocation
	//
	host_lattice = (Cell*)malloc(heapsize);
	if (host_lattice == NULL)
	{
		perror("host ram unavailable");
		exit(1);
	}
	hologram << <GRID, BLOCK >> > (dev_lattice);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		puts("KERNEL error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
}

void closeApp()
{
	cudaStatus = hipGraphicsUnregisterResource(cuda_resource);
	if (cudaStatus != hipSuccess)
	{
		puts("unregister error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
	hipFree(dev_lattice);
	hipDeviceReset();
	free(host_lattice);
	printf("finished.\n");
}

void updateCamera()
{
	cameraFront[0] = cos(glm_rad(yaw)) * cos(glm_rad(pitch));
	cameraFront[1] = sin(glm_rad(pitch));
	cameraFront[2] = sin(glm_rad(yaw)) * cos(glm_rad(pitch));
	glm_normalize(cameraFront);
	cameraPos[0] = -SIDE * cameraFront[0];
	cameraPos[1] = -SIDE * cameraFront[1];
	cameraPos[2] = -SIDE * cameraFront[2];
	//
	// Assemble the view matrix
	//
	vec3 sum;
	glm_vec3_add(cameraPos, cameraFront, sum);
	glm_lookat(cameraPos, sum, cameraUp, view);
	//
	int loc = glGetUniformLocation(shaderProgram, "view");
	glUniformMatrix4fv(loc, 1, GL_FALSE, view[0]);
}

int initOpenGL(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitWindowSize(800, 600);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
	char s[] = "                              ";
	sprintf(s, "Automaton %dx%dx%dx%dx%d", SIDE, SIDE, SIDE, SIDE2, 2);
	glutCreateWindow(s);
	printf("\tGPU: %s\n", glGetString(GL_VERSION));
	GLenum err = glewInit();
	if (GLEW_OK != err)
	{
		printf("glew init %s\n", glewGetErrorString(err)); fflush(stdout);
		return -1;
	}
	//
	// Create shaders
	//
	int success;
	char infoLog[512];
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		perror("Vertex shader failed.\n");
		return -1;
	}
	unsigned int fragmentShader;
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		perror("Vertex shader failed.\n");
		return -1;
	}
	//
	// Link shaders
	//
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);
	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if (!success)
	{
		glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
		perror("Linking error.\n");
		return -1;
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);
	for (int i = 0; i < SIDE3; i++)
	{
		colors[i][0] = 0;
		colors[i][1] = 1;
		colors[i][2] = 1;
	}
	glUseProgram(shaderProgram);
	//
	// Create vbos
	//
	unsigned int cubeVBO;
	glGenBuffers(1, &cubeVBO);
	glBindBuffer(GL_ARRAY_BUFFER, cubeVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(cubeVertices), cubeVertices, GL_STATIC_DRAW);
	//
	glGenBuffers(1, &colorVBO);
	glBindBuffer(GL_ARRAY_BUFFER, colorVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vec3) * SIDE3, &colors[0], GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	// Create vao
	//
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);
	//
	// Add vbos
	//
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, cubeVBO);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	glEnableVertexAttribArray(1);
	glBindBuffer(GL_ARRAY_BUFFER, colorVBO);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, (void*)(0));
	glVertexAttribDivisor(1, 1);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//
	// Connect color vbo to cuda
	//
	cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_resource, colorVBO, cudaGraphicsMapFlagsNone);
	if (cudaStatus != hipSuccess)
	{
		puts("connect error");
		perror(hipGetErrorString(cudaStatus));
		exit(1);
	}
	//
	// Create the projection matrix
	//
	glUseProgram(shaderProgram);
	glm_ortho(-1, 1, -1, 1, -1.0f, 200, projection);
	//
	// Depth and transparency
	//
	//glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND); 
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	//glDepthFunc(GL_LESS);
	//
	// Upadte uniforms
	//
	int loc = glGetUniformLocation(shaderProgram, "projection");
	glUniformMatrix4fv(loc, 1, GL_FALSE, projection[0]);
	#if ORDER == 4
	vec3 scale = { 0.008, 0.008, 0.008 };
	#else if ORDER == 5
	vec3 scale = { 0.003, 0.003, 0.003 };
	#endif
	glm_scale(model, scale);
	loc = glGetUniformLocation(shaderProgram, "model");
	glUniformMatrix4fv(loc, 1, GL_FALSE, model[0]);
	//
	// Update camera
	//
	updateCamera();
	glClearColor(0.1, 0.2, 0.2, 1.0);
	glUseProgram(shaderProgram);
	//
	// Define callback routines
	//
	glutKeyboardFunc(&keyboard);
	glutSpecialFunc(&specialKeys);
	glutIdleFunc(&animation);
	glutDisplayFunc(&display);
	//
	return 0;
}

/* 
 * Program entry point.
 */
int main(int argc, char** argv)
{
	initCuda();
	//printResults(true);
	initOpenGL(argc, argv);
	start = GetTickCount();
	glutMainLoop();
	return EXIT_SUCCESS;
}
