#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//#include <stdint.h>
//#include <iostream>

#include "automaton.cuh"

__device__ bool compareCells(Cell* cell, int xyz)
{
    int x = xyz & (SIDE - 1);
    int y = (xyz >> ORDER) & (SIDE - 1);
    int z = (xyz >> (2 * ORDER));
    //
    /*
    if (cell->t != 0)
        return false;
    if(cell->noise != cell->floor)
        return false;
//    cell->b = 0;
    if (cell->synch != 0)
        return false;
//    cell->ctrl = 0;
//    cell->flash = 0;
    if(cell->sine != 0)
        return false;
    if(cell->cosine != SIDE / 2)
        return false;
    */
    if(!ISNULL(cell->o))
        return false;
    //
    // Cell belongs to the hologram?
    //
    /*
    if (z == SIDE / 2 && (x + SIDE * y) == cell->floor)
    {
        // Initialize charges, spin and momentum
        //
        if(cell->f != 1)
            return false;
        if (x < SIDE / 2 && (cell->charge & D_MASK) == 0)
            return false;
        //
        unsigned char tiling = (x % 2) ^ (y % 2);
        if (tiling)
        {
            if ((cell->charge & Q_MASK) == 0)
                return false;
        }
        else
        {
            if ((cell->charge & C_MASK) != C_MASK)
                return false;
            if ((cell->charge & W_MASK) == 0)
                return false;
        }
        //
        // Test spin and momentum
        //
        char s[3] = { 0,0,0 }, p[3] = { 0,0,0 };
        if (x == SIDE - 1)
        {
            // Enforce monopole
            //
            s[2] = (cell->charge & D_MASK) ? -SIDE / 2 : +SIDE / 2;
            p[2] = (cell->floor % 2) ? +SIDE / 2 : -SIDE / 2;
            if (cell->s[2] != s[2])
                return false;
            if (cell->p[2] != p[2])
                return false;
        }
        else
        {
            // Isotropic distribution
            //
            switch ((x + SIDE * y) % 6)
            {
            case 0:
                p[0] = x - SIDE / 2;
                p[1] = y - SIDE / 2;
                p[2] = SIDE / 2;
                //
                s[0] = y - SIDE / 2;
                s[1] = x - SIDE / 2;
                s[2] = -SIDE / 2;
                break;
            case 1:
                p[0] = SIDE / 2;
                p[1] = x - SIDE / 2;
                p[2] = y - SIDE / 2;
                //
                s[0] = -SIDE / 2;
                s[1] = y - SIDE / 2;
                s[2] = x - SIDE / 2;
                break;
            case 2:
                p[0] = y - SIDE / 2;
                p[1] = SIDE / 2;
                p[2] = x - SIDE / 2;
                //
                s[0] = x - SIDE / 2;
                s[1] = y - SIDE / 2;
                s[2] = SIDE / 2;
                break;
            case 3:
                p[0] = y - SIDE / 2;
                p[1] = x - SIDE / 2;
                p[2] = -SIDE / 2;
                //
                s[0] = x - SIDE / 2;
                s[1] = y - SIDE / 2;
                s[2] = SIDE / 2;
                break;
            case 4:
                p[0] = -SIDE / 2;
                p[1] = y - SIDE / 2;
                p[2] = x - SIDE / 2;
                //
                s[0] = SIDE / 2;
                s[1] = x - SIDE / 2;
                s[2] = y - SIDE / 2;
                break;
            case 5:
                p[0] = x - SIDE / 2;
                p[1] = -SIDE / 2;
                p[2] = y - SIDE / 2;
                //
                s[0] = y - SIDE / 2;
                s[1] = SIDE / 2;
                s[2] = x - SIDE / 2;
                break;
            }
            if (cell->s[0] != s[0])
                return false;
            if (cell->s[1] != s[1])
                return false;
            if (cell->s[2] != s[2])
                return false;
            if (cell->p[0] != p[0])
                return false;
            if (cell->p[1] != p[1])
                return false;
            if (cell->p[2] != p[2])
                return false;
        }
    }
    else
    {
        // Does not belong to the hologram
        //
        if (!ISNULL(cell->p))
            return false;
        if (!ISNULL(cell->s))
            return false;
        if(cell->charge != 0)
            return false;
        if (cell->f != 0)
            return false;
    }
    */
    return true;
}

__global__ void poincare(Cell* lattice, int *count)
{
    // Calculate 3d index
    //
    bool test = false;
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        // Build one lattice
        //
        Cell* cell = lattice + xyz;
        for (int floor = 0; floor < SIDE2; floor++)
        {
            if (!compareCells(cell, xyz))
            {
                test = true; 
                break;
            }
            cell = nextV(cell);
        }
        if (test)
        {
            atomicAdd(count, 1);
        }
    }
}

