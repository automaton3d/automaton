#include "hip/hip_runtime.h"
/*
 * tree2.c
 *
 *  Created on: 13 de abr de 2017
 *      Author: Alexandre
 */

#include "automaton.h"

__device__ char dirs[6][3] = { { +1, 0, 0 }, { -1, 0, 0 }, { 0, +1, 0 }, { 0, -1, 0}, { 0, 0, +1 }, { 0, 0, -1 } };

 /*
  * Tests whether the direction dir is a valid path in the visit-once-tree.
  */
__device__ bool isAllowed(int dir, char p[3], unsigned char d0)
{
	int d1 = p[0] * p[0] + p[1] * p[1] + p[2] * p[2];
	int x = p[0] + dirs[dir][0];
	int y = p[1] + dirs[dir][1];
	int z = p[2] + dirs[dir][2];
	int d2 = x * x + y * y + z * z;
	if (d2 <= d1)
		return false;
	//
	// Wrapping test
	//
	if (x == SIDE / 2 + 1 || x == -SIDE / 2 || y == SIDE / 2 + 1 || y == -SIDE / 2 || z == SIDE / 2 + 1 || z == -SIDE / 2)
		return false;
	//
	// Root
	//
	int level = abs(x) + abs(y) + abs(z);
	if (level == 1)
		return true;
	//
	// x axis
	//
	if (x > 0 && y == 0 && z == 0 && dir == 0)
		return true;
	else if (x < 0 && y == 0 && z == 0 && dir == 1)
		return true;
	//
	// y axis
	//
	else if (x == 0 && y > 0 && z == 0 && dir == 2)
		return true;
	else if (x == 0 && y < 0 && z == 0 && dir == 3)
		return true;
	//
	// z axis
	//
	else if (x == 0 && y == 0 && z > 0 && dir == 4)
		return true;
	else if (x == 0 && y == 0 && z < 0 && dir == 5)
		return true;
	//
	// xy plane
	//
	else if (x > 0 && y > 0 && z == 0)
	{
		if (level % 2 == 1)
			return (dir == 0 && d0 == 2);
		else
			return (dir == 2 && d0 == 0);
	}
	else if (x < 0 && y > 0 && z == 0)
	{
		if (level % 2 == 1)
			return (dir == 1 && d0 == 2);
		else
			return (dir == 2 && d0 == 1);
	}
	else if (x > 0 && y < 0 && z == 0)
	{
		if (level % 2 == 1)
			return (dir == 0 && d0 == 3);
		else
			return (dir == 3 && d0 == 0);
	}
	else if (x < 0 && y < 0 && z == 0)
	{
		if (level % 2 == 1)
			return (dir == 1 && d0 == 3);
		else
			return (dir == 3 && d0 == 1);
	}
	//
	// yz plane
	//
	else if (x == 0 && y > 0 && z > 0)
	{
		if (level % 2 == 0)
			return (dir == 4 && d0 == 2);
		else
			return (dir == 2 && d0 == 4);
	}
	else if (x == 0 && y < 0 && z > 0)
	{
		if (level % 2 == 0)
			return (dir == 4 && d0 == 3);
		else
			return (dir == 3 && d0 == 4);
	}
	else if (x == 0 && y > 0 && z < 0)
	{
		if (level % 2 == 0)
			return (dir == 5 && d0 == 2);
		else
			return (dir == 2 && d0 == 5);
	}
	else if (x == 0 && y < 0 && z < 0)
	{
		if (level % 2 == 0)
			return (dir == 5 && d0 == 3);
		else
			return (dir == 3 && d0 == 5);
	}
	//
	// zx plane
	//
	else if (x > 0 && y == 0 && z > 0)
	{
		if (level % 2 == 1)
			return (dir == 4 && d0 == 0);
		else
			return (dir == 0 && d0 == 4);
	}
	else if (x < 0 && y == 0 && z > 0)
	{
		if (level % 2 == 1)
			return (dir == 4 && d0 == 1);
		else
			return (dir == 1 && d0 == 4);
	}
	else if (x > 0 && y == 0 && z < 0)
	{
		if (level % 2 == 1)
			return (dir == 5 && d0 == 0);
		else
			return (dir == 0 && d0 == 5);
	}
	else if (x < 0 && y == 0 && z < 0)
	{
		if (level % 2 == 1)
			return (dir == 5 && d0 == 1);
		else
			return (dir == 1 && d0 == 5);
	}
	else
	{
		// Spirals
		//
		int x0 = x + SIDE / 2;
		int y0 = y + SIDE / 2;
		int z0 = z + SIDE / 2;
		//
		switch (level % 3)
		{
		case 0:
			if (x0 != SIDE / 2 && y0 != SIDE / 2)
				return (z0 > SIDE / 2 && dir == 4) || (z0 < SIDE / 2 && dir == 5);
			break;
		case 1:
			if (y0 != SIDE / 2 && z0 != SIDE / 2)
				return (x0 > SIDE / 2 && dir == 0) || (x0 < SIDE / 2 && dir == 1);
			break;
		case 2:
			if (x0 != SIDE / 2 && z0 != SIDE / 2)
				return (y0 > SIDE / 2 && dir == 2) || (y0 < SIDE / 2 && dir == 3);
			break;
		}
	}
	return false;
}
