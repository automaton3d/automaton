#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "automaton.h"

/*
 * Tests whether the direction dir is a valid path in the visit-once-tree.
 */
__device__ bool isAllowed(int dir, char vdir[3], char o[3], unsigned char d0)
{
    // Calculate new origin vector
    //
    int x = o[0] + vdir[0];
    int y = o[1] + vdir[1];
    int z = o[2] + vdir[2];
    //
    // Test for expansion
    //
    int d1 = MOD2(o);
    int d2 = x * x + y * y + z * z;
    if (d2 <= d1)
        return false;
    //
    // Wrapping test
    //
    if (x == S + 1 || x == -S || y == S + 1 || y == -S || z ==S + 1 || z == -S)
        return false;
    //
    // Root allows all six directions
    //
    int level = abs(x) + abs(y) + abs(z);
    if (level == 1)
        return true;
    //
    // x axis
    //
    if (x > 0 && y == 0 && z == 0 && dir == 0)
        return true;
    else if (x < 0 && y == 0 && z == 0 && dir == 1)
        return true;
    //
    // y axis
    //
    else if (x == 0 && y > 0 && z == 0 && dir == 2)
        return true;
    else if (x == 0 && y < 0 && z == 0 && dir == 3)
        return true;
    //
    // z axis
    //
    else if (x == 0 && y == 0 && z > 0 && dir == 4)
        return true;
    else if (x == 0 && y == 0 && z < 0 && dir == 5)
        return true;
    //
    // xy plane
    //
    else if (x > 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 2);
        else
            return (dir == 2 && d0 == 0);
    }
    else if (x < 0 && y > 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 2);
        else
            return (dir == 2 && d0 == 1);
    }
    else if (x > 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 0 && d0 == 3);
        else
            return (dir == 3 && d0 == 0);
    }
    else if (x < 0 && y < 0 && z == 0)
    {
        if (level % 2 == 1)
            return (dir == 1 && d0 == 3);
        else
            return (dir == 3 && d0 == 1);
    }
    //
    // yz plane
    //
    else if (x == 0 && y > 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 2);
        else
            return (dir == 2 && d0 == 4);
    }
    else if (x == 0 && y < 0 && z > 0)
    {
        if (level % 2 == 0)
            return (dir == 4 && d0 == 3);
        else
            return (dir == 3 && d0 == 4);
    }
    else if (x == 0 && y > 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 2);
        else
            return (dir == 2 && d0 == 5);
    }
    else if (x == 0 && y < 0 && z < 0)
    {
        if (level % 2 == 0)
            return (dir == 5 && d0 == 3);
        else
            return (dir == 3 && d0 == 5);
    }
    //
    // zx plane
    //
    else if (x > 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 0);
        else
            return (dir == 0 && d0 == 4);
    }
    else if (x < 0 && y == 0 && z > 0)
    {
        if (level % 2 == 1)
            return (dir == 4 && d0 == 1);
        else
            return (dir == 1 && d0 == 4);
    }
    else if (x > 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 0);
        else
            return (dir == 0 && d0 == 5);
    }
    else if (x < 0 && y == 0 && z < 0)
    {
        if (level % 2 == 1)
            return (dir == 5 && d0 == 1);
        else
            return (dir == 1 && d0 == 5);
    }
    else
    {
        // Spirals
        //
        int x0 = x + S;
        int y0 = y + S;
        int z0 = z + S;
        //
        switch (level % 3)
        {
        case 0:
            if (x0 != S && y0 != S)
                return (z0 > S && dir == 4) || (z0 < S && dir == 5);
            break;
        case 1:
            if (y0 != S && z0 != S)
                return (x0 > S && dir == 0) || (x0 < S && dir == 1);
            break;
        case 2:
            if (x0 != S && z0 != S)
                return (y0 > S && dir == 2) || (y0 < S && dir == 3);
            break;
        }
    }
    return false;
}

__device__ Cell* getPointer(int dir, Cell *draft, char* vdir)
{
    Cell* neighbor = draft;
    switch (dir)
    {
    case 0:
        *vdir = +1;
        if (draft->wrap & 0x40)
            neighbor -= (SIDE - 1);
        else
            neighbor++;
        break;
    case 1:
        *vdir = -1;
        if (draft->wrap & 0x80)
            neighbor += (SIDE - 1);
        else
            neighbor--;
        break;
    case 2:
        *(++vdir) = +1;
        if (draft->wrap & 0x10)
            neighbor -= (SIDE2 - SIDE);
        else
            neighbor += SIDE;
        break;
    case 3:
        *(++vdir) = -1;
        if (draft->wrap & 0x20)
            neighbor += (SIDE2 - SIDE);
        else
            neighbor -= SIDE;
        break;
    case 4:
        *(++(++vdir)) = +1;
        if (draft->wrap & 0x04)
            neighbor -= (SIDE3 - SIDE2);
        else
            neighbor += SIDE2;
        break;
    case 5:
        *(++(++vdir)) = -1;
        if (draft->wrap & 0x08)
            neighbor = draft + (SIDE3 - SIDE2);
        else
            neighbor = draft - SIDE2;
        break;
    }
    return neighbor;
}

__device__ void spread(Cell* stable, Cell* draft, int floor)
{
    // Update tracking info
    //
    if (draft->ctrl > 0)
    {
        // Track decay
        //
        draft->phi *= (1 - 1 / (2 * draft->t));
        //
        // Minsky circle algorithm
        //
        int xNew = draft->cosine - (draft->sine >> SHIFT);
        int yNew = draft->sine + (draft->cosine >> SHIFT);
        draft->cosine = xNew;
        draft->sine = yNew;
        //
        draft->ctrl--;
    }
    //
    // Spread cell contents if not empty
    //
    if (draft->f > 0)
    {
        draft->t++; 
        //
        // Re-emmited?
        //
        if (stable->flash && ALIGNED(stable->o, stable->pole))
        {
            RESET(draft->o);
            draft->t = 0;
        }
        //
        // Explore von Neumann directions
        //
        Cell* neighbor;
        for (int dir = 0; dir < 6; dir++)
        {
            char vdir[3] = { 0, 0, 0 };
            neighbor = getPointer(dir, draft, (char*)vdir);
            //
            // Test if branch is legal
            //
            if(isAllowed(dir, vdir, draft->o, draft->dir))
            {
                // Superluminal signal spreads not synchronized
                //
                if (stable->flash)
                {
                    neighbor->flash = stable->flash - 1;
                }
                //
                // Bubble cells spread synchronized
                //
                if (draft->t * draft->t > draft->synch)
                {
                    neighbor->t = draft->t;
                    neighbor->dir = dir;
                    neighbor->f = stable->f;
                    neighbor->b = stable->b;
                    neighbor->charge = stable->charge;
                    //
                    neighbor->o[0] = stable->o[0] + vdir[0];
                    neighbor->o[1] = stable->o[1] + vdir[1];
                    neighbor->o[2] = stable->o[2] + vdir[2];
                    //
                    COPY(neighbor->s, stable->s);
                    COPY(neighbor->p, stable->p);
                    //
                    // Schedule for spherical evolution
                    //
                    neighbor->synch = LIGHT2 * MOD2(neighbor->o);
                    //
                    draft->f = 0;
                    RESET(draft->p);
                }
            }
        }
    }
}

__global__ void expand(Cell* lattice)
{
    long xyz = blockDim.x * blockIdx.x + threadIdx.x;
    if (xyz < SIDE3)
    {
        Cell* draft = lattice + xyz;
        Cell* stable = lattice + xyz + SIDE2 * SIDE3;
        if (draft->active)
        {
            Cell* temp = draft;
            draft = stable;
            stable = temp;
        }
        for (int v = 0; v < SIDE2; v++)
        {
            spread(stable, draft, stable->floor);
            //
            // Next register
            //
            draft = nextV(draft);
            stable = nextV(stable);
        }
    }
}
